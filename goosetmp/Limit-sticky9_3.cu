#include "hip/hip_runtime.h"
               __global__ void
               f03_calculator(int ioff_, int ni_, int nj_, double eps2, f03_jp_t *f03_jp_, f03_ip_t *f03_ip_, f03_result_t *f03_result_)
               {
                   extern __shared__ char smembuf_[];
                   int kbdim_ = blockDim.x;
                   
                   f03_jp_t * f03_jp_smem_ = (f03_jp_t *)smembuf_;
                   
                   f03_result_t * f03_result_smem_ = (f03_result_t *)smembuf_;
                   double dx_0_, dx_1_, dx_2_, r2, rinv, mrinv;
                   double pot_i_wcache_ = 0.0f;

                   int njdiv_ = gridDim.x;
                   int jbid_ = blockIdx.x;
                   int ibid_ = blockIdx.y;
                   int tid_ = threadIdx.x;
                   int npipe_ = (ni_ - 1) / 1 + 1;
                   int nvalidthread_ = npipe_ - kbdim_ * ibid_;
                   if (nvalidthread_ > kbdim_) {
                       nvalidthread_ = kbdim_;
                   }
                   int njsub_ = (nj_ - 1) / njdiv_ + 1;
                   int joff0_ = njsub_ * jbid_;
                   int joff1_ = joff0_ + njsub_;
                   if (joff1_ > nj_) {
                       joff1_ = nj_;
                   }
                   int jstride_ = 1;
                   if (nvalidthread_ <= kbdim_ / 2) jstride_ = 2;
if (nvalidthread_ <= kbdim_ / 4) jstride_ = 4;

                   int njhsub_ = kbdim_ / jstride_;
                   int jstart_ = tid_ / njhsub_;
                   int isrc_ = kbdim_ * ibid_ + tid_ % njhsub_;
                   int idst_ = njdiv_ * isrc_ + jbid_;
                   int icnt_ = ioff_ + isrc_;
                   for (int joff_ = joff0_; joff_ < joff1_; joff_ += kbdim_) {
                       int jsrc_ = tid_+joff_;
                       __syncthreads();
                       #if 1
                     {
                         float4 *srcbuf_ = (float4 *)(f03_jp_ + joff_);
                         float4 *dstbuf_ = (float4 *)smembuf_;
                         for (int icpy = 0; icpy < sizeof(f03_jp_t)/sizeof(float4); icpy++) {
                             dstbuf_[tid_] = srcbuf_[tid_];
                             dstbuf_ += kbdim_;
                             srcbuf_ += kbdim_;
                         }
                     }
#else
                     f03_jp_smem_[tid_] = f03_jp_[jsrc_];
#endif

                       __syncthreads();
                       int jsup_ = kbdim_;
                       if (joff_ + jsup_ > joff1_) {
                           jsup_ = joff1_ - joff_;
                       }
                       if (jsup_ < kbdim_) {
                           for (int j_ = jstart_; j_ < jsup_; j_+= jstride_) {
                               dx_0_ = f03_jp_smem_[j_].x_j_0_ - f03_ip_[isrc_].x_i_0_;
dx_1_ = f03_jp_smem_[j_].x_j_1_ - f03_ip_[isrc_].x_i_1_;
dx_2_ = f03_jp_smem_[j_].x_j_2_ - f03_ip_[isrc_].x_i_2_;
r2 = dx_0_ * dx_0_ + dx_1_ * dx_1_ + dx_2_ * dx_2_ + eps2;
rinv = rsqrt(r2);
mrinv = rinv * f03_jp_smem_[j_].m_j_;
pot_i_wcache_ -= mrinv;

                           }
                       }
                       else {    
                           for (int j_ = jstart_; j_ < kbdim_; j_+= jstride_ * 8) {
                                       // loop 0
dx_0_ = f03_jp_smem_[j_ + jstride_ * 0].x_j_0_ - f03_ip_[isrc_].x_i_0_;
dx_1_ = f03_jp_smem_[j_ + jstride_ * 0].x_j_1_ - f03_ip_[isrc_].x_i_1_;
dx_2_ = f03_jp_smem_[j_ + jstride_ * 0].x_j_2_ - f03_ip_[isrc_].x_i_2_;
r2 = dx_0_ * dx_0_ + dx_1_ * dx_1_ + dx_2_ * dx_2_ + eps2;
rinv = rsqrt(r2);
mrinv = rinv * f03_jp_smem_[j_ + jstride_ * 0].m_j_;
pot_i_wcache_ -= mrinv;
        // loop 1
dx_0_ = f03_jp_smem_[j_ + jstride_ * 1].x_j_0_ - f03_ip_[isrc_].x_i_0_;
dx_1_ = f03_jp_smem_[j_ + jstride_ * 1].x_j_1_ - f03_ip_[isrc_].x_i_1_;
dx_2_ = f03_jp_smem_[j_ + jstride_ * 1].x_j_2_ - f03_ip_[isrc_].x_i_2_;
r2 = dx_0_ * dx_0_ + dx_1_ * dx_1_ + dx_2_ * dx_2_ + eps2;
rinv = rsqrt(r2);
mrinv = rinv * f03_jp_smem_[j_ + jstride_ * 1].m_j_;
pot_i_wcache_ -= mrinv;
        // loop 2
dx_0_ = f03_jp_smem_[j_ + jstride_ * 2].x_j_0_ - f03_ip_[isrc_].x_i_0_;
dx_1_ = f03_jp_smem_[j_ + jstride_ * 2].x_j_1_ - f03_ip_[isrc_].x_i_1_;
dx_2_ = f03_jp_smem_[j_ + jstride_ * 2].x_j_2_ - f03_ip_[isrc_].x_i_2_;
r2 = dx_0_ * dx_0_ + dx_1_ * dx_1_ + dx_2_ * dx_2_ + eps2;
rinv = rsqrt(r2);
mrinv = rinv * f03_jp_smem_[j_ + jstride_ * 2].m_j_;
pot_i_wcache_ -= mrinv;
        // loop 3
dx_0_ = f03_jp_smem_[j_ + jstride_ * 3].x_j_0_ - f03_ip_[isrc_].x_i_0_;
dx_1_ = f03_jp_smem_[j_ + jstride_ * 3].x_j_1_ - f03_ip_[isrc_].x_i_1_;
dx_2_ = f03_jp_smem_[j_ + jstride_ * 3].x_j_2_ - f03_ip_[isrc_].x_i_2_;
r2 = dx_0_ * dx_0_ + dx_1_ * dx_1_ + dx_2_ * dx_2_ + eps2;
rinv = rsqrt(r2);
mrinv = rinv * f03_jp_smem_[j_ + jstride_ * 3].m_j_;
pot_i_wcache_ -= mrinv;
        // loop 4
dx_0_ = f03_jp_smem_[j_ + jstride_ * 4].x_j_0_ - f03_ip_[isrc_].x_i_0_;
dx_1_ = f03_jp_smem_[j_ + jstride_ * 4].x_j_1_ - f03_ip_[isrc_].x_i_1_;
dx_2_ = f03_jp_smem_[j_ + jstride_ * 4].x_j_2_ - f03_ip_[isrc_].x_i_2_;
r2 = dx_0_ * dx_0_ + dx_1_ * dx_1_ + dx_2_ * dx_2_ + eps2;
rinv = rsqrt(r2);
mrinv = rinv * f03_jp_smem_[j_ + jstride_ * 4].m_j_;
pot_i_wcache_ -= mrinv;
        // loop 5
dx_0_ = f03_jp_smem_[j_ + jstride_ * 5].x_j_0_ - f03_ip_[isrc_].x_i_0_;
dx_1_ = f03_jp_smem_[j_ + jstride_ * 5].x_j_1_ - f03_ip_[isrc_].x_i_1_;
dx_2_ = f03_jp_smem_[j_ + jstride_ * 5].x_j_2_ - f03_ip_[isrc_].x_i_2_;
r2 = dx_0_ * dx_0_ + dx_1_ * dx_1_ + dx_2_ * dx_2_ + eps2;
rinv = rsqrt(r2);
mrinv = rinv * f03_jp_smem_[j_ + jstride_ * 5].m_j_;
pot_i_wcache_ -= mrinv;
        // loop 6
dx_0_ = f03_jp_smem_[j_ + jstride_ * 6].x_j_0_ - f03_ip_[isrc_].x_i_0_;
dx_1_ = f03_jp_smem_[j_ + jstride_ * 6].x_j_1_ - f03_ip_[isrc_].x_i_1_;
dx_2_ = f03_jp_smem_[j_ + jstride_ * 6].x_j_2_ - f03_ip_[isrc_].x_i_2_;
r2 = dx_0_ * dx_0_ + dx_1_ * dx_1_ + dx_2_ * dx_2_ + eps2;
rinv = rsqrt(r2);
mrinv = rinv * f03_jp_smem_[j_ + jstride_ * 6].m_j_;
pot_i_wcache_ -= mrinv;
        // loop 7
dx_0_ = f03_jp_smem_[j_ + jstride_ * 7].x_j_0_ - f03_ip_[isrc_].x_i_0_;
dx_1_ = f03_jp_smem_[j_ + jstride_ * 7].x_j_1_ - f03_ip_[isrc_].x_i_1_;
dx_2_ = f03_jp_smem_[j_ + jstride_ * 7].x_j_2_ - f03_ip_[isrc_].x_i_2_;
r2 = dx_0_ * dx_0_ + dx_1_ * dx_1_ + dx_2_ * dx_2_ + eps2;
rinv = rsqrt(r2);
mrinv = rinv * f03_jp_smem_[j_ + jstride_ * 7].m_j_;
pot_i_wcache_ -= mrinv;

                           }
                       }
                   }
                   __syncthreads();
                   f03_result_smem_[tid_].pot_i_ = pot_i_wcache_;
                   __syncthreads();
                   if (jstride_ > 1) {if (tid_ < kbdim_ / 2) {f03_result_smem_[tid_].pot_i_ += f03_result_smem_[tid_ + kbdim_ / 2].pot_i_;}// __syncthreads(); // this is not necessary since kbdim_ / 2 <= warp size.
}if (jstride_ > 2) {if (tid_ < kbdim_ / 4) {f03_result_smem_[tid_].pot_i_ += f03_result_smem_[tid_ + kbdim_ / 4].pot_i_;}// __syncthreads(); // this is not necessary since kbdim_ / 4 <= warp size.
}
                   __syncthreads();

#if 1
                   if (tid_ < nvalidthread_) {
                       int idstoff_ = njdiv_ * kbdim_ * ibid_ + jbid_ + njdiv_ * (tid_ % njhsub_);
                       float4 *srcbuf_ = (float4 *) (f03_result_smem_ + tid_);
                       float4 *dstbuf_ = (float4 *) (f03_result_ + idstoff_);
                       for (int icpy_ = 0; icpy_ < sizeof(f03_result_t) / sizeof(float4); icpy_++) {
                           dstbuf_[icpy_] = srcbuf_[icpy_];
                       }
                   }
#else
                   if (tid_ < nvalidthread_) {
                       f03_result_[idst_] = f03_result_smem_[tid_];
                   }
#endif
               }

/*
 * njdiv_    : # of result fragments per result packet.
 * njdiv_ru_ : njdiv_ rounded up to a power of two.
 * rbdim_    : # of result fragments to be reduced to (rbdim_ / njdiv_) result packets.
 */
               __global__ void
               f03_reducer(int njdiv_, int njdiv_ru_, f03_result_t *f03_result_, f03_result_t *f03_result_sub_)
               {
                   extern __shared__ char smembuf_[];
                   int rbdim_ = blockDim.x;
                   f03_result_t * f03_result_smem_ = (f03_result_t *)smembuf_;
                   f03_result_t * f03_result_smem_packed_ = (f03_result_t *)(smembuf_ + rbdim_ * sizeof(f03_result_t));
                   int tid_ = threadIdx.x;
                   int bid_ = blockIdx.x;
                   int isrc_ = rbdim_ * bid_ + tid_;
                   int ndst_ = rbdim_ / njdiv_;
                   int idst_ = ndst_ * bid_ + tid_;
                   __syncthreads();
                   #if 1
                     {
                         float4 *srcbuf_ = (float4 *)(f03_result_sub_ + rbdim_ * bid_);
                         float4 *dstbuf_ = (float4 *)smembuf_;
                         for (int icpy = 0; icpy < sizeof(f03_result_t)/sizeof(float4); icpy++) {
                             dstbuf_[tid_] = srcbuf_[tid_];
                             dstbuf_ += rbdim_;
                             srcbuf_ += rbdim_;
                         }
                     }
#else
                     f03_result_smem_[tid_] = f03_result_sub_[isrc_];
#endif

                   __syncthreads();

                   int n_ = njdiv_ru_;
                   while (n_ > 1) {
                       n_ /= 2;
                       int ipartner_ = tid_ + n_;
                       if (tid_ % njdiv_ < n_ && ipartner_ % njdiv_ru_ < njdiv_) {
                           f03_result_smem_[tid_].pot_i_ += f03_result_smem_[ipartner_].pot_i_;
                       }
                       __syncthreads(); // this is not necessary if rbdim_ <= warp size.
                   }
                   __syncthreads();
                   if (tid_ % njdiv_ == 0) {
                       int ipack_ = tid_ / njdiv_;
                       f03_result_smem_packed_[ipack_] = f03_result_smem_[tid_];
                   }
                   __syncthreads();
#if 1
                   {
                       float4 *srcbuf_ = (float4 *) f03_result_smem_packed_;
                       float4 *dstbuf_ = (float4 *) (f03_result_ + ndst_ * bid_);
                       if (tid_ < ndst_) {
                           for (int icpy = 0; icpy < sizeof(f03_result_t) / sizeof(float4); icpy++) {
                               dstbuf_[tid_] = srcbuf_[tid_];
                               dstbuf_ += ndst_;
                               srcbuf_ += ndst_;
                           }
                       }
                   }
#else
                   if (tid_ < ndst_) {
                       f03_result_[idst_] = f03_result_smem_packed_[tid_];
                   }
#endif
               }
