#include "hip/hip_runtime.h"
               __global__ void
               f02_calculator(int ioff_, int ni_, int nj_, double x_0_0_, double x_0_1_, double x_0_2_, double eps2, double m_0_, f02_jp_t *f02_jp_, f02_ip_t *f02_ip_, f02_result_t *f02_result_)
               {
                   extern __shared__ char smembuf_[];
                   int kbdim_ = blockDim.x;
                   
                   f02_jp_t * f02_jp_smem_ = (f02_jp_t *)smembuf_;
                   
                   f02_result_t * f02_result_smem_ = (f02_result_t *)smembuf_;
                   double dxb_0_, dxc_0_, dxbc_0_, dvbc_0_, dxb_1_, dxc_1_, dxbc_1_, dvbc_1_, dxb_2_, dxc_2_, dxbc_2_, dvbc_2_, r1b2, r1b2e, r1be, r1c2, r1c2e, r1ce, rbc2, rbc2e, rbce, mr3inv;
                   double ac_i_0_wcache_ = 0.0f;
double ac_i_1_wcache_ = 0.0f;
double ac_i_2_wcache_ = 0.0f;

                   int njdiv_ = gridDim.x;
                   int jbid_ = blockIdx.x;
                   int ibid_ = blockIdx.y;
                   int tid_ = threadIdx.x;
                   int npipe_ = (ni_ - 1) / 1 + 1;
                   int nvalidthread_ = npipe_ - kbdim_ * ibid_;
                   if (nvalidthread_ > kbdim_) {
                       nvalidthread_ = kbdim_;
                   }
                   int njsub_ = (nj_ - 1) / njdiv_ + 1;
                   int joff0_ = njsub_ * jbid_;
                   int joff1_ = joff0_ + njsub_;
                   if (joff1_ > nj_) {
                       joff1_ = nj_;
                   }
                   int jstride_ = 1;
                   if (nvalidthread_ <= kbdim_ / 2) jstride_ = 2;
if (nvalidthread_ <= kbdim_ / 4) jstride_ = 4;

                   int njhsub_ = kbdim_ / jstride_;
                   int jstart_ = tid_ / njhsub_;
                   int isrc_ = kbdim_ * ibid_ + tid_ % njhsub_;
                   int idst_ = njdiv_ * isrc_ + jbid_;
                   int icnt_ = ioff_ + isrc_;
                   for (int joff_ = joff0_; joff_ < joff1_; joff_ += kbdim_) {
                       int jsrc_ = tid_+joff_;
                       __syncthreads();
                       #if 1
                     {
                         float4 *srcbuf_ = (float4 *)(f02_jp_ + joff_);
                         float4 *dstbuf_ = (float4 *)smembuf_;
                         for (int icpy = 0; icpy < sizeof(f02_jp_t)/sizeof(float4); icpy++) {
                             dstbuf_[tid_] = srcbuf_[tid_];
                             dstbuf_ += kbdim_;
                             srcbuf_ += kbdim_;
                         }
                     }
#else
                     f02_jp_smem_[tid_] = f02_jp_[jsrc_];
#endif

                       __syncthreads();
                       int jsup_ = kbdim_;
                       if (joff_ + jsup_ > joff1_) {
                           jsup_ = joff1_ - joff_;
                       }
                       if (jsup_ < kbdim_) {
                           for (int j_ = jstart_; j_ < jsup_; j_+= jstride_) {
                               dxb_0_ = x_0_0_ - f02_ip_[isrc_].x_i_0_;
dxc_0_ = x_0_0_ - f02_jp_smem_[j_].x_j_0_;
dxbc_0_ = f02_jp_smem_[j_].x_j_0_ - f02_ip_[isrc_].x_i_0_;
dvbc_0_ = f02_jp_smem_[j_].v_j_0_ - f02_ip_[isrc_].v_i_0_;
dxb_1_ = x_0_1_ - f02_ip_[isrc_].x_i_1_;
dxc_1_ = x_0_1_ - f02_jp_smem_[j_].x_j_1_;
dxbc_1_ = f02_jp_smem_[j_].x_j_1_ - f02_ip_[isrc_].x_i_1_;
dvbc_1_ = f02_jp_smem_[j_].v_j_1_ - f02_ip_[isrc_].v_i_1_;
dxb_2_ = x_0_2_ - f02_ip_[isrc_].x_i_2_;
dxc_2_ = x_0_2_ - f02_jp_smem_[j_].x_j_2_;
dxbc_2_ = f02_jp_smem_[j_].x_j_2_ - f02_ip_[isrc_].x_i_2_;
dvbc_2_ = f02_jp_smem_[j_].v_j_2_ - f02_ip_[isrc_].v_i_2_;
r1b2 = dxb_0_ * dxb_0_ + dxb_1_ * dxb_1_ + dxb_2_ * dxb_2_;
r1b2e = r1b2 + eps2;
r1be = rsqrt(r1b2e);
r1c2 = dxc_0_ * dxc_0_ + dxc_1_ * dxc_1_ + dxc_2_ * dxc_2_;
r1c2e = r1c2 + eps2;
r1ce = rsqrt(r1c2e);
rbc2 = dxbc_0_ * dxbc_0_ + dxbc_1_ * dxbc_1_ + dxbc_2_ * dxbc_2_;
rbc2e = rbc2 + eps2;
rbce = rsqrt(rbc2e);
mr3inv = f02_jp_smem_[j_].m_j_ * rbce * rbce * rbce;
ac_i_0_wcache_ += f02_jp_smem_[j_].m_j_ * r1be * r1be * r1be * m_0_ * (4.0 * rbce + 1.25 * r1ce + 0.25 * (r1b2 - rbc2) * r1ce * r1ce * r1ce) * dxb_0_ + mr3inv * m_0_ * (4.0 * r1be + 1.25 * r1ce + 0.25 * (-r1b2 + rbc2) * r1ce * r1ce * r1ce) * dxbc_0_ - 3.5 * f02_jp_smem_[j_].m_j_ * m_0_ * r1ce * r1ce * r1ce * (rbce - r1be) * dxc_0_ - f02_jp_smem_[j_].m_j_ * r1be * r1be * r1be * (4.0 * (f02_ip_[isrc_].v_i_0_ * f02_jp_smem_[j_].v_j_0_ + f02_ip_[isrc_].v_i_1_ * f02_jp_smem_[j_].v_j_1_ + f02_ip_[isrc_].v_i_2_ * f02_jp_smem_[j_].v_j_2_) * dxb_0_ - 3.0 * (f02_jp_smem_[j_].v_j_0_ * dxb_0_ + f02_jp_smem_[j_].v_j_1_ * dxb_1_ + f02_jp_smem_[j_].v_j_2_ * dxb_2_) * f02_ip_[isrc_].v_i_0_ - 4.0 * (f02_ip_[isrc_].v_i_0_ * dxb_0_ + f02_ip_[isrc_].v_i_1_ * dxb_1_ + f02_ip_[isrc_].v_i_2_ * dxb_2_) * f02_jp_smem_[j_].v_j_0_) + mr3inv * (f02_ip_[isrc_].v_i_0_ * f02_ip_[isrc_].v_i_0_ + f02_ip_[isrc_].v_i_1_ * f02_ip_[isrc_].v_i_1_ + f02_ip_[isrc_].v_i_2_ * f02_ip_[isrc_].v_i_2_ - 2.0 * (dvbc_0_ * dvbc_0_ + dvbc_1_ * dvbc_1_ + dvbc_2_ * dvbc_2_) + 1.5 * (f02_jp_smem_[j_].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_].v_j_2_ * dxbc_2_) * (f02_jp_smem_[j_].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_].v_j_2_ * dxbc_2_) / rbc2e) * dxbc_0_ + mr3inv * (dxbc_0_ * (4.0 * f02_ip_[isrc_].v_i_0_ - 3.0 * f02_jp_smem_[j_].v_j_0_) + dxbc_1_ * (4.0 * f02_ip_[isrc_].v_i_1_ - 3.0 * f02_jp_smem_[j_].v_j_1_) + dxbc_2_ * (4.0 * f02_ip_[isrc_].v_i_2_ - 3.0 * f02_jp_smem_[j_].v_j_2_)) * dvbc_0_;
ac_i_1_wcache_ += f02_jp_smem_[j_].m_j_ * r1be * r1be * r1be * m_0_ * (4.0 * rbce + 1.25 * r1ce + 0.25 * (r1b2 - rbc2) * r1ce * r1ce * r1ce) * dxb_1_ + mr3inv * m_0_ * (4.0 * r1be + 1.25 * r1ce + 0.25 * (-r1b2 + rbc2) * r1ce * r1ce * r1ce) * dxbc_1_ - 3.5 * f02_jp_smem_[j_].m_j_ * m_0_ * r1ce * r1ce * r1ce * (rbce - r1be) * dxc_1_ - f02_jp_smem_[j_].m_j_ * r1be * r1be * r1be * (4.0 * (f02_ip_[isrc_].v_i_0_ * f02_jp_smem_[j_].v_j_0_ + f02_ip_[isrc_].v_i_1_ * f02_jp_smem_[j_].v_j_1_ + f02_ip_[isrc_].v_i_2_ * f02_jp_smem_[j_].v_j_2_) * dxb_1_ - 3.0 * (f02_jp_smem_[j_].v_j_0_ * dxb_0_ + f02_jp_smem_[j_].v_j_1_ * dxb_1_ + f02_jp_smem_[j_].v_j_2_ * dxb_2_) * f02_ip_[isrc_].v_i_1_ - 4.0 * (f02_ip_[isrc_].v_i_0_ * dxb_0_ + f02_ip_[isrc_].v_i_1_ * dxb_1_ + f02_ip_[isrc_].v_i_2_ * dxb_2_) * f02_jp_smem_[j_].v_j_1_) + mr3inv * (f02_ip_[isrc_].v_i_0_ * f02_ip_[isrc_].v_i_0_ + f02_ip_[isrc_].v_i_1_ * f02_ip_[isrc_].v_i_1_ + f02_ip_[isrc_].v_i_2_ * f02_ip_[isrc_].v_i_2_ - 2.0 * (dvbc_0_ * dvbc_0_ + dvbc_1_ * dvbc_1_ + dvbc_2_ * dvbc_2_) + 1.5 * (f02_jp_smem_[j_].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_].v_j_2_ * dxbc_2_) * (f02_jp_smem_[j_].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_].v_j_2_ * dxbc_2_) / rbc2e) * dxbc_1_ + mr3inv * (dxbc_0_ * (4.0 * f02_ip_[isrc_].v_i_0_ - 3.0 * f02_jp_smem_[j_].v_j_0_) + dxbc_1_ * (4.0 * f02_ip_[isrc_].v_i_1_ - 3.0 * f02_jp_smem_[j_].v_j_1_) + dxbc_2_ * (4.0 * f02_ip_[isrc_].v_i_2_ - 3.0 * f02_jp_smem_[j_].v_j_2_)) * dvbc_1_;
ac_i_2_wcache_ += f02_jp_smem_[j_].m_j_ * r1be * r1be * r1be * m_0_ * (4.0 * rbce + 1.25 * r1ce + 0.25 * (r1b2 - rbc2) * r1ce * r1ce * r1ce) * dxb_2_ + mr3inv * m_0_ * (4.0 * r1be + 1.25 * r1ce + 0.25 * (-r1b2 + rbc2) * r1ce * r1ce * r1ce) * dxbc_2_ - 3.5 * f02_jp_smem_[j_].m_j_ * m_0_ * r1ce * r1ce * r1ce * (rbce - r1be) * dxc_2_ - f02_jp_smem_[j_].m_j_ * r1be * r1be * r1be * (4.0 * (f02_ip_[isrc_].v_i_0_ * f02_jp_smem_[j_].v_j_0_ + f02_ip_[isrc_].v_i_1_ * f02_jp_smem_[j_].v_j_1_ + f02_ip_[isrc_].v_i_2_ * f02_jp_smem_[j_].v_j_2_) * dxb_2_ - 3.0 * (f02_jp_smem_[j_].v_j_0_ * dxb_0_ + f02_jp_smem_[j_].v_j_1_ * dxb_1_ + f02_jp_smem_[j_].v_j_2_ * dxb_2_) * f02_ip_[isrc_].v_i_2_ - 4.0 * (f02_ip_[isrc_].v_i_0_ * dxb_0_ + f02_ip_[isrc_].v_i_1_ * dxb_1_ + f02_ip_[isrc_].v_i_2_ * dxb_2_) * f02_jp_smem_[j_].v_j_2_) + mr3inv * (f02_ip_[isrc_].v_i_0_ * f02_ip_[isrc_].v_i_0_ + f02_ip_[isrc_].v_i_1_ * f02_ip_[isrc_].v_i_1_ + f02_ip_[isrc_].v_i_2_ * f02_ip_[isrc_].v_i_2_ - 2.0 * (dvbc_0_ * dvbc_0_ + dvbc_1_ * dvbc_1_ + dvbc_2_ * dvbc_2_) + 1.5 * (f02_jp_smem_[j_].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_].v_j_2_ * dxbc_2_) * (f02_jp_smem_[j_].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_].v_j_2_ * dxbc_2_) / rbc2e) * dxbc_2_ + mr3inv * (dxbc_0_ * (4.0 * f02_ip_[isrc_].v_i_0_ - 3.0 * f02_jp_smem_[j_].v_j_0_) + dxbc_1_ * (4.0 * f02_ip_[isrc_].v_i_1_ - 3.0 * f02_jp_smem_[j_].v_j_1_) + dxbc_2_ * (4.0 * f02_ip_[isrc_].v_i_2_ - 3.0 * f02_jp_smem_[j_].v_j_2_)) * dvbc_2_;

                           }
                       }
                       else {    
                           for (int j_ = jstart_; j_ < kbdim_; j_+= jstride_ * 8) {
                                       // loop 0
dxb_0_ = x_0_0_ - f02_ip_[isrc_].x_i_0_;
dxc_0_ = x_0_0_ - f02_jp_smem_[j_ + jstride_ * 0].x_j_0_;
dxbc_0_ = f02_jp_smem_[j_ + jstride_ * 0].x_j_0_ - f02_ip_[isrc_].x_i_0_;
dvbc_0_ = f02_jp_smem_[j_ + jstride_ * 0].v_j_0_ - f02_ip_[isrc_].v_i_0_;
dxb_1_ = x_0_1_ - f02_ip_[isrc_].x_i_1_;
dxc_1_ = x_0_1_ - f02_jp_smem_[j_ + jstride_ * 0].x_j_1_;
dxbc_1_ = f02_jp_smem_[j_ + jstride_ * 0].x_j_1_ - f02_ip_[isrc_].x_i_1_;
dvbc_1_ = f02_jp_smem_[j_ + jstride_ * 0].v_j_1_ - f02_ip_[isrc_].v_i_1_;
dxb_2_ = x_0_2_ - f02_ip_[isrc_].x_i_2_;
dxc_2_ = x_0_2_ - f02_jp_smem_[j_ + jstride_ * 0].x_j_2_;
dxbc_2_ = f02_jp_smem_[j_ + jstride_ * 0].x_j_2_ - f02_ip_[isrc_].x_i_2_;
dvbc_2_ = f02_jp_smem_[j_ + jstride_ * 0].v_j_2_ - f02_ip_[isrc_].v_i_2_;
r1b2 = dxb_0_ * dxb_0_ + dxb_1_ * dxb_1_ + dxb_2_ * dxb_2_;
r1b2e = r1b2 + eps2;
r1be = rsqrt(r1b2e);
r1c2 = dxc_0_ * dxc_0_ + dxc_1_ * dxc_1_ + dxc_2_ * dxc_2_;
r1c2e = r1c2 + eps2;
r1ce = rsqrt(r1c2e);
rbc2 = dxbc_0_ * dxbc_0_ + dxbc_1_ * dxbc_1_ + dxbc_2_ * dxbc_2_;
rbc2e = rbc2 + eps2;
rbce = rsqrt(rbc2e);
mr3inv = f02_jp_smem_[j_ + jstride_ * 0].m_j_ * rbce * rbce * rbce;
ac_i_0_wcache_ += f02_jp_smem_[j_ + jstride_ * 0].m_j_ * r1be * r1be * r1be * m_0_ * (4.0 * rbce + 1.25 * r1ce + 0.25 * (r1b2 - rbc2) * r1ce * r1ce * r1ce) * dxb_0_ + mr3inv * m_0_ * (4.0 * r1be + 1.25 * r1ce + 0.25 * (-r1b2 + rbc2) * r1ce * r1ce * r1ce) * dxbc_0_ - 3.5 * f02_jp_smem_[j_ + jstride_ * 0].m_j_ * m_0_ * r1ce * r1ce * r1ce * (rbce - r1be) * dxc_0_ - f02_jp_smem_[j_ + jstride_ * 0].m_j_ * r1be * r1be * r1be * (4.0 * (f02_ip_[isrc_].v_i_0_ * f02_jp_smem_[j_ + jstride_ * 0].v_j_0_ + f02_ip_[isrc_].v_i_1_ * f02_jp_smem_[j_ + jstride_ * 0].v_j_1_ + f02_ip_[isrc_].v_i_2_ * f02_jp_smem_[j_ + jstride_ * 0].v_j_2_) * dxb_0_ - 3.0 * (f02_jp_smem_[j_ + jstride_ * 0].v_j_0_ * dxb_0_ + f02_jp_smem_[j_ + jstride_ * 0].v_j_1_ * dxb_1_ + f02_jp_smem_[j_ + jstride_ * 0].v_j_2_ * dxb_2_) * f02_ip_[isrc_].v_i_0_ - 4.0 * (f02_ip_[isrc_].v_i_0_ * dxb_0_ + f02_ip_[isrc_].v_i_1_ * dxb_1_ + f02_ip_[isrc_].v_i_2_ * dxb_2_) * f02_jp_smem_[j_ + jstride_ * 0].v_j_0_) + mr3inv * (f02_ip_[isrc_].v_i_0_ * f02_ip_[isrc_].v_i_0_ + f02_ip_[isrc_].v_i_1_ * f02_ip_[isrc_].v_i_1_ + f02_ip_[isrc_].v_i_2_ * f02_ip_[isrc_].v_i_2_ - 2.0 * (dvbc_0_ * dvbc_0_ + dvbc_1_ * dvbc_1_ + dvbc_2_ * dvbc_2_) + 1.5 * (f02_jp_smem_[j_ + jstride_ * 0].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 0].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 0].v_j_2_ * dxbc_2_) * (f02_jp_smem_[j_ + jstride_ * 0].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 0].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 0].v_j_2_ * dxbc_2_) / rbc2e) * dxbc_0_ + mr3inv * (dxbc_0_ * (4.0 * f02_ip_[isrc_].v_i_0_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 0].v_j_0_) + dxbc_1_ * (4.0 * f02_ip_[isrc_].v_i_1_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 0].v_j_1_) + dxbc_2_ * (4.0 * f02_ip_[isrc_].v_i_2_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 0].v_j_2_)) * dvbc_0_;
ac_i_1_wcache_ += f02_jp_smem_[j_ + jstride_ * 0].m_j_ * r1be * r1be * r1be * m_0_ * (4.0 * rbce + 1.25 * r1ce + 0.25 * (r1b2 - rbc2) * r1ce * r1ce * r1ce) * dxb_1_ + mr3inv * m_0_ * (4.0 * r1be + 1.25 * r1ce + 0.25 * (-r1b2 + rbc2) * r1ce * r1ce * r1ce) * dxbc_1_ - 3.5 * f02_jp_smem_[j_ + jstride_ * 0].m_j_ * m_0_ * r1ce * r1ce * r1ce * (rbce - r1be) * dxc_1_ - f02_jp_smem_[j_ + jstride_ * 0].m_j_ * r1be * r1be * r1be * (4.0 * (f02_ip_[isrc_].v_i_0_ * f02_jp_smem_[j_ + jstride_ * 0].v_j_0_ + f02_ip_[isrc_].v_i_1_ * f02_jp_smem_[j_ + jstride_ * 0].v_j_1_ + f02_ip_[isrc_].v_i_2_ * f02_jp_smem_[j_ + jstride_ * 0].v_j_2_) * dxb_1_ - 3.0 * (f02_jp_smem_[j_ + jstride_ * 0].v_j_0_ * dxb_0_ + f02_jp_smem_[j_ + jstride_ * 0].v_j_1_ * dxb_1_ + f02_jp_smem_[j_ + jstride_ * 0].v_j_2_ * dxb_2_) * f02_ip_[isrc_].v_i_1_ - 4.0 * (f02_ip_[isrc_].v_i_0_ * dxb_0_ + f02_ip_[isrc_].v_i_1_ * dxb_1_ + f02_ip_[isrc_].v_i_2_ * dxb_2_) * f02_jp_smem_[j_ + jstride_ * 0].v_j_1_) + mr3inv * (f02_ip_[isrc_].v_i_0_ * f02_ip_[isrc_].v_i_0_ + f02_ip_[isrc_].v_i_1_ * f02_ip_[isrc_].v_i_1_ + f02_ip_[isrc_].v_i_2_ * f02_ip_[isrc_].v_i_2_ - 2.0 * (dvbc_0_ * dvbc_0_ + dvbc_1_ * dvbc_1_ + dvbc_2_ * dvbc_2_) + 1.5 * (f02_jp_smem_[j_ + jstride_ * 0].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 0].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 0].v_j_2_ * dxbc_2_) * (f02_jp_smem_[j_ + jstride_ * 0].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 0].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 0].v_j_2_ * dxbc_2_) / rbc2e) * dxbc_1_ + mr3inv * (dxbc_0_ * (4.0 * f02_ip_[isrc_].v_i_0_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 0].v_j_0_) + dxbc_1_ * (4.0 * f02_ip_[isrc_].v_i_1_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 0].v_j_1_) + dxbc_2_ * (4.0 * f02_ip_[isrc_].v_i_2_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 0].v_j_2_)) * dvbc_1_;
ac_i_2_wcache_ += f02_jp_smem_[j_ + jstride_ * 0].m_j_ * r1be * r1be * r1be * m_0_ * (4.0 * rbce + 1.25 * r1ce + 0.25 * (r1b2 - rbc2) * r1ce * r1ce * r1ce) * dxb_2_ + mr3inv * m_0_ * (4.0 * r1be + 1.25 * r1ce + 0.25 * (-r1b2 + rbc2) * r1ce * r1ce * r1ce) * dxbc_2_ - 3.5 * f02_jp_smem_[j_ + jstride_ * 0].m_j_ * m_0_ * r1ce * r1ce * r1ce * (rbce - r1be) * dxc_2_ - f02_jp_smem_[j_ + jstride_ * 0].m_j_ * r1be * r1be * r1be * (4.0 * (f02_ip_[isrc_].v_i_0_ * f02_jp_smem_[j_ + jstride_ * 0].v_j_0_ + f02_ip_[isrc_].v_i_1_ * f02_jp_smem_[j_ + jstride_ * 0].v_j_1_ + f02_ip_[isrc_].v_i_2_ * f02_jp_smem_[j_ + jstride_ * 0].v_j_2_) * dxb_2_ - 3.0 * (f02_jp_smem_[j_ + jstride_ * 0].v_j_0_ * dxb_0_ + f02_jp_smem_[j_ + jstride_ * 0].v_j_1_ * dxb_1_ + f02_jp_smem_[j_ + jstride_ * 0].v_j_2_ * dxb_2_) * f02_ip_[isrc_].v_i_2_ - 4.0 * (f02_ip_[isrc_].v_i_0_ * dxb_0_ + f02_ip_[isrc_].v_i_1_ * dxb_1_ + f02_ip_[isrc_].v_i_2_ * dxb_2_) * f02_jp_smem_[j_ + jstride_ * 0].v_j_2_) + mr3inv * (f02_ip_[isrc_].v_i_0_ * f02_ip_[isrc_].v_i_0_ + f02_ip_[isrc_].v_i_1_ * f02_ip_[isrc_].v_i_1_ + f02_ip_[isrc_].v_i_2_ * f02_ip_[isrc_].v_i_2_ - 2.0 * (dvbc_0_ * dvbc_0_ + dvbc_1_ * dvbc_1_ + dvbc_2_ * dvbc_2_) + 1.5 * (f02_jp_smem_[j_ + jstride_ * 0].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 0].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 0].v_j_2_ * dxbc_2_) * (f02_jp_smem_[j_ + jstride_ * 0].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 0].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 0].v_j_2_ * dxbc_2_) / rbc2e) * dxbc_2_ + mr3inv * (dxbc_0_ * (4.0 * f02_ip_[isrc_].v_i_0_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 0].v_j_0_) + dxbc_1_ * (4.0 * f02_ip_[isrc_].v_i_1_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 0].v_j_1_) + dxbc_2_ * (4.0 * f02_ip_[isrc_].v_i_2_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 0].v_j_2_)) * dvbc_2_;
        // loop 1
dxb_0_ = x_0_0_ - f02_ip_[isrc_].x_i_0_;
dxc_0_ = x_0_0_ - f02_jp_smem_[j_ + jstride_ * 1].x_j_0_;
dxbc_0_ = f02_jp_smem_[j_ + jstride_ * 1].x_j_0_ - f02_ip_[isrc_].x_i_0_;
dvbc_0_ = f02_jp_smem_[j_ + jstride_ * 1].v_j_0_ - f02_ip_[isrc_].v_i_0_;
dxb_1_ = x_0_1_ - f02_ip_[isrc_].x_i_1_;
dxc_1_ = x_0_1_ - f02_jp_smem_[j_ + jstride_ * 1].x_j_1_;
dxbc_1_ = f02_jp_smem_[j_ + jstride_ * 1].x_j_1_ - f02_ip_[isrc_].x_i_1_;
dvbc_1_ = f02_jp_smem_[j_ + jstride_ * 1].v_j_1_ - f02_ip_[isrc_].v_i_1_;
dxb_2_ = x_0_2_ - f02_ip_[isrc_].x_i_2_;
dxc_2_ = x_0_2_ - f02_jp_smem_[j_ + jstride_ * 1].x_j_2_;
dxbc_2_ = f02_jp_smem_[j_ + jstride_ * 1].x_j_2_ - f02_ip_[isrc_].x_i_2_;
dvbc_2_ = f02_jp_smem_[j_ + jstride_ * 1].v_j_2_ - f02_ip_[isrc_].v_i_2_;
r1b2 = dxb_0_ * dxb_0_ + dxb_1_ * dxb_1_ + dxb_2_ * dxb_2_;
r1b2e = r1b2 + eps2;
r1be = rsqrt(r1b2e);
r1c2 = dxc_0_ * dxc_0_ + dxc_1_ * dxc_1_ + dxc_2_ * dxc_2_;
r1c2e = r1c2 + eps2;
r1ce = rsqrt(r1c2e);
rbc2 = dxbc_0_ * dxbc_0_ + dxbc_1_ * dxbc_1_ + dxbc_2_ * dxbc_2_;
rbc2e = rbc2 + eps2;
rbce = rsqrt(rbc2e);
mr3inv = f02_jp_smem_[j_ + jstride_ * 1].m_j_ * rbce * rbce * rbce;
ac_i_0_wcache_ += f02_jp_smem_[j_ + jstride_ * 1].m_j_ * r1be * r1be * r1be * m_0_ * (4.0 * rbce + 1.25 * r1ce + 0.25 * (r1b2 - rbc2) * r1ce * r1ce * r1ce) * dxb_0_ + mr3inv * m_0_ * (4.0 * r1be + 1.25 * r1ce + 0.25 * (-r1b2 + rbc2) * r1ce * r1ce * r1ce) * dxbc_0_ - 3.5 * f02_jp_smem_[j_ + jstride_ * 1].m_j_ * m_0_ * r1ce * r1ce * r1ce * (rbce - r1be) * dxc_0_ - f02_jp_smem_[j_ + jstride_ * 1].m_j_ * r1be * r1be * r1be * (4.0 * (f02_ip_[isrc_].v_i_0_ * f02_jp_smem_[j_ + jstride_ * 1].v_j_0_ + f02_ip_[isrc_].v_i_1_ * f02_jp_smem_[j_ + jstride_ * 1].v_j_1_ + f02_ip_[isrc_].v_i_2_ * f02_jp_smem_[j_ + jstride_ * 1].v_j_2_) * dxb_0_ - 3.0 * (f02_jp_smem_[j_ + jstride_ * 1].v_j_0_ * dxb_0_ + f02_jp_smem_[j_ + jstride_ * 1].v_j_1_ * dxb_1_ + f02_jp_smem_[j_ + jstride_ * 1].v_j_2_ * dxb_2_) * f02_ip_[isrc_].v_i_0_ - 4.0 * (f02_ip_[isrc_].v_i_0_ * dxb_0_ + f02_ip_[isrc_].v_i_1_ * dxb_1_ + f02_ip_[isrc_].v_i_2_ * dxb_2_) * f02_jp_smem_[j_ + jstride_ * 1].v_j_0_) + mr3inv * (f02_ip_[isrc_].v_i_0_ * f02_ip_[isrc_].v_i_0_ + f02_ip_[isrc_].v_i_1_ * f02_ip_[isrc_].v_i_1_ + f02_ip_[isrc_].v_i_2_ * f02_ip_[isrc_].v_i_2_ - 2.0 * (dvbc_0_ * dvbc_0_ + dvbc_1_ * dvbc_1_ + dvbc_2_ * dvbc_2_) + 1.5 * (f02_jp_smem_[j_ + jstride_ * 1].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 1].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 1].v_j_2_ * dxbc_2_) * (f02_jp_smem_[j_ + jstride_ * 1].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 1].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 1].v_j_2_ * dxbc_2_) / rbc2e) * dxbc_0_ + mr3inv * (dxbc_0_ * (4.0 * f02_ip_[isrc_].v_i_0_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 1].v_j_0_) + dxbc_1_ * (4.0 * f02_ip_[isrc_].v_i_1_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 1].v_j_1_) + dxbc_2_ * (4.0 * f02_ip_[isrc_].v_i_2_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 1].v_j_2_)) * dvbc_0_;
ac_i_1_wcache_ += f02_jp_smem_[j_ + jstride_ * 1].m_j_ * r1be * r1be * r1be * m_0_ * (4.0 * rbce + 1.25 * r1ce + 0.25 * (r1b2 - rbc2) * r1ce * r1ce * r1ce) * dxb_1_ + mr3inv * m_0_ * (4.0 * r1be + 1.25 * r1ce + 0.25 * (-r1b2 + rbc2) * r1ce * r1ce * r1ce) * dxbc_1_ - 3.5 * f02_jp_smem_[j_ + jstride_ * 1].m_j_ * m_0_ * r1ce * r1ce * r1ce * (rbce - r1be) * dxc_1_ - f02_jp_smem_[j_ + jstride_ * 1].m_j_ * r1be * r1be * r1be * (4.0 * (f02_ip_[isrc_].v_i_0_ * f02_jp_smem_[j_ + jstride_ * 1].v_j_0_ + f02_ip_[isrc_].v_i_1_ * f02_jp_smem_[j_ + jstride_ * 1].v_j_1_ + f02_ip_[isrc_].v_i_2_ * f02_jp_smem_[j_ + jstride_ * 1].v_j_2_) * dxb_1_ - 3.0 * (f02_jp_smem_[j_ + jstride_ * 1].v_j_0_ * dxb_0_ + f02_jp_smem_[j_ + jstride_ * 1].v_j_1_ * dxb_1_ + f02_jp_smem_[j_ + jstride_ * 1].v_j_2_ * dxb_2_) * f02_ip_[isrc_].v_i_1_ - 4.0 * (f02_ip_[isrc_].v_i_0_ * dxb_0_ + f02_ip_[isrc_].v_i_1_ * dxb_1_ + f02_ip_[isrc_].v_i_2_ * dxb_2_) * f02_jp_smem_[j_ + jstride_ * 1].v_j_1_) + mr3inv * (f02_ip_[isrc_].v_i_0_ * f02_ip_[isrc_].v_i_0_ + f02_ip_[isrc_].v_i_1_ * f02_ip_[isrc_].v_i_1_ + f02_ip_[isrc_].v_i_2_ * f02_ip_[isrc_].v_i_2_ - 2.0 * (dvbc_0_ * dvbc_0_ + dvbc_1_ * dvbc_1_ + dvbc_2_ * dvbc_2_) + 1.5 * (f02_jp_smem_[j_ + jstride_ * 1].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 1].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 1].v_j_2_ * dxbc_2_) * (f02_jp_smem_[j_ + jstride_ * 1].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 1].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 1].v_j_2_ * dxbc_2_) / rbc2e) * dxbc_1_ + mr3inv * (dxbc_0_ * (4.0 * f02_ip_[isrc_].v_i_0_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 1].v_j_0_) + dxbc_1_ * (4.0 * f02_ip_[isrc_].v_i_1_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 1].v_j_1_) + dxbc_2_ * (4.0 * f02_ip_[isrc_].v_i_2_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 1].v_j_2_)) * dvbc_1_;
ac_i_2_wcache_ += f02_jp_smem_[j_ + jstride_ * 1].m_j_ * r1be * r1be * r1be * m_0_ * (4.0 * rbce + 1.25 * r1ce + 0.25 * (r1b2 - rbc2) * r1ce * r1ce * r1ce) * dxb_2_ + mr3inv * m_0_ * (4.0 * r1be + 1.25 * r1ce + 0.25 * (-r1b2 + rbc2) * r1ce * r1ce * r1ce) * dxbc_2_ - 3.5 * f02_jp_smem_[j_ + jstride_ * 1].m_j_ * m_0_ * r1ce * r1ce * r1ce * (rbce - r1be) * dxc_2_ - f02_jp_smem_[j_ + jstride_ * 1].m_j_ * r1be * r1be * r1be * (4.0 * (f02_ip_[isrc_].v_i_0_ * f02_jp_smem_[j_ + jstride_ * 1].v_j_0_ + f02_ip_[isrc_].v_i_1_ * f02_jp_smem_[j_ + jstride_ * 1].v_j_1_ + f02_ip_[isrc_].v_i_2_ * f02_jp_smem_[j_ + jstride_ * 1].v_j_2_) * dxb_2_ - 3.0 * (f02_jp_smem_[j_ + jstride_ * 1].v_j_0_ * dxb_0_ + f02_jp_smem_[j_ + jstride_ * 1].v_j_1_ * dxb_1_ + f02_jp_smem_[j_ + jstride_ * 1].v_j_2_ * dxb_2_) * f02_ip_[isrc_].v_i_2_ - 4.0 * (f02_ip_[isrc_].v_i_0_ * dxb_0_ + f02_ip_[isrc_].v_i_1_ * dxb_1_ + f02_ip_[isrc_].v_i_2_ * dxb_2_) * f02_jp_smem_[j_ + jstride_ * 1].v_j_2_) + mr3inv * (f02_ip_[isrc_].v_i_0_ * f02_ip_[isrc_].v_i_0_ + f02_ip_[isrc_].v_i_1_ * f02_ip_[isrc_].v_i_1_ + f02_ip_[isrc_].v_i_2_ * f02_ip_[isrc_].v_i_2_ - 2.0 * (dvbc_0_ * dvbc_0_ + dvbc_1_ * dvbc_1_ + dvbc_2_ * dvbc_2_) + 1.5 * (f02_jp_smem_[j_ + jstride_ * 1].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 1].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 1].v_j_2_ * dxbc_2_) * (f02_jp_smem_[j_ + jstride_ * 1].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 1].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 1].v_j_2_ * dxbc_2_) / rbc2e) * dxbc_2_ + mr3inv * (dxbc_0_ * (4.0 * f02_ip_[isrc_].v_i_0_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 1].v_j_0_) + dxbc_1_ * (4.0 * f02_ip_[isrc_].v_i_1_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 1].v_j_1_) + dxbc_2_ * (4.0 * f02_ip_[isrc_].v_i_2_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 1].v_j_2_)) * dvbc_2_;
        // loop 2
dxb_0_ = x_0_0_ - f02_ip_[isrc_].x_i_0_;
dxc_0_ = x_0_0_ - f02_jp_smem_[j_ + jstride_ * 2].x_j_0_;
dxbc_0_ = f02_jp_smem_[j_ + jstride_ * 2].x_j_0_ - f02_ip_[isrc_].x_i_0_;
dvbc_0_ = f02_jp_smem_[j_ + jstride_ * 2].v_j_0_ - f02_ip_[isrc_].v_i_0_;
dxb_1_ = x_0_1_ - f02_ip_[isrc_].x_i_1_;
dxc_1_ = x_0_1_ - f02_jp_smem_[j_ + jstride_ * 2].x_j_1_;
dxbc_1_ = f02_jp_smem_[j_ + jstride_ * 2].x_j_1_ - f02_ip_[isrc_].x_i_1_;
dvbc_1_ = f02_jp_smem_[j_ + jstride_ * 2].v_j_1_ - f02_ip_[isrc_].v_i_1_;
dxb_2_ = x_0_2_ - f02_ip_[isrc_].x_i_2_;
dxc_2_ = x_0_2_ - f02_jp_smem_[j_ + jstride_ * 2].x_j_2_;
dxbc_2_ = f02_jp_smem_[j_ + jstride_ * 2].x_j_2_ - f02_ip_[isrc_].x_i_2_;
dvbc_2_ = f02_jp_smem_[j_ + jstride_ * 2].v_j_2_ - f02_ip_[isrc_].v_i_2_;
r1b2 = dxb_0_ * dxb_0_ + dxb_1_ * dxb_1_ + dxb_2_ * dxb_2_;
r1b2e = r1b2 + eps2;
r1be = rsqrt(r1b2e);
r1c2 = dxc_0_ * dxc_0_ + dxc_1_ * dxc_1_ + dxc_2_ * dxc_2_;
r1c2e = r1c2 + eps2;
r1ce = rsqrt(r1c2e);
rbc2 = dxbc_0_ * dxbc_0_ + dxbc_1_ * dxbc_1_ + dxbc_2_ * dxbc_2_;
rbc2e = rbc2 + eps2;
rbce = rsqrt(rbc2e);
mr3inv = f02_jp_smem_[j_ + jstride_ * 2].m_j_ * rbce * rbce * rbce;
ac_i_0_wcache_ += f02_jp_smem_[j_ + jstride_ * 2].m_j_ * r1be * r1be * r1be * m_0_ * (4.0 * rbce + 1.25 * r1ce + 0.25 * (r1b2 - rbc2) * r1ce * r1ce * r1ce) * dxb_0_ + mr3inv * m_0_ * (4.0 * r1be + 1.25 * r1ce + 0.25 * (-r1b2 + rbc2) * r1ce * r1ce * r1ce) * dxbc_0_ - 3.5 * f02_jp_smem_[j_ + jstride_ * 2].m_j_ * m_0_ * r1ce * r1ce * r1ce * (rbce - r1be) * dxc_0_ - f02_jp_smem_[j_ + jstride_ * 2].m_j_ * r1be * r1be * r1be * (4.0 * (f02_ip_[isrc_].v_i_0_ * f02_jp_smem_[j_ + jstride_ * 2].v_j_0_ + f02_ip_[isrc_].v_i_1_ * f02_jp_smem_[j_ + jstride_ * 2].v_j_1_ + f02_ip_[isrc_].v_i_2_ * f02_jp_smem_[j_ + jstride_ * 2].v_j_2_) * dxb_0_ - 3.0 * (f02_jp_smem_[j_ + jstride_ * 2].v_j_0_ * dxb_0_ + f02_jp_smem_[j_ + jstride_ * 2].v_j_1_ * dxb_1_ + f02_jp_smem_[j_ + jstride_ * 2].v_j_2_ * dxb_2_) * f02_ip_[isrc_].v_i_0_ - 4.0 * (f02_ip_[isrc_].v_i_0_ * dxb_0_ + f02_ip_[isrc_].v_i_1_ * dxb_1_ + f02_ip_[isrc_].v_i_2_ * dxb_2_) * f02_jp_smem_[j_ + jstride_ * 2].v_j_0_) + mr3inv * (f02_ip_[isrc_].v_i_0_ * f02_ip_[isrc_].v_i_0_ + f02_ip_[isrc_].v_i_1_ * f02_ip_[isrc_].v_i_1_ + f02_ip_[isrc_].v_i_2_ * f02_ip_[isrc_].v_i_2_ - 2.0 * (dvbc_0_ * dvbc_0_ + dvbc_1_ * dvbc_1_ + dvbc_2_ * dvbc_2_) + 1.5 * (f02_jp_smem_[j_ + jstride_ * 2].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 2].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 2].v_j_2_ * dxbc_2_) * (f02_jp_smem_[j_ + jstride_ * 2].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 2].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 2].v_j_2_ * dxbc_2_) / rbc2e) * dxbc_0_ + mr3inv * (dxbc_0_ * (4.0 * f02_ip_[isrc_].v_i_0_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 2].v_j_0_) + dxbc_1_ * (4.0 * f02_ip_[isrc_].v_i_1_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 2].v_j_1_) + dxbc_2_ * (4.0 * f02_ip_[isrc_].v_i_2_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 2].v_j_2_)) * dvbc_0_;
ac_i_1_wcache_ += f02_jp_smem_[j_ + jstride_ * 2].m_j_ * r1be * r1be * r1be * m_0_ * (4.0 * rbce + 1.25 * r1ce + 0.25 * (r1b2 - rbc2) * r1ce * r1ce * r1ce) * dxb_1_ + mr3inv * m_0_ * (4.0 * r1be + 1.25 * r1ce + 0.25 * (-r1b2 + rbc2) * r1ce * r1ce * r1ce) * dxbc_1_ - 3.5 * f02_jp_smem_[j_ + jstride_ * 2].m_j_ * m_0_ * r1ce * r1ce * r1ce * (rbce - r1be) * dxc_1_ - f02_jp_smem_[j_ + jstride_ * 2].m_j_ * r1be * r1be * r1be * (4.0 * (f02_ip_[isrc_].v_i_0_ * f02_jp_smem_[j_ + jstride_ * 2].v_j_0_ + f02_ip_[isrc_].v_i_1_ * f02_jp_smem_[j_ + jstride_ * 2].v_j_1_ + f02_ip_[isrc_].v_i_2_ * f02_jp_smem_[j_ + jstride_ * 2].v_j_2_) * dxb_1_ - 3.0 * (f02_jp_smem_[j_ + jstride_ * 2].v_j_0_ * dxb_0_ + f02_jp_smem_[j_ + jstride_ * 2].v_j_1_ * dxb_1_ + f02_jp_smem_[j_ + jstride_ * 2].v_j_2_ * dxb_2_) * f02_ip_[isrc_].v_i_1_ - 4.0 * (f02_ip_[isrc_].v_i_0_ * dxb_0_ + f02_ip_[isrc_].v_i_1_ * dxb_1_ + f02_ip_[isrc_].v_i_2_ * dxb_2_) * f02_jp_smem_[j_ + jstride_ * 2].v_j_1_) + mr3inv * (f02_ip_[isrc_].v_i_0_ * f02_ip_[isrc_].v_i_0_ + f02_ip_[isrc_].v_i_1_ * f02_ip_[isrc_].v_i_1_ + f02_ip_[isrc_].v_i_2_ * f02_ip_[isrc_].v_i_2_ - 2.0 * (dvbc_0_ * dvbc_0_ + dvbc_1_ * dvbc_1_ + dvbc_2_ * dvbc_2_) + 1.5 * (f02_jp_smem_[j_ + jstride_ * 2].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 2].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 2].v_j_2_ * dxbc_2_) * (f02_jp_smem_[j_ + jstride_ * 2].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 2].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 2].v_j_2_ * dxbc_2_) / rbc2e) * dxbc_1_ + mr3inv * (dxbc_0_ * (4.0 * f02_ip_[isrc_].v_i_0_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 2].v_j_0_) + dxbc_1_ * (4.0 * f02_ip_[isrc_].v_i_1_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 2].v_j_1_) + dxbc_2_ * (4.0 * f02_ip_[isrc_].v_i_2_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 2].v_j_2_)) * dvbc_1_;
ac_i_2_wcache_ += f02_jp_smem_[j_ + jstride_ * 2].m_j_ * r1be * r1be * r1be * m_0_ * (4.0 * rbce + 1.25 * r1ce + 0.25 * (r1b2 - rbc2) * r1ce * r1ce * r1ce) * dxb_2_ + mr3inv * m_0_ * (4.0 * r1be + 1.25 * r1ce + 0.25 * (-r1b2 + rbc2) * r1ce * r1ce * r1ce) * dxbc_2_ - 3.5 * f02_jp_smem_[j_ + jstride_ * 2].m_j_ * m_0_ * r1ce * r1ce * r1ce * (rbce - r1be) * dxc_2_ - f02_jp_smem_[j_ + jstride_ * 2].m_j_ * r1be * r1be * r1be * (4.0 * (f02_ip_[isrc_].v_i_0_ * f02_jp_smem_[j_ + jstride_ * 2].v_j_0_ + f02_ip_[isrc_].v_i_1_ * f02_jp_smem_[j_ + jstride_ * 2].v_j_1_ + f02_ip_[isrc_].v_i_2_ * f02_jp_smem_[j_ + jstride_ * 2].v_j_2_) * dxb_2_ - 3.0 * (f02_jp_smem_[j_ + jstride_ * 2].v_j_0_ * dxb_0_ + f02_jp_smem_[j_ + jstride_ * 2].v_j_1_ * dxb_1_ + f02_jp_smem_[j_ + jstride_ * 2].v_j_2_ * dxb_2_) * f02_ip_[isrc_].v_i_2_ - 4.0 * (f02_ip_[isrc_].v_i_0_ * dxb_0_ + f02_ip_[isrc_].v_i_1_ * dxb_1_ + f02_ip_[isrc_].v_i_2_ * dxb_2_) * f02_jp_smem_[j_ + jstride_ * 2].v_j_2_) + mr3inv * (f02_ip_[isrc_].v_i_0_ * f02_ip_[isrc_].v_i_0_ + f02_ip_[isrc_].v_i_1_ * f02_ip_[isrc_].v_i_1_ + f02_ip_[isrc_].v_i_2_ * f02_ip_[isrc_].v_i_2_ - 2.0 * (dvbc_0_ * dvbc_0_ + dvbc_1_ * dvbc_1_ + dvbc_2_ * dvbc_2_) + 1.5 * (f02_jp_smem_[j_ + jstride_ * 2].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 2].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 2].v_j_2_ * dxbc_2_) * (f02_jp_smem_[j_ + jstride_ * 2].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 2].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 2].v_j_2_ * dxbc_2_) / rbc2e) * dxbc_2_ + mr3inv * (dxbc_0_ * (4.0 * f02_ip_[isrc_].v_i_0_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 2].v_j_0_) + dxbc_1_ * (4.0 * f02_ip_[isrc_].v_i_1_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 2].v_j_1_) + dxbc_2_ * (4.0 * f02_ip_[isrc_].v_i_2_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 2].v_j_2_)) * dvbc_2_;
        // loop 3
dxb_0_ = x_0_0_ - f02_ip_[isrc_].x_i_0_;
dxc_0_ = x_0_0_ - f02_jp_smem_[j_ + jstride_ * 3].x_j_0_;
dxbc_0_ = f02_jp_smem_[j_ + jstride_ * 3].x_j_0_ - f02_ip_[isrc_].x_i_0_;
dvbc_0_ = f02_jp_smem_[j_ + jstride_ * 3].v_j_0_ - f02_ip_[isrc_].v_i_0_;
dxb_1_ = x_0_1_ - f02_ip_[isrc_].x_i_1_;
dxc_1_ = x_0_1_ - f02_jp_smem_[j_ + jstride_ * 3].x_j_1_;
dxbc_1_ = f02_jp_smem_[j_ + jstride_ * 3].x_j_1_ - f02_ip_[isrc_].x_i_1_;
dvbc_1_ = f02_jp_smem_[j_ + jstride_ * 3].v_j_1_ - f02_ip_[isrc_].v_i_1_;
dxb_2_ = x_0_2_ - f02_ip_[isrc_].x_i_2_;
dxc_2_ = x_0_2_ - f02_jp_smem_[j_ + jstride_ * 3].x_j_2_;
dxbc_2_ = f02_jp_smem_[j_ + jstride_ * 3].x_j_2_ - f02_ip_[isrc_].x_i_2_;
dvbc_2_ = f02_jp_smem_[j_ + jstride_ * 3].v_j_2_ - f02_ip_[isrc_].v_i_2_;
r1b2 = dxb_0_ * dxb_0_ + dxb_1_ * dxb_1_ + dxb_2_ * dxb_2_;
r1b2e = r1b2 + eps2;
r1be = rsqrt(r1b2e);
r1c2 = dxc_0_ * dxc_0_ + dxc_1_ * dxc_1_ + dxc_2_ * dxc_2_;
r1c2e = r1c2 + eps2;
r1ce = rsqrt(r1c2e);
rbc2 = dxbc_0_ * dxbc_0_ + dxbc_1_ * dxbc_1_ + dxbc_2_ * dxbc_2_;
rbc2e = rbc2 + eps2;
rbce = rsqrt(rbc2e);
mr3inv = f02_jp_smem_[j_ + jstride_ * 3].m_j_ * rbce * rbce * rbce;
ac_i_0_wcache_ += f02_jp_smem_[j_ + jstride_ * 3].m_j_ * r1be * r1be * r1be * m_0_ * (4.0 * rbce + 1.25 * r1ce + 0.25 * (r1b2 - rbc2) * r1ce * r1ce * r1ce) * dxb_0_ + mr3inv * m_0_ * (4.0 * r1be + 1.25 * r1ce + 0.25 * (-r1b2 + rbc2) * r1ce * r1ce * r1ce) * dxbc_0_ - 3.5 * f02_jp_smem_[j_ + jstride_ * 3].m_j_ * m_0_ * r1ce * r1ce * r1ce * (rbce - r1be) * dxc_0_ - f02_jp_smem_[j_ + jstride_ * 3].m_j_ * r1be * r1be * r1be * (4.0 * (f02_ip_[isrc_].v_i_0_ * f02_jp_smem_[j_ + jstride_ * 3].v_j_0_ + f02_ip_[isrc_].v_i_1_ * f02_jp_smem_[j_ + jstride_ * 3].v_j_1_ + f02_ip_[isrc_].v_i_2_ * f02_jp_smem_[j_ + jstride_ * 3].v_j_2_) * dxb_0_ - 3.0 * (f02_jp_smem_[j_ + jstride_ * 3].v_j_0_ * dxb_0_ + f02_jp_smem_[j_ + jstride_ * 3].v_j_1_ * dxb_1_ + f02_jp_smem_[j_ + jstride_ * 3].v_j_2_ * dxb_2_) * f02_ip_[isrc_].v_i_0_ - 4.0 * (f02_ip_[isrc_].v_i_0_ * dxb_0_ + f02_ip_[isrc_].v_i_1_ * dxb_1_ + f02_ip_[isrc_].v_i_2_ * dxb_2_) * f02_jp_smem_[j_ + jstride_ * 3].v_j_0_) + mr3inv * (f02_ip_[isrc_].v_i_0_ * f02_ip_[isrc_].v_i_0_ + f02_ip_[isrc_].v_i_1_ * f02_ip_[isrc_].v_i_1_ + f02_ip_[isrc_].v_i_2_ * f02_ip_[isrc_].v_i_2_ - 2.0 * (dvbc_0_ * dvbc_0_ + dvbc_1_ * dvbc_1_ + dvbc_2_ * dvbc_2_) + 1.5 * (f02_jp_smem_[j_ + jstride_ * 3].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 3].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 3].v_j_2_ * dxbc_2_) * (f02_jp_smem_[j_ + jstride_ * 3].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 3].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 3].v_j_2_ * dxbc_2_) / rbc2e) * dxbc_0_ + mr3inv * (dxbc_0_ * (4.0 * f02_ip_[isrc_].v_i_0_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 3].v_j_0_) + dxbc_1_ * (4.0 * f02_ip_[isrc_].v_i_1_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 3].v_j_1_) + dxbc_2_ * (4.0 * f02_ip_[isrc_].v_i_2_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 3].v_j_2_)) * dvbc_0_;
ac_i_1_wcache_ += f02_jp_smem_[j_ + jstride_ * 3].m_j_ * r1be * r1be * r1be * m_0_ * (4.0 * rbce + 1.25 * r1ce + 0.25 * (r1b2 - rbc2) * r1ce * r1ce * r1ce) * dxb_1_ + mr3inv * m_0_ * (4.0 * r1be + 1.25 * r1ce + 0.25 * (-r1b2 + rbc2) * r1ce * r1ce * r1ce) * dxbc_1_ - 3.5 * f02_jp_smem_[j_ + jstride_ * 3].m_j_ * m_0_ * r1ce * r1ce * r1ce * (rbce - r1be) * dxc_1_ - f02_jp_smem_[j_ + jstride_ * 3].m_j_ * r1be * r1be * r1be * (4.0 * (f02_ip_[isrc_].v_i_0_ * f02_jp_smem_[j_ + jstride_ * 3].v_j_0_ + f02_ip_[isrc_].v_i_1_ * f02_jp_smem_[j_ + jstride_ * 3].v_j_1_ + f02_ip_[isrc_].v_i_2_ * f02_jp_smem_[j_ + jstride_ * 3].v_j_2_) * dxb_1_ - 3.0 * (f02_jp_smem_[j_ + jstride_ * 3].v_j_0_ * dxb_0_ + f02_jp_smem_[j_ + jstride_ * 3].v_j_1_ * dxb_1_ + f02_jp_smem_[j_ + jstride_ * 3].v_j_2_ * dxb_2_) * f02_ip_[isrc_].v_i_1_ - 4.0 * (f02_ip_[isrc_].v_i_0_ * dxb_0_ + f02_ip_[isrc_].v_i_1_ * dxb_1_ + f02_ip_[isrc_].v_i_2_ * dxb_2_) * f02_jp_smem_[j_ + jstride_ * 3].v_j_1_) + mr3inv * (f02_ip_[isrc_].v_i_0_ * f02_ip_[isrc_].v_i_0_ + f02_ip_[isrc_].v_i_1_ * f02_ip_[isrc_].v_i_1_ + f02_ip_[isrc_].v_i_2_ * f02_ip_[isrc_].v_i_2_ - 2.0 * (dvbc_0_ * dvbc_0_ + dvbc_1_ * dvbc_1_ + dvbc_2_ * dvbc_2_) + 1.5 * (f02_jp_smem_[j_ + jstride_ * 3].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 3].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 3].v_j_2_ * dxbc_2_) * (f02_jp_smem_[j_ + jstride_ * 3].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 3].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 3].v_j_2_ * dxbc_2_) / rbc2e) * dxbc_1_ + mr3inv * (dxbc_0_ * (4.0 * f02_ip_[isrc_].v_i_0_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 3].v_j_0_) + dxbc_1_ * (4.0 * f02_ip_[isrc_].v_i_1_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 3].v_j_1_) + dxbc_2_ * (4.0 * f02_ip_[isrc_].v_i_2_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 3].v_j_2_)) * dvbc_1_;
ac_i_2_wcache_ += f02_jp_smem_[j_ + jstride_ * 3].m_j_ * r1be * r1be * r1be * m_0_ * (4.0 * rbce + 1.25 * r1ce + 0.25 * (r1b2 - rbc2) * r1ce * r1ce * r1ce) * dxb_2_ + mr3inv * m_0_ * (4.0 * r1be + 1.25 * r1ce + 0.25 * (-r1b2 + rbc2) * r1ce * r1ce * r1ce) * dxbc_2_ - 3.5 * f02_jp_smem_[j_ + jstride_ * 3].m_j_ * m_0_ * r1ce * r1ce * r1ce * (rbce - r1be) * dxc_2_ - f02_jp_smem_[j_ + jstride_ * 3].m_j_ * r1be * r1be * r1be * (4.0 * (f02_ip_[isrc_].v_i_0_ * f02_jp_smem_[j_ + jstride_ * 3].v_j_0_ + f02_ip_[isrc_].v_i_1_ * f02_jp_smem_[j_ + jstride_ * 3].v_j_1_ + f02_ip_[isrc_].v_i_2_ * f02_jp_smem_[j_ + jstride_ * 3].v_j_2_) * dxb_2_ - 3.0 * (f02_jp_smem_[j_ + jstride_ * 3].v_j_0_ * dxb_0_ + f02_jp_smem_[j_ + jstride_ * 3].v_j_1_ * dxb_1_ + f02_jp_smem_[j_ + jstride_ * 3].v_j_2_ * dxb_2_) * f02_ip_[isrc_].v_i_2_ - 4.0 * (f02_ip_[isrc_].v_i_0_ * dxb_0_ + f02_ip_[isrc_].v_i_1_ * dxb_1_ + f02_ip_[isrc_].v_i_2_ * dxb_2_) * f02_jp_smem_[j_ + jstride_ * 3].v_j_2_) + mr3inv * (f02_ip_[isrc_].v_i_0_ * f02_ip_[isrc_].v_i_0_ + f02_ip_[isrc_].v_i_1_ * f02_ip_[isrc_].v_i_1_ + f02_ip_[isrc_].v_i_2_ * f02_ip_[isrc_].v_i_2_ - 2.0 * (dvbc_0_ * dvbc_0_ + dvbc_1_ * dvbc_1_ + dvbc_2_ * dvbc_2_) + 1.5 * (f02_jp_smem_[j_ + jstride_ * 3].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 3].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 3].v_j_2_ * dxbc_2_) * (f02_jp_smem_[j_ + jstride_ * 3].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 3].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 3].v_j_2_ * dxbc_2_) / rbc2e) * dxbc_2_ + mr3inv * (dxbc_0_ * (4.0 * f02_ip_[isrc_].v_i_0_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 3].v_j_0_) + dxbc_1_ * (4.0 * f02_ip_[isrc_].v_i_1_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 3].v_j_1_) + dxbc_2_ * (4.0 * f02_ip_[isrc_].v_i_2_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 3].v_j_2_)) * dvbc_2_;
        // loop 4
dxb_0_ = x_0_0_ - f02_ip_[isrc_].x_i_0_;
dxc_0_ = x_0_0_ - f02_jp_smem_[j_ + jstride_ * 4].x_j_0_;
dxbc_0_ = f02_jp_smem_[j_ + jstride_ * 4].x_j_0_ - f02_ip_[isrc_].x_i_0_;
dvbc_0_ = f02_jp_smem_[j_ + jstride_ * 4].v_j_0_ - f02_ip_[isrc_].v_i_0_;
dxb_1_ = x_0_1_ - f02_ip_[isrc_].x_i_1_;
dxc_1_ = x_0_1_ - f02_jp_smem_[j_ + jstride_ * 4].x_j_1_;
dxbc_1_ = f02_jp_smem_[j_ + jstride_ * 4].x_j_1_ - f02_ip_[isrc_].x_i_1_;
dvbc_1_ = f02_jp_smem_[j_ + jstride_ * 4].v_j_1_ - f02_ip_[isrc_].v_i_1_;
dxb_2_ = x_0_2_ - f02_ip_[isrc_].x_i_2_;
dxc_2_ = x_0_2_ - f02_jp_smem_[j_ + jstride_ * 4].x_j_2_;
dxbc_2_ = f02_jp_smem_[j_ + jstride_ * 4].x_j_2_ - f02_ip_[isrc_].x_i_2_;
dvbc_2_ = f02_jp_smem_[j_ + jstride_ * 4].v_j_2_ - f02_ip_[isrc_].v_i_2_;
r1b2 = dxb_0_ * dxb_0_ + dxb_1_ * dxb_1_ + dxb_2_ * dxb_2_;
r1b2e = r1b2 + eps2;
r1be = rsqrt(r1b2e);
r1c2 = dxc_0_ * dxc_0_ + dxc_1_ * dxc_1_ + dxc_2_ * dxc_2_;
r1c2e = r1c2 + eps2;
r1ce = rsqrt(r1c2e);
rbc2 = dxbc_0_ * dxbc_0_ + dxbc_1_ * dxbc_1_ + dxbc_2_ * dxbc_2_;
rbc2e = rbc2 + eps2;
rbce = rsqrt(rbc2e);
mr3inv = f02_jp_smem_[j_ + jstride_ * 4].m_j_ * rbce * rbce * rbce;
ac_i_0_wcache_ += f02_jp_smem_[j_ + jstride_ * 4].m_j_ * r1be * r1be * r1be * m_0_ * (4.0 * rbce + 1.25 * r1ce + 0.25 * (r1b2 - rbc2) * r1ce * r1ce * r1ce) * dxb_0_ + mr3inv * m_0_ * (4.0 * r1be + 1.25 * r1ce + 0.25 * (-r1b2 + rbc2) * r1ce * r1ce * r1ce) * dxbc_0_ - 3.5 * f02_jp_smem_[j_ + jstride_ * 4].m_j_ * m_0_ * r1ce * r1ce * r1ce * (rbce - r1be) * dxc_0_ - f02_jp_smem_[j_ + jstride_ * 4].m_j_ * r1be * r1be * r1be * (4.0 * (f02_ip_[isrc_].v_i_0_ * f02_jp_smem_[j_ + jstride_ * 4].v_j_0_ + f02_ip_[isrc_].v_i_1_ * f02_jp_smem_[j_ + jstride_ * 4].v_j_1_ + f02_ip_[isrc_].v_i_2_ * f02_jp_smem_[j_ + jstride_ * 4].v_j_2_) * dxb_0_ - 3.0 * (f02_jp_smem_[j_ + jstride_ * 4].v_j_0_ * dxb_0_ + f02_jp_smem_[j_ + jstride_ * 4].v_j_1_ * dxb_1_ + f02_jp_smem_[j_ + jstride_ * 4].v_j_2_ * dxb_2_) * f02_ip_[isrc_].v_i_0_ - 4.0 * (f02_ip_[isrc_].v_i_0_ * dxb_0_ + f02_ip_[isrc_].v_i_1_ * dxb_1_ + f02_ip_[isrc_].v_i_2_ * dxb_2_) * f02_jp_smem_[j_ + jstride_ * 4].v_j_0_) + mr3inv * (f02_ip_[isrc_].v_i_0_ * f02_ip_[isrc_].v_i_0_ + f02_ip_[isrc_].v_i_1_ * f02_ip_[isrc_].v_i_1_ + f02_ip_[isrc_].v_i_2_ * f02_ip_[isrc_].v_i_2_ - 2.0 * (dvbc_0_ * dvbc_0_ + dvbc_1_ * dvbc_1_ + dvbc_2_ * dvbc_2_) + 1.5 * (f02_jp_smem_[j_ + jstride_ * 4].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 4].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 4].v_j_2_ * dxbc_2_) * (f02_jp_smem_[j_ + jstride_ * 4].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 4].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 4].v_j_2_ * dxbc_2_) / rbc2e) * dxbc_0_ + mr3inv * (dxbc_0_ * (4.0 * f02_ip_[isrc_].v_i_0_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 4].v_j_0_) + dxbc_1_ * (4.0 * f02_ip_[isrc_].v_i_1_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 4].v_j_1_) + dxbc_2_ * (4.0 * f02_ip_[isrc_].v_i_2_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 4].v_j_2_)) * dvbc_0_;
ac_i_1_wcache_ += f02_jp_smem_[j_ + jstride_ * 4].m_j_ * r1be * r1be * r1be * m_0_ * (4.0 * rbce + 1.25 * r1ce + 0.25 * (r1b2 - rbc2) * r1ce * r1ce * r1ce) * dxb_1_ + mr3inv * m_0_ * (4.0 * r1be + 1.25 * r1ce + 0.25 * (-r1b2 + rbc2) * r1ce * r1ce * r1ce) * dxbc_1_ - 3.5 * f02_jp_smem_[j_ + jstride_ * 4].m_j_ * m_0_ * r1ce * r1ce * r1ce * (rbce - r1be) * dxc_1_ - f02_jp_smem_[j_ + jstride_ * 4].m_j_ * r1be * r1be * r1be * (4.0 * (f02_ip_[isrc_].v_i_0_ * f02_jp_smem_[j_ + jstride_ * 4].v_j_0_ + f02_ip_[isrc_].v_i_1_ * f02_jp_smem_[j_ + jstride_ * 4].v_j_1_ + f02_ip_[isrc_].v_i_2_ * f02_jp_smem_[j_ + jstride_ * 4].v_j_2_) * dxb_1_ - 3.0 * (f02_jp_smem_[j_ + jstride_ * 4].v_j_0_ * dxb_0_ + f02_jp_smem_[j_ + jstride_ * 4].v_j_1_ * dxb_1_ + f02_jp_smem_[j_ + jstride_ * 4].v_j_2_ * dxb_2_) * f02_ip_[isrc_].v_i_1_ - 4.0 * (f02_ip_[isrc_].v_i_0_ * dxb_0_ + f02_ip_[isrc_].v_i_1_ * dxb_1_ + f02_ip_[isrc_].v_i_2_ * dxb_2_) * f02_jp_smem_[j_ + jstride_ * 4].v_j_1_) + mr3inv * (f02_ip_[isrc_].v_i_0_ * f02_ip_[isrc_].v_i_0_ + f02_ip_[isrc_].v_i_1_ * f02_ip_[isrc_].v_i_1_ + f02_ip_[isrc_].v_i_2_ * f02_ip_[isrc_].v_i_2_ - 2.0 * (dvbc_0_ * dvbc_0_ + dvbc_1_ * dvbc_1_ + dvbc_2_ * dvbc_2_) + 1.5 * (f02_jp_smem_[j_ + jstride_ * 4].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 4].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 4].v_j_2_ * dxbc_2_) * (f02_jp_smem_[j_ + jstride_ * 4].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 4].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 4].v_j_2_ * dxbc_2_) / rbc2e) * dxbc_1_ + mr3inv * (dxbc_0_ * (4.0 * f02_ip_[isrc_].v_i_0_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 4].v_j_0_) + dxbc_1_ * (4.0 * f02_ip_[isrc_].v_i_1_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 4].v_j_1_) + dxbc_2_ * (4.0 * f02_ip_[isrc_].v_i_2_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 4].v_j_2_)) * dvbc_1_;
ac_i_2_wcache_ += f02_jp_smem_[j_ + jstride_ * 4].m_j_ * r1be * r1be * r1be * m_0_ * (4.0 * rbce + 1.25 * r1ce + 0.25 * (r1b2 - rbc2) * r1ce * r1ce * r1ce) * dxb_2_ + mr3inv * m_0_ * (4.0 * r1be + 1.25 * r1ce + 0.25 * (-r1b2 + rbc2) * r1ce * r1ce * r1ce) * dxbc_2_ - 3.5 * f02_jp_smem_[j_ + jstride_ * 4].m_j_ * m_0_ * r1ce * r1ce * r1ce * (rbce - r1be) * dxc_2_ - f02_jp_smem_[j_ + jstride_ * 4].m_j_ * r1be * r1be * r1be * (4.0 * (f02_ip_[isrc_].v_i_0_ * f02_jp_smem_[j_ + jstride_ * 4].v_j_0_ + f02_ip_[isrc_].v_i_1_ * f02_jp_smem_[j_ + jstride_ * 4].v_j_1_ + f02_ip_[isrc_].v_i_2_ * f02_jp_smem_[j_ + jstride_ * 4].v_j_2_) * dxb_2_ - 3.0 * (f02_jp_smem_[j_ + jstride_ * 4].v_j_0_ * dxb_0_ + f02_jp_smem_[j_ + jstride_ * 4].v_j_1_ * dxb_1_ + f02_jp_smem_[j_ + jstride_ * 4].v_j_2_ * dxb_2_) * f02_ip_[isrc_].v_i_2_ - 4.0 * (f02_ip_[isrc_].v_i_0_ * dxb_0_ + f02_ip_[isrc_].v_i_1_ * dxb_1_ + f02_ip_[isrc_].v_i_2_ * dxb_2_) * f02_jp_smem_[j_ + jstride_ * 4].v_j_2_) + mr3inv * (f02_ip_[isrc_].v_i_0_ * f02_ip_[isrc_].v_i_0_ + f02_ip_[isrc_].v_i_1_ * f02_ip_[isrc_].v_i_1_ + f02_ip_[isrc_].v_i_2_ * f02_ip_[isrc_].v_i_2_ - 2.0 * (dvbc_0_ * dvbc_0_ + dvbc_1_ * dvbc_1_ + dvbc_2_ * dvbc_2_) + 1.5 * (f02_jp_smem_[j_ + jstride_ * 4].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 4].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 4].v_j_2_ * dxbc_2_) * (f02_jp_smem_[j_ + jstride_ * 4].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 4].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 4].v_j_2_ * dxbc_2_) / rbc2e) * dxbc_2_ + mr3inv * (dxbc_0_ * (4.0 * f02_ip_[isrc_].v_i_0_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 4].v_j_0_) + dxbc_1_ * (4.0 * f02_ip_[isrc_].v_i_1_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 4].v_j_1_) + dxbc_2_ * (4.0 * f02_ip_[isrc_].v_i_2_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 4].v_j_2_)) * dvbc_2_;
        // loop 5
dxb_0_ = x_0_0_ - f02_ip_[isrc_].x_i_0_;
dxc_0_ = x_0_0_ - f02_jp_smem_[j_ + jstride_ * 5].x_j_0_;
dxbc_0_ = f02_jp_smem_[j_ + jstride_ * 5].x_j_0_ - f02_ip_[isrc_].x_i_0_;
dvbc_0_ = f02_jp_smem_[j_ + jstride_ * 5].v_j_0_ - f02_ip_[isrc_].v_i_0_;
dxb_1_ = x_0_1_ - f02_ip_[isrc_].x_i_1_;
dxc_1_ = x_0_1_ - f02_jp_smem_[j_ + jstride_ * 5].x_j_1_;
dxbc_1_ = f02_jp_smem_[j_ + jstride_ * 5].x_j_1_ - f02_ip_[isrc_].x_i_1_;
dvbc_1_ = f02_jp_smem_[j_ + jstride_ * 5].v_j_1_ - f02_ip_[isrc_].v_i_1_;
dxb_2_ = x_0_2_ - f02_ip_[isrc_].x_i_2_;
dxc_2_ = x_0_2_ - f02_jp_smem_[j_ + jstride_ * 5].x_j_2_;
dxbc_2_ = f02_jp_smem_[j_ + jstride_ * 5].x_j_2_ - f02_ip_[isrc_].x_i_2_;
dvbc_2_ = f02_jp_smem_[j_ + jstride_ * 5].v_j_2_ - f02_ip_[isrc_].v_i_2_;
r1b2 = dxb_0_ * dxb_0_ + dxb_1_ * dxb_1_ + dxb_2_ * dxb_2_;
r1b2e = r1b2 + eps2;
r1be = rsqrt(r1b2e);
r1c2 = dxc_0_ * dxc_0_ + dxc_1_ * dxc_1_ + dxc_2_ * dxc_2_;
r1c2e = r1c2 + eps2;
r1ce = rsqrt(r1c2e);
rbc2 = dxbc_0_ * dxbc_0_ + dxbc_1_ * dxbc_1_ + dxbc_2_ * dxbc_2_;
rbc2e = rbc2 + eps2;
rbce = rsqrt(rbc2e);
mr3inv = f02_jp_smem_[j_ + jstride_ * 5].m_j_ * rbce * rbce * rbce;
ac_i_0_wcache_ += f02_jp_smem_[j_ + jstride_ * 5].m_j_ * r1be * r1be * r1be * m_0_ * (4.0 * rbce + 1.25 * r1ce + 0.25 * (r1b2 - rbc2) * r1ce * r1ce * r1ce) * dxb_0_ + mr3inv * m_0_ * (4.0 * r1be + 1.25 * r1ce + 0.25 * (-r1b2 + rbc2) * r1ce * r1ce * r1ce) * dxbc_0_ - 3.5 * f02_jp_smem_[j_ + jstride_ * 5].m_j_ * m_0_ * r1ce * r1ce * r1ce * (rbce - r1be) * dxc_0_ - f02_jp_smem_[j_ + jstride_ * 5].m_j_ * r1be * r1be * r1be * (4.0 * (f02_ip_[isrc_].v_i_0_ * f02_jp_smem_[j_ + jstride_ * 5].v_j_0_ + f02_ip_[isrc_].v_i_1_ * f02_jp_smem_[j_ + jstride_ * 5].v_j_1_ + f02_ip_[isrc_].v_i_2_ * f02_jp_smem_[j_ + jstride_ * 5].v_j_2_) * dxb_0_ - 3.0 * (f02_jp_smem_[j_ + jstride_ * 5].v_j_0_ * dxb_0_ + f02_jp_smem_[j_ + jstride_ * 5].v_j_1_ * dxb_1_ + f02_jp_smem_[j_ + jstride_ * 5].v_j_2_ * dxb_2_) * f02_ip_[isrc_].v_i_0_ - 4.0 * (f02_ip_[isrc_].v_i_0_ * dxb_0_ + f02_ip_[isrc_].v_i_1_ * dxb_1_ + f02_ip_[isrc_].v_i_2_ * dxb_2_) * f02_jp_smem_[j_ + jstride_ * 5].v_j_0_) + mr3inv * (f02_ip_[isrc_].v_i_0_ * f02_ip_[isrc_].v_i_0_ + f02_ip_[isrc_].v_i_1_ * f02_ip_[isrc_].v_i_1_ + f02_ip_[isrc_].v_i_2_ * f02_ip_[isrc_].v_i_2_ - 2.0 * (dvbc_0_ * dvbc_0_ + dvbc_1_ * dvbc_1_ + dvbc_2_ * dvbc_2_) + 1.5 * (f02_jp_smem_[j_ + jstride_ * 5].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 5].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 5].v_j_2_ * dxbc_2_) * (f02_jp_smem_[j_ + jstride_ * 5].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 5].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 5].v_j_2_ * dxbc_2_) / rbc2e) * dxbc_0_ + mr3inv * (dxbc_0_ * (4.0 * f02_ip_[isrc_].v_i_0_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 5].v_j_0_) + dxbc_1_ * (4.0 * f02_ip_[isrc_].v_i_1_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 5].v_j_1_) + dxbc_2_ * (4.0 * f02_ip_[isrc_].v_i_2_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 5].v_j_2_)) * dvbc_0_;
ac_i_1_wcache_ += f02_jp_smem_[j_ + jstride_ * 5].m_j_ * r1be * r1be * r1be * m_0_ * (4.0 * rbce + 1.25 * r1ce + 0.25 * (r1b2 - rbc2) * r1ce * r1ce * r1ce) * dxb_1_ + mr3inv * m_0_ * (4.0 * r1be + 1.25 * r1ce + 0.25 * (-r1b2 + rbc2) * r1ce * r1ce * r1ce) * dxbc_1_ - 3.5 * f02_jp_smem_[j_ + jstride_ * 5].m_j_ * m_0_ * r1ce * r1ce * r1ce * (rbce - r1be) * dxc_1_ - f02_jp_smem_[j_ + jstride_ * 5].m_j_ * r1be * r1be * r1be * (4.0 * (f02_ip_[isrc_].v_i_0_ * f02_jp_smem_[j_ + jstride_ * 5].v_j_0_ + f02_ip_[isrc_].v_i_1_ * f02_jp_smem_[j_ + jstride_ * 5].v_j_1_ + f02_ip_[isrc_].v_i_2_ * f02_jp_smem_[j_ + jstride_ * 5].v_j_2_) * dxb_1_ - 3.0 * (f02_jp_smem_[j_ + jstride_ * 5].v_j_0_ * dxb_0_ + f02_jp_smem_[j_ + jstride_ * 5].v_j_1_ * dxb_1_ + f02_jp_smem_[j_ + jstride_ * 5].v_j_2_ * dxb_2_) * f02_ip_[isrc_].v_i_1_ - 4.0 * (f02_ip_[isrc_].v_i_0_ * dxb_0_ + f02_ip_[isrc_].v_i_1_ * dxb_1_ + f02_ip_[isrc_].v_i_2_ * dxb_2_) * f02_jp_smem_[j_ + jstride_ * 5].v_j_1_) + mr3inv * (f02_ip_[isrc_].v_i_0_ * f02_ip_[isrc_].v_i_0_ + f02_ip_[isrc_].v_i_1_ * f02_ip_[isrc_].v_i_1_ + f02_ip_[isrc_].v_i_2_ * f02_ip_[isrc_].v_i_2_ - 2.0 * (dvbc_0_ * dvbc_0_ + dvbc_1_ * dvbc_1_ + dvbc_2_ * dvbc_2_) + 1.5 * (f02_jp_smem_[j_ + jstride_ * 5].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 5].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 5].v_j_2_ * dxbc_2_) * (f02_jp_smem_[j_ + jstride_ * 5].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 5].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 5].v_j_2_ * dxbc_2_) / rbc2e) * dxbc_1_ + mr3inv * (dxbc_0_ * (4.0 * f02_ip_[isrc_].v_i_0_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 5].v_j_0_) + dxbc_1_ * (4.0 * f02_ip_[isrc_].v_i_1_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 5].v_j_1_) + dxbc_2_ * (4.0 * f02_ip_[isrc_].v_i_2_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 5].v_j_2_)) * dvbc_1_;
ac_i_2_wcache_ += f02_jp_smem_[j_ + jstride_ * 5].m_j_ * r1be * r1be * r1be * m_0_ * (4.0 * rbce + 1.25 * r1ce + 0.25 * (r1b2 - rbc2) * r1ce * r1ce * r1ce) * dxb_2_ + mr3inv * m_0_ * (4.0 * r1be + 1.25 * r1ce + 0.25 * (-r1b2 + rbc2) * r1ce * r1ce * r1ce) * dxbc_2_ - 3.5 * f02_jp_smem_[j_ + jstride_ * 5].m_j_ * m_0_ * r1ce * r1ce * r1ce * (rbce - r1be) * dxc_2_ - f02_jp_smem_[j_ + jstride_ * 5].m_j_ * r1be * r1be * r1be * (4.0 * (f02_ip_[isrc_].v_i_0_ * f02_jp_smem_[j_ + jstride_ * 5].v_j_0_ + f02_ip_[isrc_].v_i_1_ * f02_jp_smem_[j_ + jstride_ * 5].v_j_1_ + f02_ip_[isrc_].v_i_2_ * f02_jp_smem_[j_ + jstride_ * 5].v_j_2_) * dxb_2_ - 3.0 * (f02_jp_smem_[j_ + jstride_ * 5].v_j_0_ * dxb_0_ + f02_jp_smem_[j_ + jstride_ * 5].v_j_1_ * dxb_1_ + f02_jp_smem_[j_ + jstride_ * 5].v_j_2_ * dxb_2_) * f02_ip_[isrc_].v_i_2_ - 4.0 * (f02_ip_[isrc_].v_i_0_ * dxb_0_ + f02_ip_[isrc_].v_i_1_ * dxb_1_ + f02_ip_[isrc_].v_i_2_ * dxb_2_) * f02_jp_smem_[j_ + jstride_ * 5].v_j_2_) + mr3inv * (f02_ip_[isrc_].v_i_0_ * f02_ip_[isrc_].v_i_0_ + f02_ip_[isrc_].v_i_1_ * f02_ip_[isrc_].v_i_1_ + f02_ip_[isrc_].v_i_2_ * f02_ip_[isrc_].v_i_2_ - 2.0 * (dvbc_0_ * dvbc_0_ + dvbc_1_ * dvbc_1_ + dvbc_2_ * dvbc_2_) + 1.5 * (f02_jp_smem_[j_ + jstride_ * 5].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 5].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 5].v_j_2_ * dxbc_2_) * (f02_jp_smem_[j_ + jstride_ * 5].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 5].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 5].v_j_2_ * dxbc_2_) / rbc2e) * dxbc_2_ + mr3inv * (dxbc_0_ * (4.0 * f02_ip_[isrc_].v_i_0_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 5].v_j_0_) + dxbc_1_ * (4.0 * f02_ip_[isrc_].v_i_1_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 5].v_j_1_) + dxbc_2_ * (4.0 * f02_ip_[isrc_].v_i_2_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 5].v_j_2_)) * dvbc_2_;
        // loop 6
dxb_0_ = x_0_0_ - f02_ip_[isrc_].x_i_0_;
dxc_0_ = x_0_0_ - f02_jp_smem_[j_ + jstride_ * 6].x_j_0_;
dxbc_0_ = f02_jp_smem_[j_ + jstride_ * 6].x_j_0_ - f02_ip_[isrc_].x_i_0_;
dvbc_0_ = f02_jp_smem_[j_ + jstride_ * 6].v_j_0_ - f02_ip_[isrc_].v_i_0_;
dxb_1_ = x_0_1_ - f02_ip_[isrc_].x_i_1_;
dxc_1_ = x_0_1_ - f02_jp_smem_[j_ + jstride_ * 6].x_j_1_;
dxbc_1_ = f02_jp_smem_[j_ + jstride_ * 6].x_j_1_ - f02_ip_[isrc_].x_i_1_;
dvbc_1_ = f02_jp_smem_[j_ + jstride_ * 6].v_j_1_ - f02_ip_[isrc_].v_i_1_;
dxb_2_ = x_0_2_ - f02_ip_[isrc_].x_i_2_;
dxc_2_ = x_0_2_ - f02_jp_smem_[j_ + jstride_ * 6].x_j_2_;
dxbc_2_ = f02_jp_smem_[j_ + jstride_ * 6].x_j_2_ - f02_ip_[isrc_].x_i_2_;
dvbc_2_ = f02_jp_smem_[j_ + jstride_ * 6].v_j_2_ - f02_ip_[isrc_].v_i_2_;
r1b2 = dxb_0_ * dxb_0_ + dxb_1_ * dxb_1_ + dxb_2_ * dxb_2_;
r1b2e = r1b2 + eps2;
r1be = rsqrt(r1b2e);
r1c2 = dxc_0_ * dxc_0_ + dxc_1_ * dxc_1_ + dxc_2_ * dxc_2_;
r1c2e = r1c2 + eps2;
r1ce = rsqrt(r1c2e);
rbc2 = dxbc_0_ * dxbc_0_ + dxbc_1_ * dxbc_1_ + dxbc_2_ * dxbc_2_;
rbc2e = rbc2 + eps2;
rbce = rsqrt(rbc2e);
mr3inv = f02_jp_smem_[j_ + jstride_ * 6].m_j_ * rbce * rbce * rbce;
ac_i_0_wcache_ += f02_jp_smem_[j_ + jstride_ * 6].m_j_ * r1be * r1be * r1be * m_0_ * (4.0 * rbce + 1.25 * r1ce + 0.25 * (r1b2 - rbc2) * r1ce * r1ce * r1ce) * dxb_0_ + mr3inv * m_0_ * (4.0 * r1be + 1.25 * r1ce + 0.25 * (-r1b2 + rbc2) * r1ce * r1ce * r1ce) * dxbc_0_ - 3.5 * f02_jp_smem_[j_ + jstride_ * 6].m_j_ * m_0_ * r1ce * r1ce * r1ce * (rbce - r1be) * dxc_0_ - f02_jp_smem_[j_ + jstride_ * 6].m_j_ * r1be * r1be * r1be * (4.0 * (f02_ip_[isrc_].v_i_0_ * f02_jp_smem_[j_ + jstride_ * 6].v_j_0_ + f02_ip_[isrc_].v_i_1_ * f02_jp_smem_[j_ + jstride_ * 6].v_j_1_ + f02_ip_[isrc_].v_i_2_ * f02_jp_smem_[j_ + jstride_ * 6].v_j_2_) * dxb_0_ - 3.0 * (f02_jp_smem_[j_ + jstride_ * 6].v_j_0_ * dxb_0_ + f02_jp_smem_[j_ + jstride_ * 6].v_j_1_ * dxb_1_ + f02_jp_smem_[j_ + jstride_ * 6].v_j_2_ * dxb_2_) * f02_ip_[isrc_].v_i_0_ - 4.0 * (f02_ip_[isrc_].v_i_0_ * dxb_0_ + f02_ip_[isrc_].v_i_1_ * dxb_1_ + f02_ip_[isrc_].v_i_2_ * dxb_2_) * f02_jp_smem_[j_ + jstride_ * 6].v_j_0_) + mr3inv * (f02_ip_[isrc_].v_i_0_ * f02_ip_[isrc_].v_i_0_ + f02_ip_[isrc_].v_i_1_ * f02_ip_[isrc_].v_i_1_ + f02_ip_[isrc_].v_i_2_ * f02_ip_[isrc_].v_i_2_ - 2.0 * (dvbc_0_ * dvbc_0_ + dvbc_1_ * dvbc_1_ + dvbc_2_ * dvbc_2_) + 1.5 * (f02_jp_smem_[j_ + jstride_ * 6].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 6].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 6].v_j_2_ * dxbc_2_) * (f02_jp_smem_[j_ + jstride_ * 6].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 6].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 6].v_j_2_ * dxbc_2_) / rbc2e) * dxbc_0_ + mr3inv * (dxbc_0_ * (4.0 * f02_ip_[isrc_].v_i_0_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 6].v_j_0_) + dxbc_1_ * (4.0 * f02_ip_[isrc_].v_i_1_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 6].v_j_1_) + dxbc_2_ * (4.0 * f02_ip_[isrc_].v_i_2_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 6].v_j_2_)) * dvbc_0_;
ac_i_1_wcache_ += f02_jp_smem_[j_ + jstride_ * 6].m_j_ * r1be * r1be * r1be * m_0_ * (4.0 * rbce + 1.25 * r1ce + 0.25 * (r1b2 - rbc2) * r1ce * r1ce * r1ce) * dxb_1_ + mr3inv * m_0_ * (4.0 * r1be + 1.25 * r1ce + 0.25 * (-r1b2 + rbc2) * r1ce * r1ce * r1ce) * dxbc_1_ - 3.5 * f02_jp_smem_[j_ + jstride_ * 6].m_j_ * m_0_ * r1ce * r1ce * r1ce * (rbce - r1be) * dxc_1_ - f02_jp_smem_[j_ + jstride_ * 6].m_j_ * r1be * r1be * r1be * (4.0 * (f02_ip_[isrc_].v_i_0_ * f02_jp_smem_[j_ + jstride_ * 6].v_j_0_ + f02_ip_[isrc_].v_i_1_ * f02_jp_smem_[j_ + jstride_ * 6].v_j_1_ + f02_ip_[isrc_].v_i_2_ * f02_jp_smem_[j_ + jstride_ * 6].v_j_2_) * dxb_1_ - 3.0 * (f02_jp_smem_[j_ + jstride_ * 6].v_j_0_ * dxb_0_ + f02_jp_smem_[j_ + jstride_ * 6].v_j_1_ * dxb_1_ + f02_jp_smem_[j_ + jstride_ * 6].v_j_2_ * dxb_2_) * f02_ip_[isrc_].v_i_1_ - 4.0 * (f02_ip_[isrc_].v_i_0_ * dxb_0_ + f02_ip_[isrc_].v_i_1_ * dxb_1_ + f02_ip_[isrc_].v_i_2_ * dxb_2_) * f02_jp_smem_[j_ + jstride_ * 6].v_j_1_) + mr3inv * (f02_ip_[isrc_].v_i_0_ * f02_ip_[isrc_].v_i_0_ + f02_ip_[isrc_].v_i_1_ * f02_ip_[isrc_].v_i_1_ + f02_ip_[isrc_].v_i_2_ * f02_ip_[isrc_].v_i_2_ - 2.0 * (dvbc_0_ * dvbc_0_ + dvbc_1_ * dvbc_1_ + dvbc_2_ * dvbc_2_) + 1.5 * (f02_jp_smem_[j_ + jstride_ * 6].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 6].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 6].v_j_2_ * dxbc_2_) * (f02_jp_smem_[j_ + jstride_ * 6].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 6].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 6].v_j_2_ * dxbc_2_) / rbc2e) * dxbc_1_ + mr3inv * (dxbc_0_ * (4.0 * f02_ip_[isrc_].v_i_0_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 6].v_j_0_) + dxbc_1_ * (4.0 * f02_ip_[isrc_].v_i_1_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 6].v_j_1_) + dxbc_2_ * (4.0 * f02_ip_[isrc_].v_i_2_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 6].v_j_2_)) * dvbc_1_;
ac_i_2_wcache_ += f02_jp_smem_[j_ + jstride_ * 6].m_j_ * r1be * r1be * r1be * m_0_ * (4.0 * rbce + 1.25 * r1ce + 0.25 * (r1b2 - rbc2) * r1ce * r1ce * r1ce) * dxb_2_ + mr3inv * m_0_ * (4.0 * r1be + 1.25 * r1ce + 0.25 * (-r1b2 + rbc2) * r1ce * r1ce * r1ce) * dxbc_2_ - 3.5 * f02_jp_smem_[j_ + jstride_ * 6].m_j_ * m_0_ * r1ce * r1ce * r1ce * (rbce - r1be) * dxc_2_ - f02_jp_smem_[j_ + jstride_ * 6].m_j_ * r1be * r1be * r1be * (4.0 * (f02_ip_[isrc_].v_i_0_ * f02_jp_smem_[j_ + jstride_ * 6].v_j_0_ + f02_ip_[isrc_].v_i_1_ * f02_jp_smem_[j_ + jstride_ * 6].v_j_1_ + f02_ip_[isrc_].v_i_2_ * f02_jp_smem_[j_ + jstride_ * 6].v_j_2_) * dxb_2_ - 3.0 * (f02_jp_smem_[j_ + jstride_ * 6].v_j_0_ * dxb_0_ + f02_jp_smem_[j_ + jstride_ * 6].v_j_1_ * dxb_1_ + f02_jp_smem_[j_ + jstride_ * 6].v_j_2_ * dxb_2_) * f02_ip_[isrc_].v_i_2_ - 4.0 * (f02_ip_[isrc_].v_i_0_ * dxb_0_ + f02_ip_[isrc_].v_i_1_ * dxb_1_ + f02_ip_[isrc_].v_i_2_ * dxb_2_) * f02_jp_smem_[j_ + jstride_ * 6].v_j_2_) + mr3inv * (f02_ip_[isrc_].v_i_0_ * f02_ip_[isrc_].v_i_0_ + f02_ip_[isrc_].v_i_1_ * f02_ip_[isrc_].v_i_1_ + f02_ip_[isrc_].v_i_2_ * f02_ip_[isrc_].v_i_2_ - 2.0 * (dvbc_0_ * dvbc_0_ + dvbc_1_ * dvbc_1_ + dvbc_2_ * dvbc_2_) + 1.5 * (f02_jp_smem_[j_ + jstride_ * 6].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 6].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 6].v_j_2_ * dxbc_2_) * (f02_jp_smem_[j_ + jstride_ * 6].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 6].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 6].v_j_2_ * dxbc_2_) / rbc2e) * dxbc_2_ + mr3inv * (dxbc_0_ * (4.0 * f02_ip_[isrc_].v_i_0_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 6].v_j_0_) + dxbc_1_ * (4.0 * f02_ip_[isrc_].v_i_1_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 6].v_j_1_) + dxbc_2_ * (4.0 * f02_ip_[isrc_].v_i_2_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 6].v_j_2_)) * dvbc_2_;
        // loop 7
dxb_0_ = x_0_0_ - f02_ip_[isrc_].x_i_0_;
dxc_0_ = x_0_0_ - f02_jp_smem_[j_ + jstride_ * 7].x_j_0_;
dxbc_0_ = f02_jp_smem_[j_ + jstride_ * 7].x_j_0_ - f02_ip_[isrc_].x_i_0_;
dvbc_0_ = f02_jp_smem_[j_ + jstride_ * 7].v_j_0_ - f02_ip_[isrc_].v_i_0_;
dxb_1_ = x_0_1_ - f02_ip_[isrc_].x_i_1_;
dxc_1_ = x_0_1_ - f02_jp_smem_[j_ + jstride_ * 7].x_j_1_;
dxbc_1_ = f02_jp_smem_[j_ + jstride_ * 7].x_j_1_ - f02_ip_[isrc_].x_i_1_;
dvbc_1_ = f02_jp_smem_[j_ + jstride_ * 7].v_j_1_ - f02_ip_[isrc_].v_i_1_;
dxb_2_ = x_0_2_ - f02_ip_[isrc_].x_i_2_;
dxc_2_ = x_0_2_ - f02_jp_smem_[j_ + jstride_ * 7].x_j_2_;
dxbc_2_ = f02_jp_smem_[j_ + jstride_ * 7].x_j_2_ - f02_ip_[isrc_].x_i_2_;
dvbc_2_ = f02_jp_smem_[j_ + jstride_ * 7].v_j_2_ - f02_ip_[isrc_].v_i_2_;
r1b2 = dxb_0_ * dxb_0_ + dxb_1_ * dxb_1_ + dxb_2_ * dxb_2_;
r1b2e = r1b2 + eps2;
r1be = rsqrt(r1b2e);
r1c2 = dxc_0_ * dxc_0_ + dxc_1_ * dxc_1_ + dxc_2_ * dxc_2_;
r1c2e = r1c2 + eps2;
r1ce = rsqrt(r1c2e);
rbc2 = dxbc_0_ * dxbc_0_ + dxbc_1_ * dxbc_1_ + dxbc_2_ * dxbc_2_;
rbc2e = rbc2 + eps2;
rbce = rsqrt(rbc2e);
mr3inv = f02_jp_smem_[j_ + jstride_ * 7].m_j_ * rbce * rbce * rbce;
ac_i_0_wcache_ += f02_jp_smem_[j_ + jstride_ * 7].m_j_ * r1be * r1be * r1be * m_0_ * (4.0 * rbce + 1.25 * r1ce + 0.25 * (r1b2 - rbc2) * r1ce * r1ce * r1ce) * dxb_0_ + mr3inv * m_0_ * (4.0 * r1be + 1.25 * r1ce + 0.25 * (-r1b2 + rbc2) * r1ce * r1ce * r1ce) * dxbc_0_ - 3.5 * f02_jp_smem_[j_ + jstride_ * 7].m_j_ * m_0_ * r1ce * r1ce * r1ce * (rbce - r1be) * dxc_0_ - f02_jp_smem_[j_ + jstride_ * 7].m_j_ * r1be * r1be * r1be * (4.0 * (f02_ip_[isrc_].v_i_0_ * f02_jp_smem_[j_ + jstride_ * 7].v_j_0_ + f02_ip_[isrc_].v_i_1_ * f02_jp_smem_[j_ + jstride_ * 7].v_j_1_ + f02_ip_[isrc_].v_i_2_ * f02_jp_smem_[j_ + jstride_ * 7].v_j_2_) * dxb_0_ - 3.0 * (f02_jp_smem_[j_ + jstride_ * 7].v_j_0_ * dxb_0_ + f02_jp_smem_[j_ + jstride_ * 7].v_j_1_ * dxb_1_ + f02_jp_smem_[j_ + jstride_ * 7].v_j_2_ * dxb_2_) * f02_ip_[isrc_].v_i_0_ - 4.0 * (f02_ip_[isrc_].v_i_0_ * dxb_0_ + f02_ip_[isrc_].v_i_1_ * dxb_1_ + f02_ip_[isrc_].v_i_2_ * dxb_2_) * f02_jp_smem_[j_ + jstride_ * 7].v_j_0_) + mr3inv * (f02_ip_[isrc_].v_i_0_ * f02_ip_[isrc_].v_i_0_ + f02_ip_[isrc_].v_i_1_ * f02_ip_[isrc_].v_i_1_ + f02_ip_[isrc_].v_i_2_ * f02_ip_[isrc_].v_i_2_ - 2.0 * (dvbc_0_ * dvbc_0_ + dvbc_1_ * dvbc_1_ + dvbc_2_ * dvbc_2_) + 1.5 * (f02_jp_smem_[j_ + jstride_ * 7].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 7].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 7].v_j_2_ * dxbc_2_) * (f02_jp_smem_[j_ + jstride_ * 7].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 7].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 7].v_j_2_ * dxbc_2_) / rbc2e) * dxbc_0_ + mr3inv * (dxbc_0_ * (4.0 * f02_ip_[isrc_].v_i_0_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 7].v_j_0_) + dxbc_1_ * (4.0 * f02_ip_[isrc_].v_i_1_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 7].v_j_1_) + dxbc_2_ * (4.0 * f02_ip_[isrc_].v_i_2_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 7].v_j_2_)) * dvbc_0_;
ac_i_1_wcache_ += f02_jp_smem_[j_ + jstride_ * 7].m_j_ * r1be * r1be * r1be * m_0_ * (4.0 * rbce + 1.25 * r1ce + 0.25 * (r1b2 - rbc2) * r1ce * r1ce * r1ce) * dxb_1_ + mr3inv * m_0_ * (4.0 * r1be + 1.25 * r1ce + 0.25 * (-r1b2 + rbc2) * r1ce * r1ce * r1ce) * dxbc_1_ - 3.5 * f02_jp_smem_[j_ + jstride_ * 7].m_j_ * m_0_ * r1ce * r1ce * r1ce * (rbce - r1be) * dxc_1_ - f02_jp_smem_[j_ + jstride_ * 7].m_j_ * r1be * r1be * r1be * (4.0 * (f02_ip_[isrc_].v_i_0_ * f02_jp_smem_[j_ + jstride_ * 7].v_j_0_ + f02_ip_[isrc_].v_i_1_ * f02_jp_smem_[j_ + jstride_ * 7].v_j_1_ + f02_ip_[isrc_].v_i_2_ * f02_jp_smem_[j_ + jstride_ * 7].v_j_2_) * dxb_1_ - 3.0 * (f02_jp_smem_[j_ + jstride_ * 7].v_j_0_ * dxb_0_ + f02_jp_smem_[j_ + jstride_ * 7].v_j_1_ * dxb_1_ + f02_jp_smem_[j_ + jstride_ * 7].v_j_2_ * dxb_2_) * f02_ip_[isrc_].v_i_1_ - 4.0 * (f02_ip_[isrc_].v_i_0_ * dxb_0_ + f02_ip_[isrc_].v_i_1_ * dxb_1_ + f02_ip_[isrc_].v_i_2_ * dxb_2_) * f02_jp_smem_[j_ + jstride_ * 7].v_j_1_) + mr3inv * (f02_ip_[isrc_].v_i_0_ * f02_ip_[isrc_].v_i_0_ + f02_ip_[isrc_].v_i_1_ * f02_ip_[isrc_].v_i_1_ + f02_ip_[isrc_].v_i_2_ * f02_ip_[isrc_].v_i_2_ - 2.0 * (dvbc_0_ * dvbc_0_ + dvbc_1_ * dvbc_1_ + dvbc_2_ * dvbc_2_) + 1.5 * (f02_jp_smem_[j_ + jstride_ * 7].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 7].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 7].v_j_2_ * dxbc_2_) * (f02_jp_smem_[j_ + jstride_ * 7].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 7].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 7].v_j_2_ * dxbc_2_) / rbc2e) * dxbc_1_ + mr3inv * (dxbc_0_ * (4.0 * f02_ip_[isrc_].v_i_0_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 7].v_j_0_) + dxbc_1_ * (4.0 * f02_ip_[isrc_].v_i_1_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 7].v_j_1_) + dxbc_2_ * (4.0 * f02_ip_[isrc_].v_i_2_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 7].v_j_2_)) * dvbc_1_;
ac_i_2_wcache_ += f02_jp_smem_[j_ + jstride_ * 7].m_j_ * r1be * r1be * r1be * m_0_ * (4.0 * rbce + 1.25 * r1ce + 0.25 * (r1b2 - rbc2) * r1ce * r1ce * r1ce) * dxb_2_ + mr3inv * m_0_ * (4.0 * r1be + 1.25 * r1ce + 0.25 * (-r1b2 + rbc2) * r1ce * r1ce * r1ce) * dxbc_2_ - 3.5 * f02_jp_smem_[j_ + jstride_ * 7].m_j_ * m_0_ * r1ce * r1ce * r1ce * (rbce - r1be) * dxc_2_ - f02_jp_smem_[j_ + jstride_ * 7].m_j_ * r1be * r1be * r1be * (4.0 * (f02_ip_[isrc_].v_i_0_ * f02_jp_smem_[j_ + jstride_ * 7].v_j_0_ + f02_ip_[isrc_].v_i_1_ * f02_jp_smem_[j_ + jstride_ * 7].v_j_1_ + f02_ip_[isrc_].v_i_2_ * f02_jp_smem_[j_ + jstride_ * 7].v_j_2_) * dxb_2_ - 3.0 * (f02_jp_smem_[j_ + jstride_ * 7].v_j_0_ * dxb_0_ + f02_jp_smem_[j_ + jstride_ * 7].v_j_1_ * dxb_1_ + f02_jp_smem_[j_ + jstride_ * 7].v_j_2_ * dxb_2_) * f02_ip_[isrc_].v_i_2_ - 4.0 * (f02_ip_[isrc_].v_i_0_ * dxb_0_ + f02_ip_[isrc_].v_i_1_ * dxb_1_ + f02_ip_[isrc_].v_i_2_ * dxb_2_) * f02_jp_smem_[j_ + jstride_ * 7].v_j_2_) + mr3inv * (f02_ip_[isrc_].v_i_0_ * f02_ip_[isrc_].v_i_0_ + f02_ip_[isrc_].v_i_1_ * f02_ip_[isrc_].v_i_1_ + f02_ip_[isrc_].v_i_2_ * f02_ip_[isrc_].v_i_2_ - 2.0 * (dvbc_0_ * dvbc_0_ + dvbc_1_ * dvbc_1_ + dvbc_2_ * dvbc_2_) + 1.5 * (f02_jp_smem_[j_ + jstride_ * 7].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 7].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 7].v_j_2_ * dxbc_2_) * (f02_jp_smem_[j_ + jstride_ * 7].v_j_0_ * dxbc_0_ + f02_jp_smem_[j_ + jstride_ * 7].v_j_1_ * dxbc_1_ + f02_jp_smem_[j_ + jstride_ * 7].v_j_2_ * dxbc_2_) / rbc2e) * dxbc_2_ + mr3inv * (dxbc_0_ * (4.0 * f02_ip_[isrc_].v_i_0_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 7].v_j_0_) + dxbc_1_ * (4.0 * f02_ip_[isrc_].v_i_1_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 7].v_j_1_) + dxbc_2_ * (4.0 * f02_ip_[isrc_].v_i_2_ - 3.0 * f02_jp_smem_[j_ + jstride_ * 7].v_j_2_)) * dvbc_2_;

                           }
                       }
                   }
                   __syncthreads();
                   f02_result_smem_[tid_].ac_i_0_ = ac_i_0_wcache_;f02_result_smem_[tid_].ac_i_1_ = ac_i_1_wcache_;f02_result_smem_[tid_].ac_i_2_ = ac_i_2_wcache_;
                   __syncthreads();
                   if (jstride_ > 1) {if (tid_ < kbdim_ / 2) {f02_result_smem_[tid_].ac_i_0_ += f02_result_smem_[tid_ + kbdim_ / 2].ac_i_0_;f02_result_smem_[tid_].ac_i_1_ += f02_result_smem_[tid_ + kbdim_ / 2].ac_i_1_;f02_result_smem_[tid_].ac_i_2_ += f02_result_smem_[tid_ + kbdim_ / 2].ac_i_2_;}// __syncthreads(); // this is not necessary since kbdim_ / 2 <= warp size.
}if (jstride_ > 2) {if (tid_ < kbdim_ / 4) {f02_result_smem_[tid_].ac_i_0_ += f02_result_smem_[tid_ + kbdim_ / 4].ac_i_0_;f02_result_smem_[tid_].ac_i_1_ += f02_result_smem_[tid_ + kbdim_ / 4].ac_i_1_;f02_result_smem_[tid_].ac_i_2_ += f02_result_smem_[tid_ + kbdim_ / 4].ac_i_2_;}// __syncthreads(); // this is not necessary since kbdim_ / 4 <= warp size.
}
                   __syncthreads();

#if 1
                   if (tid_ < nvalidthread_) {
                       int idstoff_ = njdiv_ * kbdim_ * ibid_ + jbid_ + njdiv_ * (tid_ % njhsub_);
                       float4 *srcbuf_ = (float4 *) (f02_result_smem_ + tid_);
                       float4 *dstbuf_ = (float4 *) (f02_result_ + idstoff_);
                       for (int icpy_ = 0; icpy_ < sizeof(f02_result_t) / sizeof(float4); icpy_++) {
                           dstbuf_[icpy_] = srcbuf_[icpy_];
                       }
                   }
#else
                   if (tid_ < nvalidthread_) {
                       f02_result_[idst_] = f02_result_smem_[tid_];
                   }
#endif
               }

/*
 * njdiv_    : # of result fragments per result packet.
 * njdiv_ru_ : njdiv_ rounded up to a power of two.
 * rbdim_    : # of result fragments to be reduced to (rbdim_ / njdiv_) result packets.
 */
               __global__ void
               f02_reducer(int njdiv_, int njdiv_ru_, f02_result_t *f02_result_, f02_result_t *f02_result_sub_)
               {
                   extern __shared__ char smembuf_[];
                   int rbdim_ = blockDim.x;
                   f02_result_t * f02_result_smem_ = (f02_result_t *)smembuf_;
                   f02_result_t * f02_result_smem_packed_ = (f02_result_t *)(smembuf_ + rbdim_ * sizeof(f02_result_t));
                   int tid_ = threadIdx.x;
                   int bid_ = blockIdx.x;
                   int isrc_ = rbdim_ * bid_ + tid_;
                   int ndst_ = rbdim_ / njdiv_;
                   int idst_ = ndst_ * bid_ + tid_;
                   __syncthreads();
                   #if 1
                     {
                         float4 *srcbuf_ = (float4 *)(f02_result_sub_ + rbdim_ * bid_);
                         float4 *dstbuf_ = (float4 *)smembuf_;
                         for (int icpy = 0; icpy < sizeof(f02_result_t)/sizeof(float4); icpy++) {
                             dstbuf_[tid_] = srcbuf_[tid_];
                             dstbuf_ += rbdim_;
                             srcbuf_ += rbdim_;
                         }
                     }
#else
                     f02_result_smem_[tid_] = f02_result_sub_[isrc_];
#endif

                   __syncthreads();

                   int n_ = njdiv_ru_;
                   while (n_ > 1) {
                       n_ /= 2;
                       int ipartner_ = tid_ + n_;
                       if (tid_ % njdiv_ < n_ && ipartner_ % njdiv_ru_ < njdiv_) {
                           f02_result_smem_[tid_].ac_i_0_ += f02_result_smem_[ipartner_].ac_i_0_;f02_result_smem_[tid_].ac_i_1_ += f02_result_smem_[ipartner_].ac_i_1_;f02_result_smem_[tid_].ac_i_2_ += f02_result_smem_[ipartner_].ac_i_2_;
                       }
                       __syncthreads(); // this is not necessary if rbdim_ <= warp size.
                   }
                   __syncthreads();
                   if (tid_ % njdiv_ == 0) {
                       int ipack_ = tid_ / njdiv_;
                       f02_result_smem_packed_[ipack_] = f02_result_smem_[tid_];
                   }
                   __syncthreads();
#if 1
                   {
                       float4 *srcbuf_ = (float4 *) f02_result_smem_packed_;
                       float4 *dstbuf_ = (float4 *) (f02_result_ + ndst_ * bid_);
                       if (tid_ < ndst_) {
                           for (int icpy = 0; icpy < sizeof(f02_result_t) / sizeof(float4); icpy++) {
                               dstbuf_[tid_] = srcbuf_[tid_];
                               dstbuf_ += ndst_;
                               srcbuf_ += ndst_;
                           }
                       }
                   }
#else
                   if (tid_ < ndst_) {
                       f02_result_[idst_] = f02_result_smem_packed_[tid_];
                   }
#endif
               }
