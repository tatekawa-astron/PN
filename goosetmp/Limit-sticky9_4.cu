#include "hip/hip_runtime.h"
               __global__ void
               f04_calculator(int ioff_, int ni_, int nj_, double x_0_0_, double x_0_1_, double x_0_2_, double eps2, double m_0_, f04_jp_t *f04_jp_, f04_ip_t *f04_ip_, f04_result_t *f04_result_)
               {
                   extern __shared__ char smembuf_[];
                   int kbdim_ = blockDim.x;
                   
                   f04_jp_t * f04_jp_smem_ = (f04_jp_t *)smembuf_;
                   
                   f04_result_t * f04_result_smem_ = (f04_result_t *)smembuf_;
                   double dxa_0_, dxb_0_, dxab_0_, dxa_1_, dxb_1_, dxab_1_, dxa_2_, dxb_2_, dxab_2_, r1a2e, r1ainv, r1b2e, rab2e, r1binv, rabinv, vi2;
                   double pot_pn2_i_wcache_ = 0.0f;

                   int njdiv_ = gridDim.x;
                   int jbid_ = blockIdx.x;
                   int ibid_ = blockIdx.y;
                   int tid_ = threadIdx.x;
                   int npipe_ = (ni_ - 1) / 1 + 1;
                   int nvalidthread_ = npipe_ - kbdim_ * ibid_;
                   if (nvalidthread_ > kbdim_) {
                       nvalidthread_ = kbdim_;
                   }
                   int njsub_ = (nj_ - 1) / njdiv_ + 1;
                   int joff0_ = njsub_ * jbid_;
                   int joff1_ = joff0_ + njsub_;
                   if (joff1_ > nj_) {
                       joff1_ = nj_;
                   }
                   int jstride_ = 1;
                   if (nvalidthread_ <= kbdim_ / 2) jstride_ = 2;
if (nvalidthread_ <= kbdim_ / 4) jstride_ = 4;

                   int njhsub_ = kbdim_ / jstride_;
                   int jstart_ = tid_ / njhsub_;
                   int isrc_ = kbdim_ * ibid_ + tid_ % njhsub_;
                   int idst_ = njdiv_ * isrc_ + jbid_;
                   int icnt_ = ioff_ + isrc_;
                   for (int joff_ = joff0_; joff_ < joff1_; joff_ += kbdim_) {
                       int jsrc_ = tid_+joff_;
                       __syncthreads();
                       #if 1
                     {
                         float4 *srcbuf_ = (float4 *)(f04_jp_ + joff_);
                         float4 *dstbuf_ = (float4 *)smembuf_;
                         for (int icpy = 0; icpy < sizeof(f04_jp_t)/sizeof(float4); icpy++) {
                             dstbuf_[tid_] = srcbuf_[tid_];
                             dstbuf_ += kbdim_;
                             srcbuf_ += kbdim_;
                         }
                     }
#else
                     f04_jp_smem_[tid_] = f04_jp_[jsrc_];
#endif

                       __syncthreads();
                       int jsup_ = kbdim_;
                       if (joff_ + jsup_ > joff1_) {
                           jsup_ = joff1_ - joff_;
                       }
                       if (jsup_ < kbdim_) {
                           for (int j_ = jstart_; j_ < jsup_; j_+= jstride_) {
                               dxa_0_ = f04_ip_[isrc_].x_i_0_ - x_0_0_;
dxb_0_ = f04_jp_smem_[j_].x_j_0_ - x_0_0_;
dxab_0_ = f04_jp_smem_[j_].x_j_0_ - f04_ip_[isrc_].x_i_0_;
dxa_1_ = f04_ip_[isrc_].x_i_1_ - x_0_1_;
dxb_1_ = f04_jp_smem_[j_].x_j_1_ - x_0_1_;
dxab_1_ = f04_jp_smem_[j_].x_j_1_ - f04_ip_[isrc_].x_i_1_;
dxa_2_ = f04_ip_[isrc_].x_i_2_ - x_0_2_;
dxb_2_ = f04_jp_smem_[j_].x_j_2_ - x_0_2_;
dxab_2_ = f04_jp_smem_[j_].x_j_2_ - f04_ip_[isrc_].x_i_2_;
r1a2e = dxa_0_ * dxa_0_ + dxa_1_ * dxa_1_ + dxa_2_ * dxa_2_ + eps2;
r1ainv = rsqrt(r1a2e);
r1b2e = dxb_0_ * dxb_0_ + dxb_1_ * dxb_1_ + dxb_2_ * dxb_2_ + eps2;
rab2e = dxab_0_ * dxab_0_ + dxab_1_ * dxab_1_ + dxab_2_ * dxab_2_ + eps2;
r1binv = rsqrt(r1b2e);
rabinv = rsqrt(rab2e);
vi2 = f04_ip_[isrc_].v_i_0_ * f04_ip_[isrc_].v_i_0_ + f04_ip_[isrc_].v_i_1_ * f04_ip_[isrc_].v_i_1_ + f04_ip_[isrc_].v_i_2_ * f04_ip_[isrc_].v_i_2_;
pot_pn2_i_wcache_ += 0.25 * rabinv * f04_ip_[isrc_].m_i_ * f04_jp_smem_[j_].m_j_ * (6.0 * vi2 - 7.0 * (f04_ip_[isrc_].v_i_0_ * f04_jp_smem_[j_].v_j_0_ + f04_ip_[isrc_].v_i_1_ * f04_jp_smem_[j_].v_j_1_ + f04_ip_[isrc_].v_i_2_ * f04_jp_smem_[j_].v_j_2_) - (dxab_0_ * f04_ip_[isrc_].v_i_0_ + dxab_1_ * f04_ip_[isrc_].v_i_1_ + dxab_2_ * f04_ip_[isrc_].v_i_2_) * (dxab_0_ * f04_jp_smem_[j_].v_j_0_ + dxab_1_ * f04_jp_smem_[j_].v_j_1_ + dxab_2_ * f04_jp_smem_[j_].v_j_2_) / rab2e) + m_0_ * f04_ip_[isrc_].m_i_ * f04_jp_smem_[j_].m_j_ * r1ainv * rabinv + 0.5 * m_0_ * f04_ip_[isrc_].m_i_ * f04_jp_smem_[j_].m_j_ * r1ainv * r1binv;

                           }
                       }
                       else {    
                           for (int j_ = jstart_; j_ < kbdim_; j_+= jstride_ * 8) {
                                       // loop 0
dxa_0_ = f04_ip_[isrc_].x_i_0_ - x_0_0_;
dxb_0_ = f04_jp_smem_[j_ + jstride_ * 0].x_j_0_ - x_0_0_;
dxab_0_ = f04_jp_smem_[j_ + jstride_ * 0].x_j_0_ - f04_ip_[isrc_].x_i_0_;
dxa_1_ = f04_ip_[isrc_].x_i_1_ - x_0_1_;
dxb_1_ = f04_jp_smem_[j_ + jstride_ * 0].x_j_1_ - x_0_1_;
dxab_1_ = f04_jp_smem_[j_ + jstride_ * 0].x_j_1_ - f04_ip_[isrc_].x_i_1_;
dxa_2_ = f04_ip_[isrc_].x_i_2_ - x_0_2_;
dxb_2_ = f04_jp_smem_[j_ + jstride_ * 0].x_j_2_ - x_0_2_;
dxab_2_ = f04_jp_smem_[j_ + jstride_ * 0].x_j_2_ - f04_ip_[isrc_].x_i_2_;
r1a2e = dxa_0_ * dxa_0_ + dxa_1_ * dxa_1_ + dxa_2_ * dxa_2_ + eps2;
r1ainv = rsqrt(r1a2e);
r1b2e = dxb_0_ * dxb_0_ + dxb_1_ * dxb_1_ + dxb_2_ * dxb_2_ + eps2;
rab2e = dxab_0_ * dxab_0_ + dxab_1_ * dxab_1_ + dxab_2_ * dxab_2_ + eps2;
r1binv = rsqrt(r1b2e);
rabinv = rsqrt(rab2e);
vi2 = f04_ip_[isrc_].v_i_0_ * f04_ip_[isrc_].v_i_0_ + f04_ip_[isrc_].v_i_1_ * f04_ip_[isrc_].v_i_1_ + f04_ip_[isrc_].v_i_2_ * f04_ip_[isrc_].v_i_2_;
pot_pn2_i_wcache_ += 0.25 * rabinv * f04_ip_[isrc_].m_i_ * f04_jp_smem_[j_ + jstride_ * 0].m_j_ * (6.0 * vi2 - 7.0 * (f04_ip_[isrc_].v_i_0_ * f04_jp_smem_[j_ + jstride_ * 0].v_j_0_ + f04_ip_[isrc_].v_i_1_ * f04_jp_smem_[j_ + jstride_ * 0].v_j_1_ + f04_ip_[isrc_].v_i_2_ * f04_jp_smem_[j_ + jstride_ * 0].v_j_2_) - (dxab_0_ * f04_ip_[isrc_].v_i_0_ + dxab_1_ * f04_ip_[isrc_].v_i_1_ + dxab_2_ * f04_ip_[isrc_].v_i_2_) * (dxab_0_ * f04_jp_smem_[j_ + jstride_ * 0].v_j_0_ + dxab_1_ * f04_jp_smem_[j_ + jstride_ * 0].v_j_1_ + dxab_2_ * f04_jp_smem_[j_ + jstride_ * 0].v_j_2_) / rab2e) + m_0_ * f04_ip_[isrc_].m_i_ * f04_jp_smem_[j_ + jstride_ * 0].m_j_ * r1ainv * rabinv + 0.5 * m_0_ * f04_ip_[isrc_].m_i_ * f04_jp_smem_[j_ + jstride_ * 0].m_j_ * r1ainv * r1binv;
        // loop 1
dxa_0_ = f04_ip_[isrc_].x_i_0_ - x_0_0_;
dxb_0_ = f04_jp_smem_[j_ + jstride_ * 1].x_j_0_ - x_0_0_;
dxab_0_ = f04_jp_smem_[j_ + jstride_ * 1].x_j_0_ - f04_ip_[isrc_].x_i_0_;
dxa_1_ = f04_ip_[isrc_].x_i_1_ - x_0_1_;
dxb_1_ = f04_jp_smem_[j_ + jstride_ * 1].x_j_1_ - x_0_1_;
dxab_1_ = f04_jp_smem_[j_ + jstride_ * 1].x_j_1_ - f04_ip_[isrc_].x_i_1_;
dxa_2_ = f04_ip_[isrc_].x_i_2_ - x_0_2_;
dxb_2_ = f04_jp_smem_[j_ + jstride_ * 1].x_j_2_ - x_0_2_;
dxab_2_ = f04_jp_smem_[j_ + jstride_ * 1].x_j_2_ - f04_ip_[isrc_].x_i_2_;
r1a2e = dxa_0_ * dxa_0_ + dxa_1_ * dxa_1_ + dxa_2_ * dxa_2_ + eps2;
r1ainv = rsqrt(r1a2e);
r1b2e = dxb_0_ * dxb_0_ + dxb_1_ * dxb_1_ + dxb_2_ * dxb_2_ + eps2;
rab2e = dxab_0_ * dxab_0_ + dxab_1_ * dxab_1_ + dxab_2_ * dxab_2_ + eps2;
r1binv = rsqrt(r1b2e);
rabinv = rsqrt(rab2e);
vi2 = f04_ip_[isrc_].v_i_0_ * f04_ip_[isrc_].v_i_0_ + f04_ip_[isrc_].v_i_1_ * f04_ip_[isrc_].v_i_1_ + f04_ip_[isrc_].v_i_2_ * f04_ip_[isrc_].v_i_2_;
pot_pn2_i_wcache_ += 0.25 * rabinv * f04_ip_[isrc_].m_i_ * f04_jp_smem_[j_ + jstride_ * 1].m_j_ * (6.0 * vi2 - 7.0 * (f04_ip_[isrc_].v_i_0_ * f04_jp_smem_[j_ + jstride_ * 1].v_j_0_ + f04_ip_[isrc_].v_i_1_ * f04_jp_smem_[j_ + jstride_ * 1].v_j_1_ + f04_ip_[isrc_].v_i_2_ * f04_jp_smem_[j_ + jstride_ * 1].v_j_2_) - (dxab_0_ * f04_ip_[isrc_].v_i_0_ + dxab_1_ * f04_ip_[isrc_].v_i_1_ + dxab_2_ * f04_ip_[isrc_].v_i_2_) * (dxab_0_ * f04_jp_smem_[j_ + jstride_ * 1].v_j_0_ + dxab_1_ * f04_jp_smem_[j_ + jstride_ * 1].v_j_1_ + dxab_2_ * f04_jp_smem_[j_ + jstride_ * 1].v_j_2_) / rab2e) + m_0_ * f04_ip_[isrc_].m_i_ * f04_jp_smem_[j_ + jstride_ * 1].m_j_ * r1ainv * rabinv + 0.5 * m_0_ * f04_ip_[isrc_].m_i_ * f04_jp_smem_[j_ + jstride_ * 1].m_j_ * r1ainv * r1binv;
        // loop 2
dxa_0_ = f04_ip_[isrc_].x_i_0_ - x_0_0_;
dxb_0_ = f04_jp_smem_[j_ + jstride_ * 2].x_j_0_ - x_0_0_;
dxab_0_ = f04_jp_smem_[j_ + jstride_ * 2].x_j_0_ - f04_ip_[isrc_].x_i_0_;
dxa_1_ = f04_ip_[isrc_].x_i_1_ - x_0_1_;
dxb_1_ = f04_jp_smem_[j_ + jstride_ * 2].x_j_1_ - x_0_1_;
dxab_1_ = f04_jp_smem_[j_ + jstride_ * 2].x_j_1_ - f04_ip_[isrc_].x_i_1_;
dxa_2_ = f04_ip_[isrc_].x_i_2_ - x_0_2_;
dxb_2_ = f04_jp_smem_[j_ + jstride_ * 2].x_j_2_ - x_0_2_;
dxab_2_ = f04_jp_smem_[j_ + jstride_ * 2].x_j_2_ - f04_ip_[isrc_].x_i_2_;
r1a2e = dxa_0_ * dxa_0_ + dxa_1_ * dxa_1_ + dxa_2_ * dxa_2_ + eps2;
r1ainv = rsqrt(r1a2e);
r1b2e = dxb_0_ * dxb_0_ + dxb_1_ * dxb_1_ + dxb_2_ * dxb_2_ + eps2;
rab2e = dxab_0_ * dxab_0_ + dxab_1_ * dxab_1_ + dxab_2_ * dxab_2_ + eps2;
r1binv = rsqrt(r1b2e);
rabinv = rsqrt(rab2e);
vi2 = f04_ip_[isrc_].v_i_0_ * f04_ip_[isrc_].v_i_0_ + f04_ip_[isrc_].v_i_1_ * f04_ip_[isrc_].v_i_1_ + f04_ip_[isrc_].v_i_2_ * f04_ip_[isrc_].v_i_2_;
pot_pn2_i_wcache_ += 0.25 * rabinv * f04_ip_[isrc_].m_i_ * f04_jp_smem_[j_ + jstride_ * 2].m_j_ * (6.0 * vi2 - 7.0 * (f04_ip_[isrc_].v_i_0_ * f04_jp_smem_[j_ + jstride_ * 2].v_j_0_ + f04_ip_[isrc_].v_i_1_ * f04_jp_smem_[j_ + jstride_ * 2].v_j_1_ + f04_ip_[isrc_].v_i_2_ * f04_jp_smem_[j_ + jstride_ * 2].v_j_2_) - (dxab_0_ * f04_ip_[isrc_].v_i_0_ + dxab_1_ * f04_ip_[isrc_].v_i_1_ + dxab_2_ * f04_ip_[isrc_].v_i_2_) * (dxab_0_ * f04_jp_smem_[j_ + jstride_ * 2].v_j_0_ + dxab_1_ * f04_jp_smem_[j_ + jstride_ * 2].v_j_1_ + dxab_2_ * f04_jp_smem_[j_ + jstride_ * 2].v_j_2_) / rab2e) + m_0_ * f04_ip_[isrc_].m_i_ * f04_jp_smem_[j_ + jstride_ * 2].m_j_ * r1ainv * rabinv + 0.5 * m_0_ * f04_ip_[isrc_].m_i_ * f04_jp_smem_[j_ + jstride_ * 2].m_j_ * r1ainv * r1binv;
        // loop 3
dxa_0_ = f04_ip_[isrc_].x_i_0_ - x_0_0_;
dxb_0_ = f04_jp_smem_[j_ + jstride_ * 3].x_j_0_ - x_0_0_;
dxab_0_ = f04_jp_smem_[j_ + jstride_ * 3].x_j_0_ - f04_ip_[isrc_].x_i_0_;
dxa_1_ = f04_ip_[isrc_].x_i_1_ - x_0_1_;
dxb_1_ = f04_jp_smem_[j_ + jstride_ * 3].x_j_1_ - x_0_1_;
dxab_1_ = f04_jp_smem_[j_ + jstride_ * 3].x_j_1_ - f04_ip_[isrc_].x_i_1_;
dxa_2_ = f04_ip_[isrc_].x_i_2_ - x_0_2_;
dxb_2_ = f04_jp_smem_[j_ + jstride_ * 3].x_j_2_ - x_0_2_;
dxab_2_ = f04_jp_smem_[j_ + jstride_ * 3].x_j_2_ - f04_ip_[isrc_].x_i_2_;
r1a2e = dxa_0_ * dxa_0_ + dxa_1_ * dxa_1_ + dxa_2_ * dxa_2_ + eps2;
r1ainv = rsqrt(r1a2e);
r1b2e = dxb_0_ * dxb_0_ + dxb_1_ * dxb_1_ + dxb_2_ * dxb_2_ + eps2;
rab2e = dxab_0_ * dxab_0_ + dxab_1_ * dxab_1_ + dxab_2_ * dxab_2_ + eps2;
r1binv = rsqrt(r1b2e);
rabinv = rsqrt(rab2e);
vi2 = f04_ip_[isrc_].v_i_0_ * f04_ip_[isrc_].v_i_0_ + f04_ip_[isrc_].v_i_1_ * f04_ip_[isrc_].v_i_1_ + f04_ip_[isrc_].v_i_2_ * f04_ip_[isrc_].v_i_2_;
pot_pn2_i_wcache_ += 0.25 * rabinv * f04_ip_[isrc_].m_i_ * f04_jp_smem_[j_ + jstride_ * 3].m_j_ * (6.0 * vi2 - 7.0 * (f04_ip_[isrc_].v_i_0_ * f04_jp_smem_[j_ + jstride_ * 3].v_j_0_ + f04_ip_[isrc_].v_i_1_ * f04_jp_smem_[j_ + jstride_ * 3].v_j_1_ + f04_ip_[isrc_].v_i_2_ * f04_jp_smem_[j_ + jstride_ * 3].v_j_2_) - (dxab_0_ * f04_ip_[isrc_].v_i_0_ + dxab_1_ * f04_ip_[isrc_].v_i_1_ + dxab_2_ * f04_ip_[isrc_].v_i_2_) * (dxab_0_ * f04_jp_smem_[j_ + jstride_ * 3].v_j_0_ + dxab_1_ * f04_jp_smem_[j_ + jstride_ * 3].v_j_1_ + dxab_2_ * f04_jp_smem_[j_ + jstride_ * 3].v_j_2_) / rab2e) + m_0_ * f04_ip_[isrc_].m_i_ * f04_jp_smem_[j_ + jstride_ * 3].m_j_ * r1ainv * rabinv + 0.5 * m_0_ * f04_ip_[isrc_].m_i_ * f04_jp_smem_[j_ + jstride_ * 3].m_j_ * r1ainv * r1binv;
        // loop 4
dxa_0_ = f04_ip_[isrc_].x_i_0_ - x_0_0_;
dxb_0_ = f04_jp_smem_[j_ + jstride_ * 4].x_j_0_ - x_0_0_;
dxab_0_ = f04_jp_smem_[j_ + jstride_ * 4].x_j_0_ - f04_ip_[isrc_].x_i_0_;
dxa_1_ = f04_ip_[isrc_].x_i_1_ - x_0_1_;
dxb_1_ = f04_jp_smem_[j_ + jstride_ * 4].x_j_1_ - x_0_1_;
dxab_1_ = f04_jp_smem_[j_ + jstride_ * 4].x_j_1_ - f04_ip_[isrc_].x_i_1_;
dxa_2_ = f04_ip_[isrc_].x_i_2_ - x_0_2_;
dxb_2_ = f04_jp_smem_[j_ + jstride_ * 4].x_j_2_ - x_0_2_;
dxab_2_ = f04_jp_smem_[j_ + jstride_ * 4].x_j_2_ - f04_ip_[isrc_].x_i_2_;
r1a2e = dxa_0_ * dxa_0_ + dxa_1_ * dxa_1_ + dxa_2_ * dxa_2_ + eps2;
r1ainv = rsqrt(r1a2e);
r1b2e = dxb_0_ * dxb_0_ + dxb_1_ * dxb_1_ + dxb_2_ * dxb_2_ + eps2;
rab2e = dxab_0_ * dxab_0_ + dxab_1_ * dxab_1_ + dxab_2_ * dxab_2_ + eps2;
r1binv = rsqrt(r1b2e);
rabinv = rsqrt(rab2e);
vi2 = f04_ip_[isrc_].v_i_0_ * f04_ip_[isrc_].v_i_0_ + f04_ip_[isrc_].v_i_1_ * f04_ip_[isrc_].v_i_1_ + f04_ip_[isrc_].v_i_2_ * f04_ip_[isrc_].v_i_2_;
pot_pn2_i_wcache_ += 0.25 * rabinv * f04_ip_[isrc_].m_i_ * f04_jp_smem_[j_ + jstride_ * 4].m_j_ * (6.0 * vi2 - 7.0 * (f04_ip_[isrc_].v_i_0_ * f04_jp_smem_[j_ + jstride_ * 4].v_j_0_ + f04_ip_[isrc_].v_i_1_ * f04_jp_smem_[j_ + jstride_ * 4].v_j_1_ + f04_ip_[isrc_].v_i_2_ * f04_jp_smem_[j_ + jstride_ * 4].v_j_2_) - (dxab_0_ * f04_ip_[isrc_].v_i_0_ + dxab_1_ * f04_ip_[isrc_].v_i_1_ + dxab_2_ * f04_ip_[isrc_].v_i_2_) * (dxab_0_ * f04_jp_smem_[j_ + jstride_ * 4].v_j_0_ + dxab_1_ * f04_jp_smem_[j_ + jstride_ * 4].v_j_1_ + dxab_2_ * f04_jp_smem_[j_ + jstride_ * 4].v_j_2_) / rab2e) + m_0_ * f04_ip_[isrc_].m_i_ * f04_jp_smem_[j_ + jstride_ * 4].m_j_ * r1ainv * rabinv + 0.5 * m_0_ * f04_ip_[isrc_].m_i_ * f04_jp_smem_[j_ + jstride_ * 4].m_j_ * r1ainv * r1binv;
        // loop 5
dxa_0_ = f04_ip_[isrc_].x_i_0_ - x_0_0_;
dxb_0_ = f04_jp_smem_[j_ + jstride_ * 5].x_j_0_ - x_0_0_;
dxab_0_ = f04_jp_smem_[j_ + jstride_ * 5].x_j_0_ - f04_ip_[isrc_].x_i_0_;
dxa_1_ = f04_ip_[isrc_].x_i_1_ - x_0_1_;
dxb_1_ = f04_jp_smem_[j_ + jstride_ * 5].x_j_1_ - x_0_1_;
dxab_1_ = f04_jp_smem_[j_ + jstride_ * 5].x_j_1_ - f04_ip_[isrc_].x_i_1_;
dxa_2_ = f04_ip_[isrc_].x_i_2_ - x_0_2_;
dxb_2_ = f04_jp_smem_[j_ + jstride_ * 5].x_j_2_ - x_0_2_;
dxab_2_ = f04_jp_smem_[j_ + jstride_ * 5].x_j_2_ - f04_ip_[isrc_].x_i_2_;
r1a2e = dxa_0_ * dxa_0_ + dxa_1_ * dxa_1_ + dxa_2_ * dxa_2_ + eps2;
r1ainv = rsqrt(r1a2e);
r1b2e = dxb_0_ * dxb_0_ + dxb_1_ * dxb_1_ + dxb_2_ * dxb_2_ + eps2;
rab2e = dxab_0_ * dxab_0_ + dxab_1_ * dxab_1_ + dxab_2_ * dxab_2_ + eps2;
r1binv = rsqrt(r1b2e);
rabinv = rsqrt(rab2e);
vi2 = f04_ip_[isrc_].v_i_0_ * f04_ip_[isrc_].v_i_0_ + f04_ip_[isrc_].v_i_1_ * f04_ip_[isrc_].v_i_1_ + f04_ip_[isrc_].v_i_2_ * f04_ip_[isrc_].v_i_2_;
pot_pn2_i_wcache_ += 0.25 * rabinv * f04_ip_[isrc_].m_i_ * f04_jp_smem_[j_ + jstride_ * 5].m_j_ * (6.0 * vi2 - 7.0 * (f04_ip_[isrc_].v_i_0_ * f04_jp_smem_[j_ + jstride_ * 5].v_j_0_ + f04_ip_[isrc_].v_i_1_ * f04_jp_smem_[j_ + jstride_ * 5].v_j_1_ + f04_ip_[isrc_].v_i_2_ * f04_jp_smem_[j_ + jstride_ * 5].v_j_2_) - (dxab_0_ * f04_ip_[isrc_].v_i_0_ + dxab_1_ * f04_ip_[isrc_].v_i_1_ + dxab_2_ * f04_ip_[isrc_].v_i_2_) * (dxab_0_ * f04_jp_smem_[j_ + jstride_ * 5].v_j_0_ + dxab_1_ * f04_jp_smem_[j_ + jstride_ * 5].v_j_1_ + dxab_2_ * f04_jp_smem_[j_ + jstride_ * 5].v_j_2_) / rab2e) + m_0_ * f04_ip_[isrc_].m_i_ * f04_jp_smem_[j_ + jstride_ * 5].m_j_ * r1ainv * rabinv + 0.5 * m_0_ * f04_ip_[isrc_].m_i_ * f04_jp_smem_[j_ + jstride_ * 5].m_j_ * r1ainv * r1binv;
        // loop 6
dxa_0_ = f04_ip_[isrc_].x_i_0_ - x_0_0_;
dxb_0_ = f04_jp_smem_[j_ + jstride_ * 6].x_j_0_ - x_0_0_;
dxab_0_ = f04_jp_smem_[j_ + jstride_ * 6].x_j_0_ - f04_ip_[isrc_].x_i_0_;
dxa_1_ = f04_ip_[isrc_].x_i_1_ - x_0_1_;
dxb_1_ = f04_jp_smem_[j_ + jstride_ * 6].x_j_1_ - x_0_1_;
dxab_1_ = f04_jp_smem_[j_ + jstride_ * 6].x_j_1_ - f04_ip_[isrc_].x_i_1_;
dxa_2_ = f04_ip_[isrc_].x_i_2_ - x_0_2_;
dxb_2_ = f04_jp_smem_[j_ + jstride_ * 6].x_j_2_ - x_0_2_;
dxab_2_ = f04_jp_smem_[j_ + jstride_ * 6].x_j_2_ - f04_ip_[isrc_].x_i_2_;
r1a2e = dxa_0_ * dxa_0_ + dxa_1_ * dxa_1_ + dxa_2_ * dxa_2_ + eps2;
r1ainv = rsqrt(r1a2e);
r1b2e = dxb_0_ * dxb_0_ + dxb_1_ * dxb_1_ + dxb_2_ * dxb_2_ + eps2;
rab2e = dxab_0_ * dxab_0_ + dxab_1_ * dxab_1_ + dxab_2_ * dxab_2_ + eps2;
r1binv = rsqrt(r1b2e);
rabinv = rsqrt(rab2e);
vi2 = f04_ip_[isrc_].v_i_0_ * f04_ip_[isrc_].v_i_0_ + f04_ip_[isrc_].v_i_1_ * f04_ip_[isrc_].v_i_1_ + f04_ip_[isrc_].v_i_2_ * f04_ip_[isrc_].v_i_2_;
pot_pn2_i_wcache_ += 0.25 * rabinv * f04_ip_[isrc_].m_i_ * f04_jp_smem_[j_ + jstride_ * 6].m_j_ * (6.0 * vi2 - 7.0 * (f04_ip_[isrc_].v_i_0_ * f04_jp_smem_[j_ + jstride_ * 6].v_j_0_ + f04_ip_[isrc_].v_i_1_ * f04_jp_smem_[j_ + jstride_ * 6].v_j_1_ + f04_ip_[isrc_].v_i_2_ * f04_jp_smem_[j_ + jstride_ * 6].v_j_2_) - (dxab_0_ * f04_ip_[isrc_].v_i_0_ + dxab_1_ * f04_ip_[isrc_].v_i_1_ + dxab_2_ * f04_ip_[isrc_].v_i_2_) * (dxab_0_ * f04_jp_smem_[j_ + jstride_ * 6].v_j_0_ + dxab_1_ * f04_jp_smem_[j_ + jstride_ * 6].v_j_1_ + dxab_2_ * f04_jp_smem_[j_ + jstride_ * 6].v_j_2_) / rab2e) + m_0_ * f04_ip_[isrc_].m_i_ * f04_jp_smem_[j_ + jstride_ * 6].m_j_ * r1ainv * rabinv + 0.5 * m_0_ * f04_ip_[isrc_].m_i_ * f04_jp_smem_[j_ + jstride_ * 6].m_j_ * r1ainv * r1binv;
        // loop 7
dxa_0_ = f04_ip_[isrc_].x_i_0_ - x_0_0_;
dxb_0_ = f04_jp_smem_[j_ + jstride_ * 7].x_j_0_ - x_0_0_;
dxab_0_ = f04_jp_smem_[j_ + jstride_ * 7].x_j_0_ - f04_ip_[isrc_].x_i_0_;
dxa_1_ = f04_ip_[isrc_].x_i_1_ - x_0_1_;
dxb_1_ = f04_jp_smem_[j_ + jstride_ * 7].x_j_1_ - x_0_1_;
dxab_1_ = f04_jp_smem_[j_ + jstride_ * 7].x_j_1_ - f04_ip_[isrc_].x_i_1_;
dxa_2_ = f04_ip_[isrc_].x_i_2_ - x_0_2_;
dxb_2_ = f04_jp_smem_[j_ + jstride_ * 7].x_j_2_ - x_0_2_;
dxab_2_ = f04_jp_smem_[j_ + jstride_ * 7].x_j_2_ - f04_ip_[isrc_].x_i_2_;
r1a2e = dxa_0_ * dxa_0_ + dxa_1_ * dxa_1_ + dxa_2_ * dxa_2_ + eps2;
r1ainv = rsqrt(r1a2e);
r1b2e = dxb_0_ * dxb_0_ + dxb_1_ * dxb_1_ + dxb_2_ * dxb_2_ + eps2;
rab2e = dxab_0_ * dxab_0_ + dxab_1_ * dxab_1_ + dxab_2_ * dxab_2_ + eps2;
r1binv = rsqrt(r1b2e);
rabinv = rsqrt(rab2e);
vi2 = f04_ip_[isrc_].v_i_0_ * f04_ip_[isrc_].v_i_0_ + f04_ip_[isrc_].v_i_1_ * f04_ip_[isrc_].v_i_1_ + f04_ip_[isrc_].v_i_2_ * f04_ip_[isrc_].v_i_2_;
pot_pn2_i_wcache_ += 0.25 * rabinv * f04_ip_[isrc_].m_i_ * f04_jp_smem_[j_ + jstride_ * 7].m_j_ * (6.0 * vi2 - 7.0 * (f04_ip_[isrc_].v_i_0_ * f04_jp_smem_[j_ + jstride_ * 7].v_j_0_ + f04_ip_[isrc_].v_i_1_ * f04_jp_smem_[j_ + jstride_ * 7].v_j_1_ + f04_ip_[isrc_].v_i_2_ * f04_jp_smem_[j_ + jstride_ * 7].v_j_2_) - (dxab_0_ * f04_ip_[isrc_].v_i_0_ + dxab_1_ * f04_ip_[isrc_].v_i_1_ + dxab_2_ * f04_ip_[isrc_].v_i_2_) * (dxab_0_ * f04_jp_smem_[j_ + jstride_ * 7].v_j_0_ + dxab_1_ * f04_jp_smem_[j_ + jstride_ * 7].v_j_1_ + dxab_2_ * f04_jp_smem_[j_ + jstride_ * 7].v_j_2_) / rab2e) + m_0_ * f04_ip_[isrc_].m_i_ * f04_jp_smem_[j_ + jstride_ * 7].m_j_ * r1ainv * rabinv + 0.5 * m_0_ * f04_ip_[isrc_].m_i_ * f04_jp_smem_[j_ + jstride_ * 7].m_j_ * r1ainv * r1binv;

                           }
                       }
                   }
                   __syncthreads();
                   f04_result_smem_[tid_].pot_pn2_i_ = pot_pn2_i_wcache_;
                   __syncthreads();
                   if (jstride_ > 1) {if (tid_ < kbdim_ / 2) {f04_result_smem_[tid_].pot_pn2_i_ += f04_result_smem_[tid_ + kbdim_ / 2].pot_pn2_i_;}// __syncthreads(); // this is not necessary since kbdim_ / 2 <= warp size.
}if (jstride_ > 2) {if (tid_ < kbdim_ / 4) {f04_result_smem_[tid_].pot_pn2_i_ += f04_result_smem_[tid_ + kbdim_ / 4].pot_pn2_i_;}// __syncthreads(); // this is not necessary since kbdim_ / 4 <= warp size.
}
                   __syncthreads();

#if 1
                   if (tid_ < nvalidthread_) {
                       int idstoff_ = njdiv_ * kbdim_ * ibid_ + jbid_ + njdiv_ * (tid_ % njhsub_);
                       float4 *srcbuf_ = (float4 *) (f04_result_smem_ + tid_);
                       float4 *dstbuf_ = (float4 *) (f04_result_ + idstoff_);
                       for (int icpy_ = 0; icpy_ < sizeof(f04_result_t) / sizeof(float4); icpy_++) {
                           dstbuf_[icpy_] = srcbuf_[icpy_];
                       }
                   }
#else
                   if (tid_ < nvalidthread_) {
                       f04_result_[idst_] = f04_result_smem_[tid_];
                   }
#endif
               }

/*
 * njdiv_    : # of result fragments per result packet.
 * njdiv_ru_ : njdiv_ rounded up to a power of two.
 * rbdim_    : # of result fragments to be reduced to (rbdim_ / njdiv_) result packets.
 */
               __global__ void
               f04_reducer(int njdiv_, int njdiv_ru_, f04_result_t *f04_result_, f04_result_t *f04_result_sub_)
               {
                   extern __shared__ char smembuf_[];
                   int rbdim_ = blockDim.x;
                   f04_result_t * f04_result_smem_ = (f04_result_t *)smembuf_;
                   f04_result_t * f04_result_smem_packed_ = (f04_result_t *)(smembuf_ + rbdim_ * sizeof(f04_result_t));
                   int tid_ = threadIdx.x;
                   int bid_ = blockIdx.x;
                   int isrc_ = rbdim_ * bid_ + tid_;
                   int ndst_ = rbdim_ / njdiv_;
                   int idst_ = ndst_ * bid_ + tid_;
                   __syncthreads();
                   #if 1
                     {
                         float4 *srcbuf_ = (float4 *)(f04_result_sub_ + rbdim_ * bid_);
                         float4 *dstbuf_ = (float4 *)smembuf_;
                         for (int icpy = 0; icpy < sizeof(f04_result_t)/sizeof(float4); icpy++) {
                             dstbuf_[tid_] = srcbuf_[tid_];
                             dstbuf_ += rbdim_;
                             srcbuf_ += rbdim_;
                         }
                     }
#else
                     f04_result_smem_[tid_] = f04_result_sub_[isrc_];
#endif

                   __syncthreads();

                   int n_ = njdiv_ru_;
                   while (n_ > 1) {
                       n_ /= 2;
                       int ipartner_ = tid_ + n_;
                       if (tid_ % njdiv_ < n_ && ipartner_ % njdiv_ru_ < njdiv_) {
                           f04_result_smem_[tid_].pot_pn2_i_ += f04_result_smem_[ipartner_].pot_pn2_i_;
                       }
                       __syncthreads(); // this is not necessary if rbdim_ <= warp size.
                   }
                   __syncthreads();
                   if (tid_ % njdiv_ == 0) {
                       int ipack_ = tid_ / njdiv_;
                       f04_result_smem_packed_[ipack_] = f04_result_smem_[tid_];
                   }
                   __syncthreads();
#if 1
                   {
                       float4 *srcbuf_ = (float4 *) f04_result_smem_packed_;
                       float4 *dstbuf_ = (float4 *) (f04_result_ + ndst_ * bid_);
                       if (tid_ < ndst_) {
                           for (int icpy = 0; icpy < sizeof(f04_result_t) / sizeof(float4); icpy++) {
                               dstbuf_[tid_] = srcbuf_[tid_];
                               dstbuf_ += ndst_;
                               srcbuf_ += ndst_;
                           }
                       }
                   }
#else
                   if (tid_ < ndst_) {
                       f04_result_[idst_] = f04_result_smem_packed_[tid_];
                   }
#endif
               }
