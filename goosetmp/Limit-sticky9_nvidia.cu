#include "hip/hip_runtime.h"
#include <cutil.h>
#include <cutil_inline.h>
#include <gcutil.h>
#include "Limit-sticky9_0.h"
#include "Limit-sticky9_1.h"
#include "Limit-sticky9_2.h"
#include "Limit-sticky9_3.h"
#include "Limit-sticky9_4.h"
#include "Limit-sticky9_5.h"
#include "Limit-sticky9_6.h"




#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <sys/resource.h>
#include <unistd.h>
#define DIM 3


#define REAL double


#define NMAX 270000





void
force(double (*x)[3], double (*v)[3], double *m, double eps,
     double (*a)[3], double *pot, int n)
{
  double r, r2e, r2, reinv, rinv, mrinv, mr3inv, dx[3], a1[3], a1c[3];
  double ac[270000][3];
  double dxb[3], dxc[3], dxbc[3], dvbc[3];
  double r1b2, r1b2e, r1c2, r1c2e, rbc2, rbc2e, r1be, mr1b3e, r1ce, rbce;
  double eps2;
  int i, j, k;

  eps2=eps*eps;



for (i=0;i<n;i++) {
    
for (k=0;k<3;k++) { a[i][k] = 0.0;}

    }
                      /*
                       * dispatcher for a kernel defined in 'Limit-sticky9_0.cu'.
                       *
                       * kbdim_  : # of threads per block. fixed to a multiple of warp size.
                       * nip_    : # of IPs handled in one block.
                       * njdiv_  : # of JP fragments. determined in a heuristic manner as a function of
                       *           ni, nj and device specification such as max grid size and warp size,
                       * nblock_ : # of blocks (ni * njdiv / nip) rounded up to a multiple of nip.
                       * npipe_  : len of each IP array. ni/nip_pack rounded up to a multiple of (nth * nip_pack).
                       * nj_ru_  : nj rounded up to a multiple of (nth * njdiv).
                       */
                      {
#if SHARED_HOSTBUF
                          double * GlobalMem<double>::hostbuf = NULL;
                          int GlobalMem<double>::nbytemax = 0;
#endif
                          // Below may have a room for optimization yet.
                          const int kbdim_ = 64; 
                          const int rbdim_ = 64; 
                          const int nimax_ = kbdim_ * 128; // necessary amount of main mem is larger for larger value.
                          const size_t ksmemsize_ = kbdim_ * sizeof(f00_jp_t);
                          const size_t rsmemsize_ = rbdim_ * sizeof(f00_result_t) * 2;
                          const int nip_ = kbdim_ * 1;
                          const GoosePrecision gprec_ = kGoosePrecisionDouble;
                          static int njdiv_, njdiv_ru_;
                          static int njold = 0, niold = 0;
                          static int nblockmax_, nsp_;
                          static size_t smemsizemax_;
                          static int firstcall_ = 1;
                          int bufidx_, ioff_, nisub_;
                          int ni_ru_ = ((n - 0 - 1) / nimax_ + 1) * nimax_;
                          hipError_t cuerr_;
                          if (firstcall_) {
                              int device_;
                              hipDeviceProp_t prop;
                              firstcall_ = 0;
                              cuerr_ = hipGetDevice(&device_);
                              cutilSafeCall(cuerr_);
                              cuerr_ = hipGetDeviceProperties(&prop, device_);
                              cutilSafeCall(cuerr_);
                              if (gprec_ >= kGoosePrecisionDouble) {
                                  if (prop.major < 2 && prop.minor < 3) {
                                      fprintf(stderr, "GPU architecture sm_%d%d does not support double-precision arithmetic.\n",
                                              prop.major, prop.minor);
                                      exit(1);
                                  }
                              }
                              smemsizemax_ = prop.sharedMemPerBlock;
                              if (ksmemsize_ >= smemsizemax_) {
                                  fprintf(stderr,
                                          "Shared memory consumption in f00_calculator (=%dbyte) "
                                          "reached the limit (=%dbyte). Too many jvars or fvars used.\n",
                                          ksmemsize_, smemsizemax_);
                                  exit(1);
                              }
                              if (rsmemsize_ >= smemsizemax_) {
                                  fprintf(stderr,
                                          "Shared memory consumption in f00_reducer (=%dbyte) "
                                          "reached the limit (=%dbyte). Too many fvars used.\n",
                                          rsmemsize_, smemsizemax_);
                                  exit(1);
                              }
                              nblockmax_ = prop.maxGridSize[0];
                              nsp_ = prop.multiProcessorCount * 8; // # of spream processors.
                              // fprintf(stderr, "nblockmax:%d  nsp:%d\n", nblockmax_, nsp_);
                          }
                          nisub_ = nimax_;
                          for (ioff_ = 0; ioff_ < n; ioff_ += nisub_) {
                              if (ioff_ + nisub_ > n) {
                                  nisub_ = n - ioff_;
                              }

                              int nisub_ru_ = ((nisub_ - 1) / nip_ + 1) * nip_;
                              if (njold != n || niold != nisub_) {
                                  // Adjust # of JP fragments so that large enough # of threads (to fill all SPs, 
                                  // and to hide latency of the global memory) are dispatched, at long as each
                                  // JP fragments has several times warp size.
                                  // You may want to hand tune the following part to obtain the optimal 'njdiv_'
                                  // value for a given hardware configuration.
                                  njdiv_ = njdiv_ru_ = 1;
                                  while (nisub_ru_ / nip_ * njdiv_ * 2 <= nblockmax_ &&
                                         nisub_ * njdiv_ < nsp_ * 200 &&
                                         n / njdiv_ > kbdim_ * 6 &&
                                         njdiv_ * 2 <= rbdim_) {
                                      njdiv_ *= 2;
                                  }
                                  // njdiv_ru_ is set to njdiv_ rounded up to a power of two.
                                  while (njdiv_ru_ < njdiv_) {
                                      njdiv_ru_ *= 2;
                                  }
                                  njold = n;
                                  niold = nisub_;
#if 0
                                  fprintf(stderr, "\n");
                                  fprintf(stderr, "nj:%d  njdiv:%d  nj/njdiv:%d  ni:%d  ni * njdiv:%d\n",
                                          n, njdiv_, n/njdiv_, nisub_, nisub_ * njdiv_);
                                  fprintf(stderr, "\n");
#endif
                              }
                              dim3 kthreads_(kbdim_, 1, 1);
                              dim3 kgrids_(njdiv_, nisub_ru_ / nip_, 1);
                              dim3 rthreads_(rbdim_, 1, 1);
                              int npipe_ = nisub_ru_ / 1;
                              int nrblock_ = npipe_ * njdiv_ / rbdim_;
                              dim3 rgrids_(nrblock_,1, 1);
                              int nj_ru_ = ((n - 0 - 1) / (kbdim_ * njdiv_) + 1) * (kbdim_ * njdiv_);
                              static int jbufsize_ = 0, ibufsize_ = 0, rsubbufsize_ = 0, rbufsize_ = 0;
                              static GlobalMem<f00_jp_t> f00_jp_;
                              static GlobalMem<f00_ip_t> f00_ip_;
                              static GlobalMem<f00_result_t> f00_result_;
                              static GlobalMem<f00_result_t> f00_result_sub_;
                              if (nj_ru_ > jbufsize_) {
                                  jbufsize_ = nj_ru_;
                                  f00_jp_.realloc(jbufsize_);
                              }

                              // Here we need to alloc n IPs. Note that npipe_ IPs would not be enough,
                              // since IPs on the device memory should not be overwritten during the ioff_ loop.
                              if (ni_ru_ > ibufsize_) {
                                  ibufsize_ = ni_ru_;
                                  f00_ip_.realloc(ibufsize_);
                              }
                              if (ni_ru_ > rbufsize_) {
                                  rbufsize_ = ni_ru_;
                                  f00_result_.realloc(rbufsize_);
                              }
                              if (npipe_ * njdiv_ > rsubbufsize_) {
                                  rsubbufsize_ = npipe_ * njdiv_ ;
                                  f00_result_sub_.realloc(rsubbufsize_);
                              }

                              for (j = 0, bufidx_ = 0 ; j <n; j++, bufidx_++) {f00_jp_[bufidx_].x_j_0_ = x[j][0];f00_jp_[bufidx_].x_j_1_ = x[j][1];f00_jp_[bufidx_].x_j_2_ = x[j][2];f00_jp_[bufidx_].m_j_ = m[j];}f00_jp_.htod(0, bufidx_);

                              if (ioff_ < n) { // otherwise nothing to send.
                                  int nipsend_ = npipe_;
                                  if (ioff_ + nipsend_ >= n) {
                                      nipsend_ = n - ioff_;
                                  }
                                  for (i = 0 ; i < nipsend_; i++) {f00_ip_[ioff_ + i].x_i_0_ = x[ioff_ + i][0];f00_ip_[ioff_ + i].x_i_1_ = x[ioff_ + i][1];f00_ip_[ioff_ + i].x_i_2_ = x[ioff_ + i][2];}f00_ip_.htod(ioff_, nipsend_);

                              }
                              f00_calculator<<<kgrids_, kthreads_, ksmemsize_>>>(ioff_, nisub_, n - 0, (double)eps2, f00_jp_, (f00_ip_ + ioff_), f00_result_sub_);
                              f00_reducer<<<rgrids_, rthreads_, rsmemsize_>>>(njdiv_, njdiv_ru_, (f00_result_ + ioff_), f00_result_sub_);

                              if (ioff_ < n) { // otherwise nothing to receive.
                                  int niprecv_ = npipe_;
                                  if (ioff_ + niprecv_ >= n) {
                                      niprecv_ = n - ioff_;
                                  }
                                  f00_result_.dtoh(ioff_, niprecv_);
for (i = 0 ; i < niprecv_; i++) {a[ioff_ + i][0] = f00_result_[ioff_ + i].a_i_0_;a[ioff_ + i][1] = f00_result_[ioff_ + i].a_i_1_;a[ioff_ + i][2] = f00_result_[ioff_ + i].a_i_2_;}
                              }
                          } // end of ioff loop.
                      } // end of api calls.



  for(k=0;k<3;k++) a1[k] = 0.0;



  for(i=1;i<n;i++) {
    for(k=0;k<3;k++) {
      dx[k] = x[i][k] - x[0][k];
    }
    r2 = dx[0]*dx[0] + dx[1]*dx[1] + dx[2]*dx[2];
    rinv = rsqrt(r2);
    r2e = r2 + eps2;
    reinv = rsqrt(r2e);
    mrinv = m[i]*reinv;
    mr3inv = mrinv*reinv*reinv;

    a1[0] += mr3inv * dx[0]
      * (5.0*m[0]*reinv
  -2.0*(v[i][0]*v[i][0]+v[i][1]*v[i][1]+v[i][2]*v[i][2])
  +1.5*(v[i][0]*dx[0]+v[i][1]*dx[1]+v[i][2]*dx[2])
  *(v[i][0]*dx[0]+v[i][1]*dx[1]+v[i][2]*dx[2])
  /r2)
      +3.0*mr3inv*v[i][0]
      * (v[i][0]*dx[0]+v[i][1]*dx[1]+v[i][2]*dx[2]);

    a1[1] += mr3inv * dx[1]
      * (5.0*m[0]*reinv
  -2.0*(v[i][0]*v[i][0]+v[i][1]*v[i][1]+v[i][2]*v[i][2])
  +1.5*(v[i][0]*dx[0]+v[i][1]*dx[1]+v[i][2]*dx[2])
  *(v[i][0]*dx[0]+v[i][1]*dx[1]+v[i][2]*dx[2])
  /r2)
      +3.0*mr3inv*v[i][1]
      * (v[i][0]*dx[0]+v[i][1]*dx[1]+v[i][2]*dx[2]);

    a1[2] += mr3inv * dx[2]
      * (5.0*m[0]*reinv
  -2.0*(v[i][0]*v[i][0]+v[i][1]*v[i][1]+v[i][2]*v[i][2])
  +1.5*(v[i][0]*dx[0]+v[i][1]*dx[1]+v[i][2]*dx[2])
  *(v[i][0]*dx[0]+v[i][1]*dx[1]+v[i][2]*dx[2])
  /r2)
      +3.0*mr3inv*v[i][2]
      * (v[i][0]*dx[0]+v[i][1]*dx[1]+v[i][2]*dx[2]);
  }

  for(k=0;k<3;k++) a[0][k]+=a1[k];



  for(k=0;k<3;k++) a1c[k]=0.0;
  for(i=1;i<n;i++) {
    for(k=0;k<3;k++) {
   dxb[k]=x[i][k]-x[0][k];
    }
      r1b2=dxb[0]*dxb[0]+dxb[1]*dxb[1]+dxb[2]*dxb[2];
      r1b2e=r1b2+eps2;
      r1be=rsqrt(r1b2e);
      mr1b3e=m[i]*r1be*r1be*r1be;

    a1c[0]+=4.0*mr1b3e*r1be*m[i]*dxb[0];
    a1c[1]+=4.0*mr1b3e*r1be*m[i]*dxb[1];
    a1c[2]+=4.0*mr1b3e*r1be*m[i]*dxb[2];

  }
                      /*
                       * dispatcher for a kernel defined in 'Limit-sticky9_1.cu'.
                       *
                       * kbdim_  : # of threads per block. fixed to a multiple of warp size.
                       * nip_    : # of IPs handled in one block.
                       * njdiv_  : # of JP fragments. determined in a heuristic manner as a function of
                       *           ni, nj and device specification such as max grid size and warp size,
                       * nblock_ : # of blocks (ni * njdiv / nip) rounded up to a multiple of nip.
                       * npipe_  : len of each IP array. ni/nip_pack rounded up to a multiple of (nth * nip_pack).
                       * nj_ru_  : nj rounded up to a multiple of (nth * njdiv).
                       */
                      {
#if SHARED_HOSTBUF
                          double * GlobalMem<double>::hostbuf = NULL;
                          int GlobalMem<double>::nbytemax = 0;
#endif
                          // Below may have a room for optimization yet.
                          const int kbdim_ = 64; 
                          const int rbdim_ = 64; 
                          const int nimax_ = kbdim_ * 128; // necessary amount of main mem is larger for larger value.
                          const size_t ksmemsize_ = kbdim_ * sizeof(f01_jp_t);
                          const size_t rsmemsize_ = rbdim_ * sizeof(f01_result_t) * 2;
                          const int nip_ = kbdim_ * 1;
                          const GoosePrecision gprec_ = kGoosePrecisionDouble;
                          static int njdiv_, njdiv_ru_;
                          static int njold = 0, niold = 0;
                          static int nblockmax_, nsp_;
                          static size_t smemsizemax_;
                          static int firstcall_ = 1;
                          int bufidx_, ioff_, nisub_;
                          int ni_ru_ = ((n - 1 - 1) / nimax_ + 1) * nimax_;
                          hipError_t cuerr_;
                          if (firstcall_) {
                              int device_;
                              hipDeviceProp_t prop;
                              firstcall_ = 0;
                              cuerr_ = hipGetDevice(&device_);
                              cutilSafeCall(cuerr_);
                              cuerr_ = hipGetDeviceProperties(&prop, device_);
                              cutilSafeCall(cuerr_);
                              if (gprec_ >= kGoosePrecisionDouble) {
                                  if (prop.major < 2 && prop.minor < 3) {
                                      fprintf(stderr, "GPU architecture sm_%d%d does not support double-precision arithmetic.\n",
                                              prop.major, prop.minor);
                                      exit(1);
                                  }
                              }
                              smemsizemax_ = prop.sharedMemPerBlock;
                              if (ksmemsize_ >= smemsizemax_) {
                                  fprintf(stderr,
                                          "Shared memory consumption in f01_calculator (=%dbyte) "
                                          "reached the limit (=%dbyte). Too many jvars or fvars used.\n",
                                          ksmemsize_, smemsizemax_);
                                  exit(1);
                              }
                              if (rsmemsize_ >= smemsizemax_) {
                                  fprintf(stderr,
                                          "Shared memory consumption in f01_reducer (=%dbyte) "
                                          "reached the limit (=%dbyte). Too many fvars used.\n",
                                          rsmemsize_, smemsizemax_);
                                  exit(1);
                              }
                              nblockmax_ = prop.maxGridSize[0];
                              nsp_ = prop.multiProcessorCount * 8; // # of spream processors.
                              // fprintf(stderr, "nblockmax:%d  nsp:%d\n", nblockmax_, nsp_);
                          }
                          nisub_ = nimax_;
                          for (ioff_ = 1; ioff_ < n; ioff_ += nisub_) {
                              if (ioff_ + nisub_ > n) {
                                  nisub_ = n - ioff_;
                              }

                              int nisub_ru_ = ((nisub_ - 1) / nip_ + 1) * nip_;
                              if (njold != n || niold != nisub_) {
                                  // Adjust # of JP fragments so that large enough # of threads (to fill all SPs, 
                                  // and to hide latency of the global memory) are dispatched, at long as each
                                  // JP fragments has several times warp size.
                                  // You may want to hand tune the following part to obtain the optimal 'njdiv_'
                                  // value for a given hardware configuration.
                                  njdiv_ = njdiv_ru_ = 1;
                                  while (nisub_ru_ / nip_ * njdiv_ * 2 <= nblockmax_ &&
                                         nisub_ * njdiv_ < nsp_ * 200 &&
                                         n / njdiv_ > kbdim_ * 6 &&
                                         njdiv_ * 2 <= rbdim_) {
                                      njdiv_ *= 2;
                                  }
                                  // njdiv_ru_ is set to njdiv_ rounded up to a power of two.
                                  while (njdiv_ru_ < njdiv_) {
                                      njdiv_ru_ *= 2;
                                  }
                                  njold = n;
                                  niold = nisub_;
#if 0
                                  fprintf(stderr, "\n");
                                  fprintf(stderr, "nj:%d  njdiv:%d  nj/njdiv:%d  ni:%d  ni * njdiv:%d\n",
                                          n, njdiv_, n/njdiv_, nisub_, nisub_ * njdiv_);
                                  fprintf(stderr, "\n");
#endif
                              }
                              dim3 kthreads_(kbdim_, 1, 1);
                              dim3 kgrids_(njdiv_, nisub_ru_ / nip_, 1);
                              dim3 rthreads_(rbdim_, 1, 1);
                              int npipe_ = nisub_ru_ / 1;
                              int nrblock_ = npipe_ * njdiv_ / rbdim_;
                              dim3 rgrids_(nrblock_,1, 1);
                              int nj_ru_ = ((n - 1 - 1) / (kbdim_ * njdiv_) + 1) * (kbdim_ * njdiv_);
                              static int jbufsize_ = 0, ibufsize_ = 0, rsubbufsize_ = 0, rbufsize_ = 0;
                              static GlobalMem<f01_jp_t> f01_jp_;
                              static GlobalMem<f01_ip_t> f01_ip_;
                              static GlobalMem<f01_result_t> f01_result_;
                              static GlobalMem<f01_result_t> f01_result_sub_;
                              if (nj_ru_ > jbufsize_) {
                                  jbufsize_ = nj_ru_;
                                  f01_jp_.realloc(jbufsize_);
                              }

                              // Here we need to alloc n IPs. Note that npipe_ IPs would not be enough,
                              // since IPs on the device memory should not be overwritten during the ioff_ loop.
                              if (ni_ru_ > ibufsize_) {
                                  ibufsize_ = ni_ru_;
                                  f01_ip_.realloc(ibufsize_);
                              }
                              if (ni_ru_ > rbufsize_) {
                                  rbufsize_ = ni_ru_;
                                  f01_result_.realloc(rbufsize_);
                              }
                              if (npipe_ * njdiv_ > rsubbufsize_) {
                                  rsubbufsize_ = npipe_ * njdiv_ ;
                                  f01_result_sub_.realloc(rsubbufsize_);
                              }

                              for (j = 1, bufidx_ = 0 ; j <n; j++, bufidx_++) {f01_jp_[bufidx_].x_j_0_ = x[j][0];f01_jp_[bufidx_].x_j_1_ = x[j][1];f01_jp_[bufidx_].x_j_2_ = x[j][2];f01_jp_[bufidx_].m_j_ = m[j];f01_jp_[bufidx_].v_j_0_ = v[j][0];f01_jp_[bufidx_].v_j_1_ = v[j][1];f01_jp_[bufidx_].v_j_2_ = v[j][2];}f01_jp_.htod(0, bufidx_);

                              if (ioff_ < n) { // otherwise nothing to send.
                                  int nipsend_ = npipe_;
                                  if (ioff_ + nipsend_ >= n) {
                                      nipsend_ = n - ioff_;
                                  }
                                  for (i = 0 ; i < nipsend_; i++) {f01_ip_[ioff_ + i].x_i_0_ = x[ioff_ + i][0];f01_ip_[ioff_ + i].x_i_1_ = x[ioff_ + i][1];f01_ip_[ioff_ + i].x_i_2_ = x[ioff_ + i][2];f01_ip_[ioff_ + i].m_i_ = m[ioff_ + i];f01_ip_[ioff_ + i].v_i_0_ = v[ioff_ + i][0];f01_ip_[ioff_ + i].v_i_1_ = v[ioff_ + i][1];f01_ip_[ioff_ + i].v_i_2_ = v[ioff_ + i][2];}f01_ip_.htod(ioff_, nipsend_);

                              }
                              f01_calculator<<<kgrids_, kthreads_, ksmemsize_>>>(ioff_, nisub_, n - 1, (double)x[0][0], (double)x[0][1], (double)x[0][2], (double)eps2, (double)m[0], f01_jp_, (f01_ip_ + ioff_), f01_result_sub_);
                              f01_reducer<<<rgrids_, rthreads_, rsmemsize_>>>(njdiv_, njdiv_ru_, (f01_result_ + ioff_), f01_result_sub_);

                              if (ioff_ < n) { // otherwise nothing to receive.
                                  int niprecv_ = npipe_;
                                  if (ioff_ + niprecv_ >= n) {
                                      niprecv_ = n - ioff_;
                                  }
                                  f01_result_.dtoh(ioff_, niprecv_);
for (i = 0 ; i < niprecv_; i++) {a1c[0] += f01_result_[ioff_ + i].a1c_0_;a1c[1] += f01_result_[ioff_ + i].a1c_1_;a1c[2] += f01_result_[ioff_ + i].a1c_2_;}
                              }
                          } // end of ioff loop.
                      } // end of api calls.





  for(i=1;i<n;i++) {
    for(k=0;k<3;k++) {
      dxb[k]=x[0][k]-x[i][k];
    }

    r1b2=dxb[0]*dxb[0]+dxb[1]*dxb[1]+dxb[2]*dxb[2];
    r1b2e=r1b2+eps2;
    r1be=rsqrt(r1b2e);

    mr1b3e=m[i]*r1be*r1be*r1be;

    a1c[0]-=
      mr1b3e*m[i]*dxb[0]* (4.0*r1be+1.25*eps)
      -mr1b3e*m[i]/m[0]
      *(4.0*(v[i][0]*v[i][0]+v[i][1]*v[i][1]+v[i][2]*v[i][2]) *dxb[0]
 -7.0*(v[i][0]*dxb[0]+v[i][1]*dxb[1]+v[i][2]*dxb[2])*v[i][0]);

    a1c[1]-=
      mr1b3e*m[i]*dxb[1]* (4.0*r1be+1.25*eps)
      -mr1b3e*m[i]/m[0]
      *(4.0*(v[i][0]*v[i][0]+v[i][1]*v[i][1]+v[i][2]*v[i][2]) *dxb[1]
     -7.0*(v[i][0]*dxb[0]+v[i][1]*dxb[1]+v[i][2]*dxb[2])*v[i][1]);

    a1c[2]-=
      mr1b3e*m[i]*dxb[2]* (4.0*r1be+1.25*eps)
      -mr1b3e*m[i]/m[0]
      *(4.0*(v[i][0]*v[i][0]+v[i][1]*v[i][1]+v[i][2]*v[i][2]) *dxb[2]
 -7.0*(v[i][0]*dxb[0]+v[i][1]*dxb[1]+v[i][2]*dxb[2])*v[i][2]);
  }

  for(k=0;k<3;k++) {
    a[0][k]+=a1c[k];
  }



  for(i=1;i<n;i++) {
    for(k=0;k<3;k++) {
      dxb[k]=x[0][k]-x[i][k];
    }

    r1b2=dxb[0]*dxb[0]+dxb[1]*dxb[1]+dxb[2]*dxb[2];
    r1b2e=r1b2+eps2;
    r1be=rsqrt(r1b2e);

    mrinv=m[0]*r1be;
    mr3inv=mrinv*r1be*r1be;

    a[i][0]+=mr3inv*dxb[0]
      * (4.0*mrinv-(v[i][0]*v[i][0]+v[i][1]*v[i][1]+v[i][2]*v[i][2]))
      + 4.0*mr3inv*(v[i][0]*dxb[0]+v[i][1]*dxb[1]+v[i][2]*dxb[2])*v[i][0];

    a[i][1]+=mr3inv*dxb[1]
      * (4.0*mrinv-(v[i][0]*v[i][0]+v[i][1]*v[i][1]+v[i][2]*v[i][2]))
      + 4.0*mr3inv*(v[i][0]*dxb[0]+v[i][1]*dxb[1]+v[i][2]*dxb[2])*v[i][1];

    a[i][2]+=mr3inv*dxb[2]
      * (4.0*mrinv-(v[i][0]*v[i][0]+v[i][1]*v[i][1]+v[i][2]*v[i][2]))
      + 4.0*mr3inv*(v[i][0]*dxb[0]+v[i][1]*dxb[1]+v[i][2]*dxb[2])*v[i][2];
  }



  for(i=1;i<n;i++){
    for(k=0;k<3;k++){
      ac[i][k]=0.0;
    }
  }


  for(i=1;i<n;i++){
    for(k=0;k<3;k++) {
      dxb[k]=x[i][k]-x[0][k];
    }

    r1b2=dxb[0]*dxb[0]+dxb[1]*dxb[1]+dxb[2]*dxb[2];
    r1b2e=r1b2+eps2;
    r1be=rsqrt(r1b2e);

    mr3inv=m[0]*r1be*r1be*r1be;

    a[i][0]+=5.0*mr3inv*m[i]*r1be*dxb[0]
      -m[i]*r1be*r1be*r1be
      *(4.0*(v[i][0]*v[i][0]+v[i][1]*v[i][1]+v[i][2]*v[i][2])*dxb[0]
       -7.0*(v[i][0]*dxb[0]+v[i][1]*dxb[1]+v[i][2]*dxb[2])*v[i][0]);

    a[i][1]+=5.0*mr3inv*m[i]*r1be*dxb[1]
      -m[i]*r1be*r1be*r1be
      *(4.0*(v[i][0]*v[i][0]+v[i][1]*v[i][1]+v[i][2]*v[i][2])*dxb[1]
       -7.0*(v[i][0]*dxb[0]+v[i][1]*dxb[1]+v[i][2]*dxb[2])*v[i][1]);

    a[i][2]+=5.0*mr3inv*m[i]*r1be*dxb[2]
      -m[i]*r1be*r1be*r1be
      *(4.0*(v[i][0]*v[i][0]+v[i][1]*v[i][1]+v[i][2]*v[i][2])*dxb[2]
       -7.0*(v[i][0]*dxb[0]+v[i][1]*dxb[1]+v[i][2]*dxb[2])*v[i][2]);
  }

                      /*
                       * dispatcher for a kernel defined in 'Limit-sticky9_2.cu'.
                       *
                       * kbdim_  : # of threads per block. fixed to a multiple of warp size.
                       * nip_    : # of IPs handled in one block.
                       * njdiv_  : # of JP fragments. determined in a heuristic manner as a function of
                       *           ni, nj and device specification such as max grid size and warp size,
                       * nblock_ : # of blocks (ni * njdiv / nip) rounded up to a multiple of nip.
                       * npipe_  : len of each IP array. ni/nip_pack rounded up to a multiple of (nth * nip_pack).
                       * nj_ru_  : nj rounded up to a multiple of (nth * njdiv).
                       */
                      {
#if SHARED_HOSTBUF
                          double * GlobalMem<double>::hostbuf = NULL;
                          int GlobalMem<double>::nbytemax = 0;
#endif
                          // Below may have a room for optimization yet.
                          const int kbdim_ = 64; 
                          const int rbdim_ = 64; 
                          const int nimax_ = kbdim_ * 128; // necessary amount of main mem is larger for larger value.
                          const size_t ksmemsize_ = kbdim_ * sizeof(f02_jp_t);
                          const size_t rsmemsize_ = rbdim_ * sizeof(f02_result_t) * 2;
                          const int nip_ = kbdim_ * 1;
                          const GoosePrecision gprec_ = kGoosePrecisionDouble;
                          static int njdiv_, njdiv_ru_;
                          static int njold = 0, niold = 0;
                          static int nblockmax_, nsp_;
                          static size_t smemsizemax_;
                          static int firstcall_ = 1;
                          int bufidx_, ioff_, nisub_;
                          int ni_ru_ = ((n - 1 - 1) / nimax_ + 1) * nimax_;
                          hipError_t cuerr_;
                          if (firstcall_) {
                              int device_;
                              hipDeviceProp_t prop;
                              firstcall_ = 0;
                              cuerr_ = hipGetDevice(&device_);
                              cutilSafeCall(cuerr_);
                              cuerr_ = hipGetDeviceProperties(&prop, device_);
                              cutilSafeCall(cuerr_);
                              if (gprec_ >= kGoosePrecisionDouble) {
                                  if (prop.major < 2 && prop.minor < 3) {
                                      fprintf(stderr, "GPU architecture sm_%d%d does not support double-precision arithmetic.\n",
                                              prop.major, prop.minor);
                                      exit(1);
                                  }
                              }
                              smemsizemax_ = prop.sharedMemPerBlock;
                              if (ksmemsize_ >= smemsizemax_) {
                                  fprintf(stderr,
                                          "Shared memory consumption in f02_calculator (=%dbyte) "
                                          "reached the limit (=%dbyte). Too many jvars or fvars used.\n",
                                          ksmemsize_, smemsizemax_);
                                  exit(1);
                              }
                              if (rsmemsize_ >= smemsizemax_) {
                                  fprintf(stderr,
                                          "Shared memory consumption in f02_reducer (=%dbyte) "
                                          "reached the limit (=%dbyte). Too many fvars used.\n",
                                          rsmemsize_, smemsizemax_);
                                  exit(1);
                              }
                              nblockmax_ = prop.maxGridSize[0];
                              nsp_ = prop.multiProcessorCount * 8; // # of spream processors.
                              // fprintf(stderr, "nblockmax:%d  nsp:%d\n", nblockmax_, nsp_);
                          }
                          nisub_ = nimax_;
                          for (ioff_ = 1; ioff_ < n; ioff_ += nisub_) {
                              if (ioff_ + nisub_ > n) {
                                  nisub_ = n - ioff_;
                              }

                              int nisub_ru_ = ((nisub_ - 1) / nip_ + 1) * nip_;
                              if (njold != n || niold != nisub_) {
                                  // Adjust # of JP fragments so that large enough # of threads (to fill all SPs, 
                                  // and to hide latency of the global memory) are dispatched, at long as each
                                  // JP fragments has several times warp size.
                                  // You may want to hand tune the following part to obtain the optimal 'njdiv_'
                                  // value for a given hardware configuration.
                                  njdiv_ = njdiv_ru_ = 1;
                                  while (nisub_ru_ / nip_ * njdiv_ * 2 <= nblockmax_ &&
                                         nisub_ * njdiv_ < nsp_ * 200 &&
                                         n / njdiv_ > kbdim_ * 6 &&
                                         njdiv_ * 2 <= rbdim_) {
                                      njdiv_ *= 2;
                                  }
                                  // njdiv_ru_ is set to njdiv_ rounded up to a power of two.
                                  while (njdiv_ru_ < njdiv_) {
                                      njdiv_ru_ *= 2;
                                  }
                                  njold = n;
                                  niold = nisub_;
#if 0
                                  fprintf(stderr, "\n");
                                  fprintf(stderr, "nj:%d  njdiv:%d  nj/njdiv:%d  ni:%d  ni * njdiv:%d\n",
                                          n, njdiv_, n/njdiv_, nisub_, nisub_ * njdiv_);
                                  fprintf(stderr, "\n");
#endif
                              }
                              dim3 kthreads_(kbdim_, 1, 1);
                              dim3 kgrids_(njdiv_, nisub_ru_ / nip_, 1);
                              dim3 rthreads_(rbdim_, 1, 1);
                              int npipe_ = nisub_ru_ / 1;
                              int nrblock_ = npipe_ * njdiv_ / rbdim_;
                              dim3 rgrids_(nrblock_,1, 1);
                              int nj_ru_ = ((n - 1 - 1) / (kbdim_ * njdiv_) + 1) * (kbdim_ * njdiv_);
                              static int jbufsize_ = 0, ibufsize_ = 0, rsubbufsize_ = 0, rbufsize_ = 0;
                              static GlobalMem<f02_jp_t> f02_jp_;
                              static GlobalMem<f02_ip_t> f02_ip_;
                              static GlobalMem<f02_result_t> f02_result_;
                              static GlobalMem<f02_result_t> f02_result_sub_;
                              if (nj_ru_ > jbufsize_) {
                                  jbufsize_ = nj_ru_;
                                  f02_jp_.realloc(jbufsize_);
                              }

                              // Here we need to alloc n IPs. Note that npipe_ IPs would not be enough,
                              // since IPs on the device memory should not be overwritten during the ioff_ loop.
                              if (ni_ru_ > ibufsize_) {
                                  ibufsize_ = ni_ru_;
                                  f02_ip_.realloc(ibufsize_);
                              }
                              if (ni_ru_ > rbufsize_) {
                                  rbufsize_ = ni_ru_;
                                  f02_result_.realloc(rbufsize_);
                              }
                              if (npipe_ * njdiv_ > rsubbufsize_) {
                                  rsubbufsize_ = npipe_ * njdiv_ ;
                                  f02_result_sub_.realloc(rsubbufsize_);
                              }

                              for (j = 1, bufidx_ = 0 ; j <n; j++, bufidx_++) {f02_jp_[bufidx_].x_j_0_ = x[j][0];f02_jp_[bufidx_].v_j_0_ = v[j][0];f02_jp_[bufidx_].x_j_1_ = x[j][1];f02_jp_[bufidx_].v_j_1_ = v[j][1];f02_jp_[bufidx_].x_j_2_ = x[j][2];f02_jp_[bufidx_].v_j_2_ = v[j][2];f02_jp_[bufidx_].m_j_ = m[j];}f02_jp_.htod(0, bufidx_);

                              if (ioff_ < n) { // otherwise nothing to send.
                                  int nipsend_ = npipe_;
                                  if (ioff_ + nipsend_ >= n) {
                                      nipsend_ = n - ioff_;
                                  }
                                  for (i = 0 ; i < nipsend_; i++) {f02_ip_[ioff_ + i].x_i_0_ = x[ioff_ + i][0];f02_ip_[ioff_ + i].v_i_0_ = v[ioff_ + i][0];f02_ip_[ioff_ + i].x_i_1_ = x[ioff_ + i][1];f02_ip_[ioff_ + i].v_i_1_ = v[ioff_ + i][1];f02_ip_[ioff_ + i].x_i_2_ = x[ioff_ + i][2];f02_ip_[ioff_ + i].v_i_2_ = v[ioff_ + i][2];}f02_ip_.htod(ioff_, nipsend_);

                              }
                              f02_calculator<<<kgrids_, kthreads_, ksmemsize_>>>(ioff_, nisub_, n - 1, (double)x[0][0], (double)x[0][1], (double)x[0][2], (double)eps2, (double)m[0], f02_jp_, (f02_ip_ + ioff_), f02_result_sub_);
                              f02_reducer<<<rgrids_, rthreads_, rsmemsize_>>>(njdiv_, njdiv_ru_, (f02_result_ + ioff_), f02_result_sub_);

                              if (ioff_ < n) { // otherwise nothing to receive.
                                  int niprecv_ = npipe_;
                                  if (ioff_ + niprecv_ >= n) {
                                      niprecv_ = n - ioff_;
                                  }
                                  f02_result_.dtoh(ioff_, niprecv_);
for (i = 0 ; i < niprecv_; i++) {ac[ioff_ + i][0] = f02_result_[ioff_ + i].ac_i_0_;ac[ioff_ + i][1] = f02_result_[ioff_ + i].ac_i_1_;ac[ioff_ + i][2] = f02_result_[ioff_ + i].ac_i_2_;}
                              }
                          } // end of ioff loop.
                      } // end of api calls.





  for(i=1;i<n;i++){
    for(k=0;k<3;k++){
      a[i][k]+=ac[i][k];
    }
  }

  for(i=1;i<n;i++){
    for(k=0;k<3;k++) {
      dxb[k]=x[0][k]-x[i][k];
    }

    r1b2=dxb[0]*dxb[0]+dxb[1]*dxb[1]+dxb[2]*dxb[2];
    r1b2e=r1b2+eps2;
    r1be=rsqrt(r1b2e);
    mr3inv=m[i]*r1be*r1be*r1be;

    a[i][0]-=mr3inv*m[0]*dxb[0]
      *(4.0/eps+1.25*r1be+0.25*r1b2*r1be*r1be*r1be)
      -3.5*mr3inv*m[0]*(1.0/eps-r1be)*dxb[0]
      -mr3inv
      *(4.0*(v[i][0]*v[i][0]+v[i][1]*v[i][1]+v[i][2]*v[i][2])*dxb[0]
       -7.0*(v[i][0]*dxb[0]+v[i][1]*dxb[1]+v[i][2]*dxb[2])*v[i][0]);

    a[i][1]-=mr3inv*m[0]*dxb[1]
      *(4.0/eps+1.25*r1be+0.25*r1b2*r1be*r1be*r1be)
      -3.5*mr3inv*m[0]*(1.0/eps-r1be)*dxb[1]
      -mr3inv
      *(4.0*(v[i][0]*v[i][0]+v[i][1]*v[i][1]+v[i][2]*v[i][2])*dxb[1]
       -7.0*(v[i][0]*dxb[0]+v[i][1]*dxb[1]+v[i][2]*dxb[2])*v[i][1]);

    a[i][2]-=mr3inv*m[0]*dxb[2]
      *(4.0/eps+1.25*r1be+0.25*r1b2*r1be*r1be*r1be)
      -3.5*mr3inv*m[0]*(1.0/eps-r1be)*dxb[2]
      -mr3inv
      *(4.0*(v[i][0]*v[i][0]+v[i][1]*v[i][1]+v[i][2]*v[i][2])*dxb[2]
       -7.0*(v[i][0]*dxb[0]+v[i][1]*dxb[1]+v[i][2]*dxb[2])*v[i][2]);
  }

}


void energy(double t,
            double x[270000][3],
            double v[270000][3],
            double m[270000],
            int n,
            double init_ene,
            double eps)
{
  double pot[270000], DE, ene, eps2;
  double dx[3], dxa[3], dxb[3], dxab[3];
  double r2, rinv, mrinv, vi2;
  double r1a2e, r1ainv, mr1ainv, r1b2e, r1binv, mr1binv;
  double rab2e, rabinv;
  double kin_n, pot_n;
  double pot_pn, pot_pn2[270000];
  int i,j,k;
  double cm[3];

  eps2=eps*eps;
  kin_n=0.0;
  pot_n=0.0;
  pot_pn=0.0;



  for (i=0;i<n;i++) {
    kin_n+=m[i]*(v[i][0]*v[i][0]+v[i][1]*v[i][1]+v[i][2]*v[i][2]);
  }

  kin_n*=0.5;

  for (i=0;i<n;i++) {
    pot[i]=0.0;
  }

                      /*
                       * dispatcher for a kernel defined in 'Limit-sticky9_3.cu'.
                       *
                       * kbdim_  : # of threads per block. fixed to a multiple of warp size.
                       * nip_    : # of IPs handled in one block.
                       * njdiv_  : # of JP fragments. determined in a heuristic manner as a function of
                       *           ni, nj and device specification such as max grid size and warp size,
                       * nblock_ : # of blocks (ni * njdiv / nip) rounded up to a multiple of nip.
                       * npipe_  : len of each IP array. ni/nip_pack rounded up to a multiple of (nth * nip_pack).
                       * nj_ru_  : nj rounded up to a multiple of (nth * njdiv).
                       */
                      {
#if SHARED_HOSTBUF
                          double * GlobalMem<double>::hostbuf = NULL;
                          int GlobalMem<double>::nbytemax = 0;
#endif
                          // Below may have a room for optimization yet.
                          const int kbdim_ = 64; 
                          const int rbdim_ = 64; 
                          const int nimax_ = kbdim_ * 128; // necessary amount of main mem is larger for larger value.
                          const size_t ksmemsize_ = kbdim_ * sizeof(f03_jp_t);
                          const size_t rsmemsize_ = rbdim_ * sizeof(f03_result_t) * 2;
                          const int nip_ = kbdim_ * 1;
                          const GoosePrecision gprec_ = kGoosePrecisionDouble;
                          static int njdiv_, njdiv_ru_;
                          static int njold = 0, niold = 0;
                          static int nblockmax_, nsp_;
                          static size_t smemsizemax_;
                          static int firstcall_ = 1;
                          int bufidx_, ioff_, nisub_;
                          int ni_ru_ = ((n - 0 - 1) / nimax_ + 1) * nimax_;
                          hipError_t cuerr_;
                          if (firstcall_) {
                              int device_;
                              hipDeviceProp_t prop;
                              firstcall_ = 0;
                              cuerr_ = hipGetDevice(&device_);
                              cutilSafeCall(cuerr_);
                              cuerr_ = hipGetDeviceProperties(&prop, device_);
                              cutilSafeCall(cuerr_);
                              if (gprec_ >= kGoosePrecisionDouble) {
                                  if (prop.major < 2 && prop.minor < 3) {
                                      fprintf(stderr, "GPU architecture sm_%d%d does not support double-precision arithmetic.\n",
                                              prop.major, prop.minor);
                                      exit(1);
                                  }
                              }
                              smemsizemax_ = prop.sharedMemPerBlock;
                              if (ksmemsize_ >= smemsizemax_) {
                                  fprintf(stderr,
                                          "Shared memory consumption in f03_calculator (=%dbyte) "
                                          "reached the limit (=%dbyte). Too many jvars or fvars used.\n",
                                          ksmemsize_, smemsizemax_);
                                  exit(1);
                              }
                              if (rsmemsize_ >= smemsizemax_) {
                                  fprintf(stderr,
                                          "Shared memory consumption in f03_reducer (=%dbyte) "
                                          "reached the limit (=%dbyte). Too many fvars used.\n",
                                          rsmemsize_, smemsizemax_);
                                  exit(1);
                              }
                              nblockmax_ = prop.maxGridSize[0];
                              nsp_ = prop.multiProcessorCount * 8; // # of spream processors.
                              // fprintf(stderr, "nblockmax:%d  nsp:%d\n", nblockmax_, nsp_);
                          }
                          nisub_ = nimax_;
                          for (ioff_ = 0; ioff_ < n; ioff_ += nisub_) {
                              if (ioff_ + nisub_ > n) {
                                  nisub_ = n - ioff_;
                              }

                              int nisub_ru_ = ((nisub_ - 1) / nip_ + 1) * nip_;
                              if (njold != n || niold != nisub_) {
                                  // Adjust # of JP fragments so that large enough # of threads (to fill all SPs, 
                                  // and to hide latency of the global memory) are dispatched, at long as each
                                  // JP fragments has several times warp size.
                                  // You may want to hand tune the following part to obtain the optimal 'njdiv_'
                                  // value for a given hardware configuration.
                                  njdiv_ = njdiv_ru_ = 1;
                                  while (nisub_ru_ / nip_ * njdiv_ * 2 <= nblockmax_ &&
                                         nisub_ * njdiv_ < nsp_ * 200 &&
                                         n / njdiv_ > kbdim_ * 6 &&
                                         njdiv_ * 2 <= rbdim_) {
                                      njdiv_ *= 2;
                                  }
                                  // njdiv_ru_ is set to njdiv_ rounded up to a power of two.
                                  while (njdiv_ru_ < njdiv_) {
                                      njdiv_ru_ *= 2;
                                  }
                                  njold = n;
                                  niold = nisub_;
#if 0
                                  fprintf(stderr, "\n");
                                  fprintf(stderr, "nj:%d  njdiv:%d  nj/njdiv:%d  ni:%d  ni * njdiv:%d\n",
                                          n, njdiv_, n/njdiv_, nisub_, nisub_ * njdiv_);
                                  fprintf(stderr, "\n");
#endif
                              }
                              dim3 kthreads_(kbdim_, 1, 1);
                              dim3 kgrids_(njdiv_, nisub_ru_ / nip_, 1);
                              dim3 rthreads_(rbdim_, 1, 1);
                              int npipe_ = nisub_ru_ / 1;
                              int nrblock_ = npipe_ * njdiv_ / rbdim_;
                              dim3 rgrids_(nrblock_,1, 1);
                              int nj_ru_ = ((n - 0 - 1) / (kbdim_ * njdiv_) + 1) * (kbdim_ * njdiv_);
                              static int jbufsize_ = 0, ibufsize_ = 0, rsubbufsize_ = 0, rbufsize_ = 0;
                              static GlobalMem<f03_jp_t> f03_jp_;
                              static GlobalMem<f03_ip_t> f03_ip_;
                              static GlobalMem<f03_result_t> f03_result_;
                              static GlobalMem<f03_result_t> f03_result_sub_;
                              if (nj_ru_ > jbufsize_) {
                                  jbufsize_ = nj_ru_;
                                  f03_jp_.realloc(jbufsize_);
                              }

                              // Here we need to alloc n IPs. Note that npipe_ IPs would not be enough,
                              // since IPs on the device memory should not be overwritten during the ioff_ loop.
                              if (ni_ru_ > ibufsize_) {
                                  ibufsize_ = ni_ru_;
                                  f03_ip_.realloc(ibufsize_);
                              }
                              if (ni_ru_ > rbufsize_) {
                                  rbufsize_ = ni_ru_;
                                  f03_result_.realloc(rbufsize_);
                              }
                              if (npipe_ * njdiv_ > rsubbufsize_) {
                                  rsubbufsize_ = npipe_ * njdiv_ ;
                                  f03_result_sub_.realloc(rsubbufsize_);
                              }

                              for (j = 0, bufidx_ = 0 ; j <n; j++, bufidx_++) {f03_jp_[bufidx_].x_j_0_ = x[j][0];f03_jp_[bufidx_].x_j_1_ = x[j][1];f03_jp_[bufidx_].x_j_2_ = x[j][2];f03_jp_[bufidx_].m_j_ = m[j];}f03_jp_.htod(0, bufidx_);

                              if (ioff_ < n) { // otherwise nothing to send.
                                  int nipsend_ = npipe_;
                                  if (ioff_ + nipsend_ >= n) {
                                      nipsend_ = n - ioff_;
                                  }
                                  for (i = 0 ; i < nipsend_; i++) {f03_ip_[ioff_ + i].x_i_0_ = x[ioff_ + i][0];f03_ip_[ioff_ + i].x_i_1_ = x[ioff_ + i][1];f03_ip_[ioff_ + i].x_i_2_ = x[ioff_ + i][2];}f03_ip_.htod(ioff_, nipsend_);

                              }
                              f03_calculator<<<kgrids_, kthreads_, ksmemsize_>>>(ioff_, nisub_, n - 0, (double)eps2, f03_jp_, (f03_ip_ + ioff_), f03_result_sub_);
                              f03_reducer<<<rgrids_, rthreads_, rsmemsize_>>>(njdiv_, njdiv_ru_, (f03_result_ + ioff_), f03_result_sub_);

                              if (ioff_ < n) { // otherwise nothing to receive.
                                  int niprecv_ = npipe_;
                                  if (ioff_ + niprecv_ >= n) {
                                      niprecv_ = n - ioff_;
                                  }
                                  f03_result_.dtoh(ioff_, niprecv_);
for (i = 0 ; i < niprecv_; i++) {pot[ioff_ + i] = f03_result_[ioff_ + i].pot_i_;}
                              }
                          } // end of ioff loop.
                      } // end of api calls.
for (i=0;i<n;i++) {
    pot[i]+=m[i]/sqrt(eps2);
    pot[i]*=m[i];
  }


  for (i=0;i<n;i++) {
    pot_n+=pot[i];
  }

  pot_n*=0.5;



  pot_pn=0.0;

  for (i=1;i<n;i++) {
    for (k=0;k<3;k++) dx[k]=x[i][k]-x[0][k];

    r1a2e=dx[0]*dx[0]+dx[1]*dx[1]+dx[2]*dx[2]+eps2;
    r1ainv=rsqrt(r1a2e);
    mr1ainv=m[0]*r1ainv;

    vi2=v[i][0]*v[i][0]+v[i][1]*v[i][1]+v[i][2]*v[i][2];

    pot_pn+=0.375*m[i]*vi2*vi2 + 1.5*mr1ainv*m[i]*vi2
      +0.5*mr1ainv*mr1ainv*m[i]
      +0.5*mr1ainv*m[i]
      *(3.0*vi2-7.0*(v[0][0]*v[i][0]+v[0][1]*v[i][1]+v[0][2]*v[i][2])
 -(dx[0]*v[0][0]+dx[1]*v[0][1]+dx[2]*v[0][2])
 *(dx[0]*v[i][0]+dx[1]*v[i][1]+dx[2]*v[i][2])
 *r1ainv*r1ainv);
  }

  for (i=1;i<n;i++) {
    pot_pn2[i] = 0.0;
  }

                      /*
                       * dispatcher for a kernel defined in 'Limit-sticky9_4.cu'.
                       *
                       * kbdim_  : # of threads per block. fixed to a multiple of warp size.
                       * nip_    : # of IPs handled in one block.
                       * njdiv_  : # of JP fragments. determined in a heuristic manner as a function of
                       *           ni, nj and device specification such as max grid size and warp size,
                       * nblock_ : # of blocks (ni * njdiv / nip) rounded up to a multiple of nip.
                       * npipe_  : len of each IP array. ni/nip_pack rounded up to a multiple of (nth * nip_pack).
                       * nj_ru_  : nj rounded up to a multiple of (nth * njdiv).
                       */
                      {
#if SHARED_HOSTBUF
                          double * GlobalMem<double>::hostbuf = NULL;
                          int GlobalMem<double>::nbytemax = 0;
#endif
                          // Below may have a room for optimization yet.
                          const int kbdim_ = 64; 
                          const int rbdim_ = 64; 
                          const int nimax_ = kbdim_ * 128; // necessary amount of main mem is larger for larger value.
                          const size_t ksmemsize_ = kbdim_ * sizeof(f04_jp_t);
                          const size_t rsmemsize_ = rbdim_ * sizeof(f04_result_t) * 2;
                          const int nip_ = kbdim_ * 1;
                          const GoosePrecision gprec_ = kGoosePrecisionDouble;
                          static int njdiv_, njdiv_ru_;
                          static int njold = 0, niold = 0;
                          static int nblockmax_, nsp_;
                          static size_t smemsizemax_;
                          static int firstcall_ = 1;
                          int bufidx_, ioff_, nisub_;
                          int ni_ru_ = ((n - 1 - 1) / nimax_ + 1) * nimax_;
                          hipError_t cuerr_;
                          if (firstcall_) {
                              int device_;
                              hipDeviceProp_t prop;
                              firstcall_ = 0;
                              cuerr_ = hipGetDevice(&device_);
                              cutilSafeCall(cuerr_);
                              cuerr_ = hipGetDeviceProperties(&prop, device_);
                              cutilSafeCall(cuerr_);
                              if (gprec_ >= kGoosePrecisionDouble) {
                                  if (prop.major < 2 && prop.minor < 3) {
                                      fprintf(stderr, "GPU architecture sm_%d%d does not support double-precision arithmetic.\n",
                                              prop.major, prop.minor);
                                      exit(1);
                                  }
                              }
                              smemsizemax_ = prop.sharedMemPerBlock;
                              if (ksmemsize_ >= smemsizemax_) {
                                  fprintf(stderr,
                                          "Shared memory consumption in f04_calculator (=%dbyte) "
                                          "reached the limit (=%dbyte). Too many jvars or fvars used.\n",
                                          ksmemsize_, smemsizemax_);
                                  exit(1);
                              }
                              if (rsmemsize_ >= smemsizemax_) {
                                  fprintf(stderr,
                                          "Shared memory consumption in f04_reducer (=%dbyte) "
                                          "reached the limit (=%dbyte). Too many fvars used.\n",
                                          rsmemsize_, smemsizemax_);
                                  exit(1);
                              }
                              nblockmax_ = prop.maxGridSize[0];
                              nsp_ = prop.multiProcessorCount * 8; // # of spream processors.
                              // fprintf(stderr, "nblockmax:%d  nsp:%d\n", nblockmax_, nsp_);
                          }
                          nisub_ = nimax_;
                          for (ioff_ = 1; ioff_ < n; ioff_ += nisub_) {
                              if (ioff_ + nisub_ > n) {
                                  nisub_ = n - ioff_;
                              }

                              int nisub_ru_ = ((nisub_ - 1) / nip_ + 1) * nip_;
                              if (njold != n || niold != nisub_) {
                                  // Adjust # of JP fragments so that large enough # of threads (to fill all SPs, 
                                  // and to hide latency of the global memory) are dispatched, at long as each
                                  // JP fragments has several times warp size.
                                  // You may want to hand tune the following part to obtain the optimal 'njdiv_'
                                  // value for a given hardware configuration.
                                  njdiv_ = njdiv_ru_ = 1;
                                  while (nisub_ru_ / nip_ * njdiv_ * 2 <= nblockmax_ &&
                                         nisub_ * njdiv_ < nsp_ * 200 &&
                                         n / njdiv_ > kbdim_ * 6 &&
                                         njdiv_ * 2 <= rbdim_) {
                                      njdiv_ *= 2;
                                  }
                                  // njdiv_ru_ is set to njdiv_ rounded up to a power of two.
                                  while (njdiv_ru_ < njdiv_) {
                                      njdiv_ru_ *= 2;
                                  }
                                  njold = n;
                                  niold = nisub_;
#if 0
                                  fprintf(stderr, "\n");
                                  fprintf(stderr, "nj:%d  njdiv:%d  nj/njdiv:%d  ni:%d  ni * njdiv:%d\n",
                                          n, njdiv_, n/njdiv_, nisub_, nisub_ * njdiv_);
                                  fprintf(stderr, "\n");
#endif
                              }
                              dim3 kthreads_(kbdim_, 1, 1);
                              dim3 kgrids_(njdiv_, nisub_ru_ / nip_, 1);
                              dim3 rthreads_(rbdim_, 1, 1);
                              int npipe_ = nisub_ru_ / 1;
                              int nrblock_ = npipe_ * njdiv_ / rbdim_;
                              dim3 rgrids_(nrblock_,1, 1);
                              int nj_ru_ = ((n - 1 - 1) / (kbdim_ * njdiv_) + 1) * (kbdim_ * njdiv_);
                              static int jbufsize_ = 0, ibufsize_ = 0, rsubbufsize_ = 0, rbufsize_ = 0;
                              static GlobalMem<f04_jp_t> f04_jp_;
                              static GlobalMem<f04_ip_t> f04_ip_;
                              static GlobalMem<f04_result_t> f04_result_;
                              static GlobalMem<f04_result_t> f04_result_sub_;
                              if (nj_ru_ > jbufsize_) {
                                  jbufsize_ = nj_ru_;
                                  f04_jp_.realloc(jbufsize_);
                              }

                              // Here we need to alloc n IPs. Note that npipe_ IPs would not be enough,
                              // since IPs on the device memory should not be overwritten during the ioff_ loop.
                              if (ni_ru_ > ibufsize_) {
                                  ibufsize_ = ni_ru_;
                                  f04_ip_.realloc(ibufsize_);
                              }
                              if (ni_ru_ > rbufsize_) {
                                  rbufsize_ = ni_ru_;
                                  f04_result_.realloc(rbufsize_);
                              }
                              if (npipe_ * njdiv_ > rsubbufsize_) {
                                  rsubbufsize_ = npipe_ * njdiv_ ;
                                  f04_result_sub_.realloc(rsubbufsize_);
                              }

                              for (j = 1, bufidx_ = 0 ; j <n; j++, bufidx_++) {f04_jp_[bufidx_].x_j_0_ = x[j][0];f04_jp_[bufidx_].x_j_1_ = x[j][1];f04_jp_[bufidx_].x_j_2_ = x[j][2];f04_jp_[bufidx_].m_j_ = m[j];f04_jp_[bufidx_].v_j_0_ = v[j][0];f04_jp_[bufidx_].v_j_1_ = v[j][1];f04_jp_[bufidx_].v_j_2_ = v[j][2];}f04_jp_.htod(0, bufidx_);

                              if (ioff_ < n) { // otherwise nothing to send.
                                  int nipsend_ = npipe_;
                                  if (ioff_ + nipsend_ >= n) {
                                      nipsend_ = n - ioff_;
                                  }
                                  for (i = 0 ; i < nipsend_; i++) {f04_ip_[ioff_ + i].x_i_0_ = x[ioff_ + i][0];f04_ip_[ioff_ + i].x_i_1_ = x[ioff_ + i][1];f04_ip_[ioff_ + i].x_i_2_ = x[ioff_ + i][2];f04_ip_[ioff_ + i].v_i_0_ = v[ioff_ + i][0];f04_ip_[ioff_ + i].v_i_1_ = v[ioff_ + i][1];f04_ip_[ioff_ + i].v_i_2_ = v[ioff_ + i][2];f04_ip_[ioff_ + i].m_i_ = m[ioff_ + i];}f04_ip_.htod(ioff_, nipsend_);

                              }
                              f04_calculator<<<kgrids_, kthreads_, ksmemsize_>>>(ioff_, nisub_, n - 1, (double)x[0][0], (double)x[0][1], (double)x[0][2], (double)eps2, (double)m[0], f04_jp_, (f04_ip_ + ioff_), f04_result_sub_);
                              f04_reducer<<<rgrids_, rthreads_, rsmemsize_>>>(njdiv_, njdiv_ru_, (f04_result_ + ioff_), f04_result_sub_);

                              if (ioff_ < n) { // otherwise nothing to receive.
                                  int niprecv_ = npipe_;
                                  if (ioff_ + niprecv_ >= n) {
                                      niprecv_ = n - ioff_;
                                  }
                                  f04_result_.dtoh(ioff_, niprecv_);
for (i = 0 ; i < niprecv_; i++) {pot_pn2[ioff_ + i] = f04_result_[ioff_ + i].pot_pn2_i_;}
                              }
                          } // end of ioff loop.
                      } // end of api calls.


  for (i=1;i<n;i++) {
     for (k=0;k<3;k++) {
       dxa[k]=x[i][k]-x[0][k];
     }

      r1a2e=dxa[0]*dxa[0]+dxa[1]*dxa[1]+dxa[2]*dxa[2]+eps2;
      r1ainv=rsqrt(r1a2e);
     vi2=v[i][0]*v[i][0]+v[i][1]*v[i][1]+v[i][2]*v[i][2];

     pot_pn2[i]-=0.25*m[i]*m[i]/eps*(-vi2)
                 +m[0]*m[i]*m[i]*(r1ainv/eps+0.5*r1ainv*r1ainv);
   }

  ene=kin_n+pot_n+pot_pn;

  for (i=1;i<n;i++) {
    ene+=pot_pn2[i];
  }

  DE=(init_ene-ene)/ene;

  printf("time = %g\n",t);
  printf("pot = %22.15e kin = %22.15e \n pot_pn = %22.15e \n  total= %22.15e ratio = %e\n",
  pot_n, kin_n, pot_pn, ene, kin_n/pot_n);
  printf(" DE = %e %g\n",DE,t);
}

void initial_energy(double x[270000][3],
                    double v[270000][3],
                    double m[270000],
                    int n,
                    double *init_ene,
                    double eps)
{
  double pot[270000], ene, eps2;
  double dx[3], dxa[3], dxb[3], dxab[3];
  double r2, rinv, mrinv, vi2;
  double r1a2e, r1ainv, mr1ainv, r1b2e, r1binv, mr1binv;
  double rab2e, rabinv;
  double kin_n, pot_n;
  double pot_pn, pot_pn2[270000];
  int i,j,k;
  double cm[3];

  eps2=eps*eps;
  kin_n=0.0;
  pot_n=0.0;
  pot_pn=0.0;



  for (i=0;i<n;i++) {
    kin_n+=m[i]*(v[i][0]*v[i][0]+v[i][1]*v[i][1]+v[i][2]*v[i][2]);
  }

  kin_n*=0.5;

  for (i=0;i<n;i++) {
    pot[i]=0.0;
  }

                      /*
                       * dispatcher for a kernel defined in 'Limit-sticky9_5.cu'.
                       *
                       * kbdim_  : # of threads per block. fixed to a multiple of warp size.
                       * nip_    : # of IPs handled in one block.
                       * njdiv_  : # of JP fragments. determined in a heuristic manner as a function of
                       *           ni, nj and device specification such as max grid size and warp size,
                       * nblock_ : # of blocks (ni * njdiv / nip) rounded up to a multiple of nip.
                       * npipe_  : len of each IP array. ni/nip_pack rounded up to a multiple of (nth * nip_pack).
                       * nj_ru_  : nj rounded up to a multiple of (nth * njdiv).
                       */
                      {
#if SHARED_HOSTBUF
                          double * GlobalMem<double>::hostbuf = NULL;
                          int GlobalMem<double>::nbytemax = 0;
#endif
                          // Below may have a room for optimization yet.
                          const int kbdim_ = 64; 
                          const int rbdim_ = 64; 
                          const int nimax_ = kbdim_ * 128; // necessary amount of main mem is larger for larger value.
                          const size_t ksmemsize_ = kbdim_ * sizeof(f05_jp_t);
                          const size_t rsmemsize_ = rbdim_ * sizeof(f05_result_t) * 2;
                          const int nip_ = kbdim_ * 1;
                          const GoosePrecision gprec_ = kGoosePrecisionDouble;
                          static int njdiv_, njdiv_ru_;
                          static int njold = 0, niold = 0;
                          static int nblockmax_, nsp_;
                          static size_t smemsizemax_;
                          static int firstcall_ = 1;
                          int bufidx_, ioff_, nisub_;
                          int ni_ru_ = ((n - 0 - 1) / nimax_ + 1) * nimax_;
                          hipError_t cuerr_;
                          if (firstcall_) {
                              int device_;
                              hipDeviceProp_t prop;
                              firstcall_ = 0;
                              cuerr_ = hipGetDevice(&device_);
                              cutilSafeCall(cuerr_);
                              cuerr_ = hipGetDeviceProperties(&prop, device_);
                              cutilSafeCall(cuerr_);
                              if (gprec_ >= kGoosePrecisionDouble) {
                                  if (prop.major < 2 && prop.minor < 3) {
                                      fprintf(stderr, "GPU architecture sm_%d%d does not support double-precision arithmetic.\n",
                                              prop.major, prop.minor);
                                      exit(1);
                                  }
                              }
                              smemsizemax_ = prop.sharedMemPerBlock;
                              if (ksmemsize_ >= smemsizemax_) {
                                  fprintf(stderr,
                                          "Shared memory consumption in f05_calculator (=%dbyte) "
                                          "reached the limit (=%dbyte). Too many jvars or fvars used.\n",
                                          ksmemsize_, smemsizemax_);
                                  exit(1);
                              }
                              if (rsmemsize_ >= smemsizemax_) {
                                  fprintf(stderr,
                                          "Shared memory consumption in f05_reducer (=%dbyte) "
                                          "reached the limit (=%dbyte). Too many fvars used.\n",
                                          rsmemsize_, smemsizemax_);
                                  exit(1);
                              }
                              nblockmax_ = prop.maxGridSize[0];
                              nsp_ = prop.multiProcessorCount * 8; // # of spream processors.
                              // fprintf(stderr, "nblockmax:%d  nsp:%d\n", nblockmax_, nsp_);
                          }
                          nisub_ = nimax_;
                          for (ioff_ = 0; ioff_ < n; ioff_ += nisub_) {
                              if (ioff_ + nisub_ > n) {
                                  nisub_ = n - ioff_;
                              }

                              int nisub_ru_ = ((nisub_ - 1) / nip_ + 1) * nip_;
                              if (njold != n || niold != nisub_) {
                                  // Adjust # of JP fragments so that large enough # of threads (to fill all SPs, 
                                  // and to hide latency of the global memory) are dispatched, at long as each
                                  // JP fragments has several times warp size.
                                  // You may want to hand tune the following part to obtain the optimal 'njdiv_'
                                  // value for a given hardware configuration.
                                  njdiv_ = njdiv_ru_ = 1;
                                  while (nisub_ru_ / nip_ * njdiv_ * 2 <= nblockmax_ &&
                                         nisub_ * njdiv_ < nsp_ * 200 &&
                                         n / njdiv_ > kbdim_ * 6 &&
                                         njdiv_ * 2 <= rbdim_) {
                                      njdiv_ *= 2;
                                  }
                                  // njdiv_ru_ is set to njdiv_ rounded up to a power of two.
                                  while (njdiv_ru_ < njdiv_) {
                                      njdiv_ru_ *= 2;
                                  }
                                  njold = n;
                                  niold = nisub_;
#if 0
                                  fprintf(stderr, "\n");
                                  fprintf(stderr, "nj:%d  njdiv:%d  nj/njdiv:%d  ni:%d  ni * njdiv:%d\n",
                                          n, njdiv_, n/njdiv_, nisub_, nisub_ * njdiv_);
                                  fprintf(stderr, "\n");
#endif
                              }
                              dim3 kthreads_(kbdim_, 1, 1);
                              dim3 kgrids_(njdiv_, nisub_ru_ / nip_, 1);
                              dim3 rthreads_(rbdim_, 1, 1);
                              int npipe_ = nisub_ru_ / 1;
                              int nrblock_ = npipe_ * njdiv_ / rbdim_;
                              dim3 rgrids_(nrblock_,1, 1);
                              int nj_ru_ = ((n - 0 - 1) / (kbdim_ * njdiv_) + 1) * (kbdim_ * njdiv_);
                              static int jbufsize_ = 0, ibufsize_ = 0, rsubbufsize_ = 0, rbufsize_ = 0;
                              static GlobalMem<f05_jp_t> f05_jp_;
                              static GlobalMem<f05_ip_t> f05_ip_;
                              static GlobalMem<f05_result_t> f05_result_;
                              static GlobalMem<f05_result_t> f05_result_sub_;
                              if (nj_ru_ > jbufsize_) {
                                  jbufsize_ = nj_ru_;
                                  f05_jp_.realloc(jbufsize_);
                              }

                              // Here we need to alloc n IPs. Note that npipe_ IPs would not be enough,
                              // since IPs on the device memory should not be overwritten during the ioff_ loop.
                              if (ni_ru_ > ibufsize_) {
                                  ibufsize_ = ni_ru_;
                                  f05_ip_.realloc(ibufsize_);
                              }
                              if (ni_ru_ > rbufsize_) {
                                  rbufsize_ = ni_ru_;
                                  f05_result_.realloc(rbufsize_);
                              }
                              if (npipe_ * njdiv_ > rsubbufsize_) {
                                  rsubbufsize_ = npipe_ * njdiv_ ;
                                  f05_result_sub_.realloc(rsubbufsize_);
                              }

                              for (j = 0, bufidx_ = 0 ; j <n; j++, bufidx_++) {f05_jp_[bufidx_].x_j_0_ = x[j][0];f05_jp_[bufidx_].x_j_1_ = x[j][1];f05_jp_[bufidx_].x_j_2_ = x[j][2];f05_jp_[bufidx_].m_j_ = m[j];}f05_jp_.htod(0, bufidx_);

                              if (ioff_ < n) { // otherwise nothing to send.
                                  int nipsend_ = npipe_;
                                  if (ioff_ + nipsend_ >= n) {
                                      nipsend_ = n - ioff_;
                                  }
                                  for (i = 0 ; i < nipsend_; i++) {f05_ip_[ioff_ + i].x_i_0_ = x[ioff_ + i][0];f05_ip_[ioff_ + i].x_i_1_ = x[ioff_ + i][1];f05_ip_[ioff_ + i].x_i_2_ = x[ioff_ + i][2];}f05_ip_.htod(ioff_, nipsend_);

                              }
                              f05_calculator<<<kgrids_, kthreads_, ksmemsize_>>>(ioff_, nisub_, n - 0, (double)eps2, f05_jp_, (f05_ip_ + ioff_), f05_result_sub_);
                              f05_reducer<<<rgrids_, rthreads_, rsmemsize_>>>(njdiv_, njdiv_ru_, (f05_result_ + ioff_), f05_result_sub_);

                              if (ioff_ < n) { // otherwise nothing to receive.
                                  int niprecv_ = npipe_;
                                  if (ioff_ + niprecv_ >= n) {
                                      niprecv_ = n - ioff_;
                                  }
                                  f05_result_.dtoh(ioff_, niprecv_);
for (i = 0 ; i < niprecv_; i++) {pot[ioff_ + i] = f05_result_[ioff_ + i].pot_i_;}
                              }
                          } // end of ioff loop.
                      } // end of api calls.
for (i=0;i<n;i++) {
    pot[i]+=m[i]/sqrt(eps2);
    pot[i]*=m[i];
  }


  for (i=0;i<n;i++) {
    pot_n+=pot[i];
  }

  pot_n*=0.5;



  pot_pn=0.0;

  for (i=1;i<n;i++) {
    for (k=0;k<3;k++) dx[k]=x[i][k]-x[0][k];

    r1a2e=dx[0]*dx[0]+dx[1]*dx[1]+dx[2]*dx[2]+eps2;
    r1ainv=rsqrt(r1a2e);
    mr1ainv=m[0]*r1ainv;

    vi2=v[i][0]*v[i][0]+v[i][1]*v[i][1]+v[i][2]*v[i][2];

    pot_pn+=0.375*m[i]*vi2*vi2 + 1.5*mr1ainv*m[i]*vi2
      +0.5*mr1ainv*mr1ainv*m[i]
      +0.5*mr1ainv*m[i]
      *(3.0*vi2-7.0*(v[0][0]*v[i][0]+v[0][1]*v[i][1]+v[0][2]*v[i][2])
 -(dx[0]*v[0][0]+dx[1]*v[0][1]+dx[2]*v[0][2])
 *(dx[0]*v[i][0]+dx[1]*v[i][1]+dx[2]*v[i][2])
 *r1ainv*r1ainv);
  }





  for (i=1;i<n;i++) {
    pot_pn2[i] = 0.0;
  }

                      /*
                       * dispatcher for a kernel defined in 'Limit-sticky9_6.cu'.
                       *
                       * kbdim_  : # of threads per block. fixed to a multiple of warp size.
                       * nip_    : # of IPs handled in one block.
                       * njdiv_  : # of JP fragments. determined in a heuristic manner as a function of
                       *           ni, nj and device specification such as max grid size and warp size,
                       * nblock_ : # of blocks (ni * njdiv / nip) rounded up to a multiple of nip.
                       * npipe_  : len of each IP array. ni/nip_pack rounded up to a multiple of (nth * nip_pack).
                       * nj_ru_  : nj rounded up to a multiple of (nth * njdiv).
                       */
                      {
#if SHARED_HOSTBUF
                          double * GlobalMem<double>::hostbuf = NULL;
                          int GlobalMem<double>::nbytemax = 0;
#endif
                          // Below may have a room for optimization yet.
                          const int kbdim_ = 64; 
                          const int rbdim_ = 64; 
                          const int nimax_ = kbdim_ * 128; // necessary amount of main mem is larger for larger value.
                          const size_t ksmemsize_ = kbdim_ * sizeof(f06_jp_t);
                          const size_t rsmemsize_ = rbdim_ * sizeof(f06_result_t) * 2;
                          const int nip_ = kbdim_ * 1;
                          const GoosePrecision gprec_ = kGoosePrecisionDouble;
                          static int njdiv_, njdiv_ru_;
                          static int njold = 0, niold = 0;
                          static int nblockmax_, nsp_;
                          static size_t smemsizemax_;
                          static int firstcall_ = 1;
                          int bufidx_, ioff_, nisub_;
                          int ni_ru_ = ((n - 1 - 1) / nimax_ + 1) * nimax_;
                          hipError_t cuerr_;
                          if (firstcall_) {
                              int device_;
                              hipDeviceProp_t prop;
                              firstcall_ = 0;
                              cuerr_ = hipGetDevice(&device_);
                              cutilSafeCall(cuerr_);
                              cuerr_ = hipGetDeviceProperties(&prop, device_);
                              cutilSafeCall(cuerr_);
                              if (gprec_ >= kGoosePrecisionDouble) {
                                  if (prop.major < 2 && prop.minor < 3) {
                                      fprintf(stderr, "GPU architecture sm_%d%d does not support double-precision arithmetic.\n",
                                              prop.major, prop.minor);
                                      exit(1);
                                  }
                              }
                              smemsizemax_ = prop.sharedMemPerBlock;
                              if (ksmemsize_ >= smemsizemax_) {
                                  fprintf(stderr,
                                          "Shared memory consumption in f06_calculator (=%dbyte) "
                                          "reached the limit (=%dbyte). Too many jvars or fvars used.\n",
                                          ksmemsize_, smemsizemax_);
                                  exit(1);
                              }
                              if (rsmemsize_ >= smemsizemax_) {
                                  fprintf(stderr,
                                          "Shared memory consumption in f06_reducer (=%dbyte) "
                                          "reached the limit (=%dbyte). Too many fvars used.\n",
                                          rsmemsize_, smemsizemax_);
                                  exit(1);
                              }
                              nblockmax_ = prop.maxGridSize[0];
                              nsp_ = prop.multiProcessorCount * 8; // # of spream processors.
                              // fprintf(stderr, "nblockmax:%d  nsp:%d\n", nblockmax_, nsp_);
                          }
                          nisub_ = nimax_;
                          for (ioff_ = 1; ioff_ < n; ioff_ += nisub_) {
                              if (ioff_ + nisub_ > n) {
                                  nisub_ = n - ioff_;
                              }

                              int nisub_ru_ = ((nisub_ - 1) / nip_ + 1) * nip_;
                              if (njold != n || niold != nisub_) {
                                  // Adjust # of JP fragments so that large enough # of threads (to fill all SPs, 
                                  // and to hide latency of the global memory) are dispatched, at long as each
                                  // JP fragments has several times warp size.
                                  // You may want to hand tune the following part to obtain the optimal 'njdiv_'
                                  // value for a given hardware configuration.
                                  njdiv_ = njdiv_ru_ = 1;
                                  while (nisub_ru_ / nip_ * njdiv_ * 2 <= nblockmax_ &&
                                         nisub_ * njdiv_ < nsp_ * 200 &&
                                         n / njdiv_ > kbdim_ * 6 &&
                                         njdiv_ * 2 <= rbdim_) {
                                      njdiv_ *= 2;
                                  }
                                  // njdiv_ru_ is set to njdiv_ rounded up to a power of two.
                                  while (njdiv_ru_ < njdiv_) {
                                      njdiv_ru_ *= 2;
                                  }
                                  njold = n;
                                  niold = nisub_;
#if 0
                                  fprintf(stderr, "\n");
                                  fprintf(stderr, "nj:%d  njdiv:%d  nj/njdiv:%d  ni:%d  ni * njdiv:%d\n",
                                          n, njdiv_, n/njdiv_, nisub_, nisub_ * njdiv_);
                                  fprintf(stderr, "\n");
#endif
                              }
                              dim3 kthreads_(kbdim_, 1, 1);
                              dim3 kgrids_(njdiv_, nisub_ru_ / nip_, 1);
                              dim3 rthreads_(rbdim_, 1, 1);
                              int npipe_ = nisub_ru_ / 1;
                              int nrblock_ = npipe_ * njdiv_ / rbdim_;
                              dim3 rgrids_(nrblock_,1, 1);
                              int nj_ru_ = ((n - 1 - 1) / (kbdim_ * njdiv_) + 1) * (kbdim_ * njdiv_);
                              static int jbufsize_ = 0, ibufsize_ = 0, rsubbufsize_ = 0, rbufsize_ = 0;
                              static GlobalMem<f06_jp_t> f06_jp_;
                              static GlobalMem<f06_ip_t> f06_ip_;
                              static GlobalMem<f06_result_t> f06_result_;
                              static GlobalMem<f06_result_t> f06_result_sub_;
                              if (nj_ru_ > jbufsize_) {
                                  jbufsize_ = nj_ru_;
                                  f06_jp_.realloc(jbufsize_);
                              }

                              // Here we need to alloc n IPs. Note that npipe_ IPs would not be enough,
                              // since IPs on the device memory should not be overwritten during the ioff_ loop.
                              if (ni_ru_ > ibufsize_) {
                                  ibufsize_ = ni_ru_;
                                  f06_ip_.realloc(ibufsize_);
                              }
                              if (ni_ru_ > rbufsize_) {
                                  rbufsize_ = ni_ru_;
                                  f06_result_.realloc(rbufsize_);
                              }
                              if (npipe_ * njdiv_ > rsubbufsize_) {
                                  rsubbufsize_ = npipe_ * njdiv_ ;
                                  f06_result_sub_.realloc(rsubbufsize_);
                              }

                              for (j = 1, bufidx_ = 0 ; j <n; j++, bufidx_++) {f06_jp_[bufidx_].x_j_0_ = x[j][0];f06_jp_[bufidx_].x_j_1_ = x[j][1];f06_jp_[bufidx_].x_j_2_ = x[j][2];f06_jp_[bufidx_].m_j_ = m[j];f06_jp_[bufidx_].v_j_0_ = v[j][0];f06_jp_[bufidx_].v_j_1_ = v[j][1];f06_jp_[bufidx_].v_j_2_ = v[j][2];}f06_jp_.htod(0, bufidx_);

                              if (ioff_ < n) { // otherwise nothing to send.
                                  int nipsend_ = npipe_;
                                  if (ioff_ + nipsend_ >= n) {
                                      nipsend_ = n - ioff_;
                                  }
                                  for (i = 0 ; i < nipsend_; i++) {f06_ip_[ioff_ + i].x_i_0_ = x[ioff_ + i][0];f06_ip_[ioff_ + i].x_i_1_ = x[ioff_ + i][1];f06_ip_[ioff_ + i].x_i_2_ = x[ioff_ + i][2];f06_ip_[ioff_ + i].v_i_0_ = v[ioff_ + i][0];f06_ip_[ioff_ + i].v_i_1_ = v[ioff_ + i][1];f06_ip_[ioff_ + i].v_i_2_ = v[ioff_ + i][2];f06_ip_[ioff_ + i].m_i_ = m[ioff_ + i];}f06_ip_.htod(ioff_, nipsend_);

                              }
                              f06_calculator<<<kgrids_, kthreads_, ksmemsize_>>>(ioff_, nisub_, n - 1, (double)x[0][0], (double)x[0][1], (double)x[0][2], (double)eps2, (double)m[0], f06_jp_, (f06_ip_ + ioff_), f06_result_sub_);
                              f06_reducer<<<rgrids_, rthreads_, rsmemsize_>>>(njdiv_, njdiv_ru_, (f06_result_ + ioff_), f06_result_sub_);

                              if (ioff_ < n) { // otherwise nothing to receive.
                                  int niprecv_ = npipe_;
                                  if (ioff_ + niprecv_ >= n) {
                                      niprecv_ = n - ioff_;
                                  }
                                  f06_result_.dtoh(ioff_, niprecv_);
for (i = 0 ; i < niprecv_; i++) {pot_pn2[ioff_ + i] = f06_result_[ioff_ + i].pot_pn2_i_;}
                              }
                          } // end of ioff loop.
                      } // end of api calls.


  for (i=1;i<n;i++) {
     for (k=0;k<3;k++) {
       dxa[k]=x[i][k]-x[0][k];
     }

      r1a2e=dxa[0]*dxa[0]+dxa[1]*dxa[1]+dxa[2]*dxa[2]+eps2;
      r1ainv=rsqrt(r1a2e);
     vi2=v[i][0]*v[i][0]+v[i][1]*v[i][1]+v[i][2]*v[i][2];

     pot_pn2[i]-=0.25*m[i]*m[i]/eps*(-vi2)
                 +m[0]*m[i]*m[i]*(r1ainv/eps+0.5*r1ainv*r1ainv);
   }

  ene=kin_n+pot_n+pot_pn;

  for (i=1;i<n;i++) {
    ene+=pot_pn2[i];
  }

  printf("time = %g\n",0.0);
  printf("pot = %22.15e kin = %22.15e \n pot_pn = %22.15e \n  total= %22.15e ratio = %e\n",
  pot_n, kin_n, pot_pn, ene, kin_n/pot_n);

  *init_ene = ene;
}
void push_velocity(double v[270000][3],
                   double a[270000][3],
                   double dt,
                   int n)
{
  int j,k;
  for(j=0;j<n;j++){
    for(k=0;k<3;k++) v[j][k] += dt*a[j][k];
  }
}
void push_position(double x[270000][3],
                   double v[270000][3],
                   double dt,
                   int n)
{
  int j,k;
  for(j=0;j<n;j++){
    for(k=0;k<3;k++) x[j][k] += dt*v[j][k];
  }
}
main()
{
  static double x[270000][3];
  static double v[270000][3];
  static double m[270000];
  static double a[270000][3];
  static double ah[270000][3];
  static double pot[270000];
  static double hx[4][270000][3];
  static double hv[4][270000][3];
  static double x1[270000][3];
  static double v1[270000][3];
  double dt,eps,init_ene,time;
  double deouttime,eouttime,idtinv,endtime,epsinv;
  double icm[3],ratio;
  FILE *fp2;
  int n,i,k,dim,symp;
  double lt=0.0, st=0.0;
  double hlt=0.0, hst=0.0;
  double holdtime;
  double xsize=2.0,rotint=2.0,sustained=0.0;
  int simid;
  double gintrps;
  double peak;
  FILE *fpinput, *fpout;
  double rr;
  float xtmp,ytmp,ztmp,vxtmp,vytmp,vztmp;
  double cub2;
  time=0.0;
  eps=0.001;
  dt=1.0/131072.0;
  n=10001;
  endtime=10.0;
  deouttime=0.005;
  fpout = fopen("Sphere-r10-1PN.dat","w");
  fpinput = fopen("Init10k-sphere.dat","r");
  if (!fpinput) {
    perror("data_input");
    exit(1);
  }
  x[0][0]=0.0;
  x[0][1]=0.0;
  x[0][2]=0.0;
  v[0][0]=0.0;
  v[0][1]=0.0;
  v[0][2]=0.0;
  printf ("Init \n");
  for(i=1; i<n; i++) {
    fscanf (fpinput, "%f %f %f \n", &xtmp,&ytmp,&ztmp);
    x[i][0]=xtmp;
    x[i][1]=ytmp;
    x[i][2]=ztmp;
  }
  for(i=1; i<n; i++) {
    fscanf (fpinput, "%f %f %f \n", &vxtmp,&vytmp,&vztmp);
    v[i][0]=vxtmp;
    v[i][1]=vytmp;
    v[i][2]=vztmp;
  }
  fclose(fpinput);
  m[0] =100.0/(double)n;
  for (i=1; i<n; i++) {
    m[i] = 1.0/(double)n;
  }
  printf ("start \n");
  printf("initialdata end\n");
  for(i=0;i<n;i++) {
    fprintf(fpout,"%lf %lf %lf \n",x[i][0],x[i][1],x[i][2]);
  }
  for(i=0;i<n;i++) {
    fprintf(fpout,"%lf %lf %lf \n",v[i][0],v[i][1],v[i][2]);
  }
  eouttime= time+deouttime;
  initial_energy(x,v,m,n,&init_ene,eps);
  while(time < endtime){
    static int step = 0;
    for(i=0;i<n;i++) {
      for(k=0;k<3;k++) {
 hx[0][i][k]=x[i][k];
 hv[0][i][k]=v[i][k];
      }
    }
    force(x,v,m,eps,a,pot,n);
    for(i=0;i<n;i++) {
      for(k=0;k<3;k++) {
 x1[i][k]=x[i][k]+0.5*v[i][k]*dt;
 v1[i][k]=v[i][k]+0.5*a[i][k]*dt;
 hx[1][i][k]=x1[i][k];
 hv[1][i][k]=v1[i][k];
      }
    }
    force(x1,v1,m,eps,a,pot,n);
    for(i=0;i<n;i++) {
      for(k=0;k<3;k++) {
        x1[i][k]=x[i][k]+0.5*v1[i][k]*dt;
 v1[i][k]=v[i][k]+0.5*a[i][k]*dt;
 hx[2][i][k]=x1[i][k];
 hv[2][i][k]=v1[i][k];
      }
    }
    force(x1,v1,m,eps,a,pot,n);
    for(i=0;i<n;i++) {
      for(k=0;k<3;k++) {
        x1[i][k]=x[i][k]+v1[i][k]*dt;
 v1[i][k]=v[i][k]+a[i][k]*dt;
 hx[3][i][k]=x1[i][k];
 hv[3][i][k]=v1[i][k];
      }
    }
    for(i=0;i<n;i++) {
      for(k=0;k<3;k++) {
 x[i][k]=(hx[0][i][k]+hx[3][i][k]
   +2.0*(hx[1][i][k]+hx[2][i][k]))/6.0;
 v[i][k]=(hv[0][i][k]+hv[3][i][k]
                 +2.0*(hv[1][i][k]+hv[2][i][k]))/6.0;
      }
    }
    time += dt;
    if( time >= eouttime) {
      energy(time,x,v,m,n,init_ene,eps);
      eouttime += deouttime;
      for(i=0;i<n;i++) {
 fprintf(fpout,"%lf %lf %lf \n",x[i][0],x[i][1],x[i][2]);
      }
      for(i=0;i<n;i++) {
 fprintf(fpout,"%lf %lf %lf \n",v[i][0],v[i][1],v[i][2]);
      }
    }
  }
  for(i=0;i<n;i++) {
    fprintf(fpout,"%lf %lf %lf \n",x[i][0],x[i][1],x[i][2]);
  }
  for(i=0;i<n;i++) {
    fprintf(fpout,"%lf %lf %lf \n",v[i][0],v[i][1],v[i][2]);
  }
  fclose(fpout);
  printf("%lf \n", time);
}
#include "Limit-sticky9_0.cu"
#include "Limit-sticky9_1.cu"
#include "Limit-sticky9_2.cu"
#include "Limit-sticky9_3.cu"
#include "Limit-sticky9_4.cu"
#include "Limit-sticky9_5.cu"
#include "Limit-sticky9_6.cu"
