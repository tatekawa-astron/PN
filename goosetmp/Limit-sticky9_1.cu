#include "hip/hip_runtime.h"
               __global__ void
               f01_calculator(int ioff_, int ni_, int nj_, double x_0_0_, double x_0_1_, double x_0_2_, double eps2, double m_0_, f01_jp_t *f01_jp_, f01_ip_t *f01_ip_, f01_result_t *f01_result_)
               {
                   extern __shared__ char smembuf_[];
                   int kbdim_ = blockDim.x;
                   
                   f01_jp_t * f01_jp_smem_ = (f01_jp_t *)smembuf_;
                   
                   f01_result_t * f01_result_smem_ = (f01_result_t *)smembuf_;
                   double dxb_0_, dxc_0_, dxbc_0_, dxb_1_, dxc_1_, dxbc_1_, dxb_2_, dxc_2_, dxbc_2_, r1b2, r1b2e, r1be, r1c2, r1c2e, r1ce, rbc2e, rbce, mr1b3e;
                   double a1c_0_wcache_ = 0.0f;
double a1c_1_wcache_ = 0.0f;
double a1c_2_wcache_ = 0.0f;

                   int njdiv_ = gridDim.x;
                   int jbid_ = blockIdx.x;
                   int ibid_ = blockIdx.y;
                   int tid_ = threadIdx.x;
                   int npipe_ = (ni_ - 1) / 1 + 1;
                   int nvalidthread_ = npipe_ - kbdim_ * ibid_;
                   if (nvalidthread_ > kbdim_) {
                       nvalidthread_ = kbdim_;
                   }
                   int njsub_ = (nj_ - 1) / njdiv_ + 1;
                   int joff0_ = njsub_ * jbid_;
                   int joff1_ = joff0_ + njsub_;
                   if (joff1_ > nj_) {
                       joff1_ = nj_;
                   }
                   int jstride_ = 1;
                   if (nvalidthread_ <= kbdim_ / 2) jstride_ = 2;
if (nvalidthread_ <= kbdim_ / 4) jstride_ = 4;

                   int njhsub_ = kbdim_ / jstride_;
                   int jstart_ = tid_ / njhsub_;
                   int isrc_ = kbdim_ * ibid_ + tid_ % njhsub_;
                   int idst_ = njdiv_ * isrc_ + jbid_;
                   int icnt_ = ioff_ + isrc_;
                   for (int joff_ = joff0_; joff_ < joff1_; joff_ += kbdim_) {
                       int jsrc_ = tid_+joff_;
                       __syncthreads();
                       #if 1
                     {
                         float4 *srcbuf_ = (float4 *)(f01_jp_ + joff_);
                         float4 *dstbuf_ = (float4 *)smembuf_;
                         for (int icpy = 0; icpy < sizeof(f01_jp_t)/sizeof(float4); icpy++) {
                             dstbuf_[tid_] = srcbuf_[tid_];
                             dstbuf_ += kbdim_;
                             srcbuf_ += kbdim_;
                         }
                     }
#else
                     f01_jp_smem_[tid_] = f01_jp_[jsrc_];
#endif

                       __syncthreads();
                       int jsup_ = kbdim_;
                       if (joff_ + jsup_ > joff1_) {
                           jsup_ = joff1_ - joff_;
                       }
                       if (jsup_ < kbdim_) {
                           for (int j_ = jstart_; j_ < jsup_; j_+= jstride_) {
                               dxb_0_ = f01_ip_[isrc_].x_i_0_ - x_0_0_;
dxc_0_ = f01_jp_smem_[j_].x_j_0_ - x_0_0_;
dxbc_0_ = f01_jp_smem_[j_].x_j_0_ - f01_ip_[isrc_].x_i_0_;
dxb_1_ = f01_ip_[isrc_].x_i_1_ - x_0_1_;
dxc_1_ = f01_jp_smem_[j_].x_j_1_ - x_0_1_;
dxbc_1_ = f01_jp_smem_[j_].x_j_1_ - f01_ip_[isrc_].x_i_1_;
dxb_2_ = f01_ip_[isrc_].x_i_2_ - x_0_2_;
dxc_2_ = f01_jp_smem_[j_].x_j_2_ - x_0_2_;
dxbc_2_ = f01_jp_smem_[j_].x_j_2_ - f01_ip_[isrc_].x_i_2_;
r1b2 = dxb_0_ * dxb_0_ + dxb_1_ * dxb_1_ + dxb_2_ * dxb_2_;
r1b2e = r1b2 + eps2;
r1be = rsqrt(r1b2e);
r1c2 = dxc_0_ * dxc_0_ + dxc_1_ * dxc_1_ + dxc_2_ * dxc_2_;
r1c2e = r1c2 + eps2;
r1ce = rsqrt(r1c2e);
rbc2e = dxbc_0_ * dxbc_0_ + dxbc_1_ * dxbc_1_ + dxbc_2_ * dxbc_2_ + eps2;
rbce = rsqrt(rbc2e);
mr1b3e = f01_ip_[isrc_].m_i_ * r1be * r1be * r1be;
a1c_0_wcache_ += mr1b3e * f01_jp_smem_[j_].m_j_ * dxb_0_ * (4.0 * r1ce + 1.25 * rbce - 0.25 * r1c2 / rbc2e * rbce + 0.25 * r1b2 / rbc2e * rbce) - 3.5 * (rbce * rbce * rbce) * r1be * f01_ip_[isrc_].m_i_ * f01_jp_smem_[j_].m_j_ * dxbc_0_ - mr1b3e * f01_jp_smem_[j_].m_j_ / m_0_ * (4.0 * (f01_ip_[isrc_].v_i_0_ * f01_jp_smem_[j_].v_j_0_ + f01_ip_[isrc_].v_i_1_ * f01_jp_smem_[j_].v_j_1_ + f01_ip_[isrc_].v_i_2_ * f01_jp_smem_[j_].v_j_2_) * dxb_0_ - 3.0 * (f01_ip_[isrc_].v_i_0_ * dxb_0_ + f01_ip_[isrc_].v_i_1_ * dxb_1_ + f01_ip_[isrc_].v_i_2_ * dxb_2_) * f01_jp_smem_[j_].v_j_0_ - 4.0 * (f01_jp_smem_[j_].v_j_0_ * dxb_0_ + f01_jp_smem_[j_].v_j_1_ * dxb_1_ + f01_jp_smem_[j_].v_j_2_ * dxb_2_) * f01_ip_[isrc_].v_i_0_);
a1c_1_wcache_ += mr1b3e * f01_jp_smem_[j_].m_j_ * dxb_1_ * (4.0 * r1ce + 1.25 * rbce - 0.25 * r1c2 / rbc2e * rbce + 0.25 * r1b2 / rbc2e * rbce) - 3.5 * (rbce * rbce * rbce) * r1be * f01_ip_[isrc_].m_i_ * f01_jp_smem_[j_].m_j_ * dxbc_1_ - mr1b3e * f01_jp_smem_[j_].m_j_ / m_0_ * (4.0 * (f01_ip_[isrc_].v_i_0_ * f01_jp_smem_[j_].v_j_0_ + f01_ip_[isrc_].v_i_1_ * f01_jp_smem_[j_].v_j_1_ + f01_ip_[isrc_].v_i_2_ * f01_jp_smem_[j_].v_j_2_) * dxb_1_ - 3.0 * (f01_ip_[isrc_].v_i_0_ * dxb_0_ + f01_ip_[isrc_].v_i_1_ * dxb_1_ + f01_ip_[isrc_].v_i_2_ * dxb_2_) * f01_jp_smem_[j_].v_j_1_ - 4.0 * (f01_jp_smem_[j_].v_j_0_ * dxb_0_ + f01_jp_smem_[j_].v_j_1_ * dxb_1_ + f01_jp_smem_[j_].v_j_2_ * dxb_2_) * f01_ip_[isrc_].v_i_1_);
a1c_2_wcache_ += mr1b3e * f01_jp_smem_[j_].m_j_ * dxb_2_ * (4.0 * r1ce + 1.25 * rbce - 0.25 * r1c2 / rbc2e * rbce + 0.25 * r1b2 / rbc2e * rbce) - 3.5 * (rbce * rbce * rbce) * r1be * f01_ip_[isrc_].m_i_ * f01_jp_smem_[j_].m_j_ * dxbc_2_ - mr1b3e * f01_jp_smem_[j_].m_j_ / m_0_ * (4.0 * (f01_ip_[isrc_].v_i_0_ * f01_jp_smem_[j_].v_j_0_ + f01_ip_[isrc_].v_i_1_ * f01_jp_smem_[j_].v_j_1_ + f01_ip_[isrc_].v_i_2_ * f01_jp_smem_[j_].v_j_2_) * dxb_2_ - 3.0 * (f01_ip_[isrc_].v_i_0_ * dxb_0_ + f01_ip_[isrc_].v_i_1_ * dxb_1_ + f01_ip_[isrc_].v_i_2_ * dxb_2_) * f01_jp_smem_[j_].v_j_2_ - 4.0 * (f01_jp_smem_[j_].v_j_0_ * dxb_0_ + f01_jp_smem_[j_].v_j_1_ * dxb_1_ + f01_jp_smem_[j_].v_j_2_ * dxb_2_) * f01_ip_[isrc_].v_i_2_);

                           }
                       }
                       else {    
                           for (int j_ = jstart_; j_ < kbdim_; j_+= jstride_ * 8) {
                                       // loop 0
dxb_0_ = f01_ip_[isrc_].x_i_0_ - x_0_0_;
dxc_0_ = f01_jp_smem_[j_ + jstride_ * 0].x_j_0_ - x_0_0_;
dxbc_0_ = f01_jp_smem_[j_ + jstride_ * 0].x_j_0_ - f01_ip_[isrc_].x_i_0_;
dxb_1_ = f01_ip_[isrc_].x_i_1_ - x_0_1_;
dxc_1_ = f01_jp_smem_[j_ + jstride_ * 0].x_j_1_ - x_0_1_;
dxbc_1_ = f01_jp_smem_[j_ + jstride_ * 0].x_j_1_ - f01_ip_[isrc_].x_i_1_;
dxb_2_ = f01_ip_[isrc_].x_i_2_ - x_0_2_;
dxc_2_ = f01_jp_smem_[j_ + jstride_ * 0].x_j_2_ - x_0_2_;
dxbc_2_ = f01_jp_smem_[j_ + jstride_ * 0].x_j_2_ - f01_ip_[isrc_].x_i_2_;
r1b2 = dxb_0_ * dxb_0_ + dxb_1_ * dxb_1_ + dxb_2_ * dxb_2_;
r1b2e = r1b2 + eps2;
r1be = rsqrt(r1b2e);
r1c2 = dxc_0_ * dxc_0_ + dxc_1_ * dxc_1_ + dxc_2_ * dxc_2_;
r1c2e = r1c2 + eps2;
r1ce = rsqrt(r1c2e);
rbc2e = dxbc_0_ * dxbc_0_ + dxbc_1_ * dxbc_1_ + dxbc_2_ * dxbc_2_ + eps2;
rbce = rsqrt(rbc2e);
mr1b3e = f01_ip_[isrc_].m_i_ * r1be * r1be * r1be;
a1c_0_wcache_ += mr1b3e * f01_jp_smem_[j_ + jstride_ * 0].m_j_ * dxb_0_ * (4.0 * r1ce + 1.25 * rbce - 0.25 * r1c2 / rbc2e * rbce + 0.25 * r1b2 / rbc2e * rbce) - 3.5 * (rbce * rbce * rbce) * r1be * f01_ip_[isrc_].m_i_ * f01_jp_smem_[j_ + jstride_ * 0].m_j_ * dxbc_0_ - mr1b3e * f01_jp_smem_[j_ + jstride_ * 0].m_j_ / m_0_ * (4.0 * (f01_ip_[isrc_].v_i_0_ * f01_jp_smem_[j_ + jstride_ * 0].v_j_0_ + f01_ip_[isrc_].v_i_1_ * f01_jp_smem_[j_ + jstride_ * 0].v_j_1_ + f01_ip_[isrc_].v_i_2_ * f01_jp_smem_[j_ + jstride_ * 0].v_j_2_) * dxb_0_ - 3.0 * (f01_ip_[isrc_].v_i_0_ * dxb_0_ + f01_ip_[isrc_].v_i_1_ * dxb_1_ + f01_ip_[isrc_].v_i_2_ * dxb_2_) * f01_jp_smem_[j_ + jstride_ * 0].v_j_0_ - 4.0 * (f01_jp_smem_[j_ + jstride_ * 0].v_j_0_ * dxb_0_ + f01_jp_smem_[j_ + jstride_ * 0].v_j_1_ * dxb_1_ + f01_jp_smem_[j_ + jstride_ * 0].v_j_2_ * dxb_2_) * f01_ip_[isrc_].v_i_0_);
a1c_1_wcache_ += mr1b3e * f01_jp_smem_[j_ + jstride_ * 0].m_j_ * dxb_1_ * (4.0 * r1ce + 1.25 * rbce - 0.25 * r1c2 / rbc2e * rbce + 0.25 * r1b2 / rbc2e * rbce) - 3.5 * (rbce * rbce * rbce) * r1be * f01_ip_[isrc_].m_i_ * f01_jp_smem_[j_ + jstride_ * 0].m_j_ * dxbc_1_ - mr1b3e * f01_jp_smem_[j_ + jstride_ * 0].m_j_ / m_0_ * (4.0 * (f01_ip_[isrc_].v_i_0_ * f01_jp_smem_[j_ + jstride_ * 0].v_j_0_ + f01_ip_[isrc_].v_i_1_ * f01_jp_smem_[j_ + jstride_ * 0].v_j_1_ + f01_ip_[isrc_].v_i_2_ * f01_jp_smem_[j_ + jstride_ * 0].v_j_2_) * dxb_1_ - 3.0 * (f01_ip_[isrc_].v_i_0_ * dxb_0_ + f01_ip_[isrc_].v_i_1_ * dxb_1_ + f01_ip_[isrc_].v_i_2_ * dxb_2_) * f01_jp_smem_[j_ + jstride_ * 0].v_j_1_ - 4.0 * (f01_jp_smem_[j_ + jstride_ * 0].v_j_0_ * dxb_0_ + f01_jp_smem_[j_ + jstride_ * 0].v_j_1_ * dxb_1_ + f01_jp_smem_[j_ + jstride_ * 0].v_j_2_ * dxb_2_) * f01_ip_[isrc_].v_i_1_);
a1c_2_wcache_ += mr1b3e * f01_jp_smem_[j_ + jstride_ * 0].m_j_ * dxb_2_ * (4.0 * r1ce + 1.25 * rbce - 0.25 * r1c2 / rbc2e * rbce + 0.25 * r1b2 / rbc2e * rbce) - 3.5 * (rbce * rbce * rbce) * r1be * f01_ip_[isrc_].m_i_ * f01_jp_smem_[j_ + jstride_ * 0].m_j_ * dxbc_2_ - mr1b3e * f01_jp_smem_[j_ + jstride_ * 0].m_j_ / m_0_ * (4.0 * (f01_ip_[isrc_].v_i_0_ * f01_jp_smem_[j_ + jstride_ * 0].v_j_0_ + f01_ip_[isrc_].v_i_1_ * f01_jp_smem_[j_ + jstride_ * 0].v_j_1_ + f01_ip_[isrc_].v_i_2_ * f01_jp_smem_[j_ + jstride_ * 0].v_j_2_) * dxb_2_ - 3.0 * (f01_ip_[isrc_].v_i_0_ * dxb_0_ + f01_ip_[isrc_].v_i_1_ * dxb_1_ + f01_ip_[isrc_].v_i_2_ * dxb_2_) * f01_jp_smem_[j_ + jstride_ * 0].v_j_2_ - 4.0 * (f01_jp_smem_[j_ + jstride_ * 0].v_j_0_ * dxb_0_ + f01_jp_smem_[j_ + jstride_ * 0].v_j_1_ * dxb_1_ + f01_jp_smem_[j_ + jstride_ * 0].v_j_2_ * dxb_2_) * f01_ip_[isrc_].v_i_2_);
        // loop 1
dxb_0_ = f01_ip_[isrc_].x_i_0_ - x_0_0_;
dxc_0_ = f01_jp_smem_[j_ + jstride_ * 1].x_j_0_ - x_0_0_;
dxbc_0_ = f01_jp_smem_[j_ + jstride_ * 1].x_j_0_ - f01_ip_[isrc_].x_i_0_;
dxb_1_ = f01_ip_[isrc_].x_i_1_ - x_0_1_;
dxc_1_ = f01_jp_smem_[j_ + jstride_ * 1].x_j_1_ - x_0_1_;
dxbc_1_ = f01_jp_smem_[j_ + jstride_ * 1].x_j_1_ - f01_ip_[isrc_].x_i_1_;
dxb_2_ = f01_ip_[isrc_].x_i_2_ - x_0_2_;
dxc_2_ = f01_jp_smem_[j_ + jstride_ * 1].x_j_2_ - x_0_2_;
dxbc_2_ = f01_jp_smem_[j_ + jstride_ * 1].x_j_2_ - f01_ip_[isrc_].x_i_2_;
r1b2 = dxb_0_ * dxb_0_ + dxb_1_ * dxb_1_ + dxb_2_ * dxb_2_;
r1b2e = r1b2 + eps2;
r1be = rsqrt(r1b2e);
r1c2 = dxc_0_ * dxc_0_ + dxc_1_ * dxc_1_ + dxc_2_ * dxc_2_;
r1c2e = r1c2 + eps2;
r1ce = rsqrt(r1c2e);
rbc2e = dxbc_0_ * dxbc_0_ + dxbc_1_ * dxbc_1_ + dxbc_2_ * dxbc_2_ + eps2;
rbce = rsqrt(rbc2e);
mr1b3e = f01_ip_[isrc_].m_i_ * r1be * r1be * r1be;
a1c_0_wcache_ += mr1b3e * f01_jp_smem_[j_ + jstride_ * 1].m_j_ * dxb_0_ * (4.0 * r1ce + 1.25 * rbce - 0.25 * r1c2 / rbc2e * rbce + 0.25 * r1b2 / rbc2e * rbce) - 3.5 * (rbce * rbce * rbce) * r1be * f01_ip_[isrc_].m_i_ * f01_jp_smem_[j_ + jstride_ * 1].m_j_ * dxbc_0_ - mr1b3e * f01_jp_smem_[j_ + jstride_ * 1].m_j_ / m_0_ * (4.0 * (f01_ip_[isrc_].v_i_0_ * f01_jp_smem_[j_ + jstride_ * 1].v_j_0_ + f01_ip_[isrc_].v_i_1_ * f01_jp_smem_[j_ + jstride_ * 1].v_j_1_ + f01_ip_[isrc_].v_i_2_ * f01_jp_smem_[j_ + jstride_ * 1].v_j_2_) * dxb_0_ - 3.0 * (f01_ip_[isrc_].v_i_0_ * dxb_0_ + f01_ip_[isrc_].v_i_1_ * dxb_1_ + f01_ip_[isrc_].v_i_2_ * dxb_2_) * f01_jp_smem_[j_ + jstride_ * 1].v_j_0_ - 4.0 * (f01_jp_smem_[j_ + jstride_ * 1].v_j_0_ * dxb_0_ + f01_jp_smem_[j_ + jstride_ * 1].v_j_1_ * dxb_1_ + f01_jp_smem_[j_ + jstride_ * 1].v_j_2_ * dxb_2_) * f01_ip_[isrc_].v_i_0_);
a1c_1_wcache_ += mr1b3e * f01_jp_smem_[j_ + jstride_ * 1].m_j_ * dxb_1_ * (4.0 * r1ce + 1.25 * rbce - 0.25 * r1c2 / rbc2e * rbce + 0.25 * r1b2 / rbc2e * rbce) - 3.5 * (rbce * rbce * rbce) * r1be * f01_ip_[isrc_].m_i_ * f01_jp_smem_[j_ + jstride_ * 1].m_j_ * dxbc_1_ - mr1b3e * f01_jp_smem_[j_ + jstride_ * 1].m_j_ / m_0_ * (4.0 * (f01_ip_[isrc_].v_i_0_ * f01_jp_smem_[j_ + jstride_ * 1].v_j_0_ + f01_ip_[isrc_].v_i_1_ * f01_jp_smem_[j_ + jstride_ * 1].v_j_1_ + f01_ip_[isrc_].v_i_2_ * f01_jp_smem_[j_ + jstride_ * 1].v_j_2_) * dxb_1_ - 3.0 * (f01_ip_[isrc_].v_i_0_ * dxb_0_ + f01_ip_[isrc_].v_i_1_ * dxb_1_ + f01_ip_[isrc_].v_i_2_ * dxb_2_) * f01_jp_smem_[j_ + jstride_ * 1].v_j_1_ - 4.0 * (f01_jp_smem_[j_ + jstride_ * 1].v_j_0_ * dxb_0_ + f01_jp_smem_[j_ + jstride_ * 1].v_j_1_ * dxb_1_ + f01_jp_smem_[j_ + jstride_ * 1].v_j_2_ * dxb_2_) * f01_ip_[isrc_].v_i_1_);
a1c_2_wcache_ += mr1b3e * f01_jp_smem_[j_ + jstride_ * 1].m_j_ * dxb_2_ * (4.0 * r1ce + 1.25 * rbce - 0.25 * r1c2 / rbc2e * rbce + 0.25 * r1b2 / rbc2e * rbce) - 3.5 * (rbce * rbce * rbce) * r1be * f01_ip_[isrc_].m_i_ * f01_jp_smem_[j_ + jstride_ * 1].m_j_ * dxbc_2_ - mr1b3e * f01_jp_smem_[j_ + jstride_ * 1].m_j_ / m_0_ * (4.0 * (f01_ip_[isrc_].v_i_0_ * f01_jp_smem_[j_ + jstride_ * 1].v_j_0_ + f01_ip_[isrc_].v_i_1_ * f01_jp_smem_[j_ + jstride_ * 1].v_j_1_ + f01_ip_[isrc_].v_i_2_ * f01_jp_smem_[j_ + jstride_ * 1].v_j_2_) * dxb_2_ - 3.0 * (f01_ip_[isrc_].v_i_0_ * dxb_0_ + f01_ip_[isrc_].v_i_1_ * dxb_1_ + f01_ip_[isrc_].v_i_2_ * dxb_2_) * f01_jp_smem_[j_ + jstride_ * 1].v_j_2_ - 4.0 * (f01_jp_smem_[j_ + jstride_ * 1].v_j_0_ * dxb_0_ + f01_jp_smem_[j_ + jstride_ * 1].v_j_1_ * dxb_1_ + f01_jp_smem_[j_ + jstride_ * 1].v_j_2_ * dxb_2_) * f01_ip_[isrc_].v_i_2_);
        // loop 2
dxb_0_ = f01_ip_[isrc_].x_i_0_ - x_0_0_;
dxc_0_ = f01_jp_smem_[j_ + jstride_ * 2].x_j_0_ - x_0_0_;
dxbc_0_ = f01_jp_smem_[j_ + jstride_ * 2].x_j_0_ - f01_ip_[isrc_].x_i_0_;
dxb_1_ = f01_ip_[isrc_].x_i_1_ - x_0_1_;
dxc_1_ = f01_jp_smem_[j_ + jstride_ * 2].x_j_1_ - x_0_1_;
dxbc_1_ = f01_jp_smem_[j_ + jstride_ * 2].x_j_1_ - f01_ip_[isrc_].x_i_1_;
dxb_2_ = f01_ip_[isrc_].x_i_2_ - x_0_2_;
dxc_2_ = f01_jp_smem_[j_ + jstride_ * 2].x_j_2_ - x_0_2_;
dxbc_2_ = f01_jp_smem_[j_ + jstride_ * 2].x_j_2_ - f01_ip_[isrc_].x_i_2_;
r1b2 = dxb_0_ * dxb_0_ + dxb_1_ * dxb_1_ + dxb_2_ * dxb_2_;
r1b2e = r1b2 + eps2;
r1be = rsqrt(r1b2e);
r1c2 = dxc_0_ * dxc_0_ + dxc_1_ * dxc_1_ + dxc_2_ * dxc_2_;
r1c2e = r1c2 + eps2;
r1ce = rsqrt(r1c2e);
rbc2e = dxbc_0_ * dxbc_0_ + dxbc_1_ * dxbc_1_ + dxbc_2_ * dxbc_2_ + eps2;
rbce = rsqrt(rbc2e);
mr1b3e = f01_ip_[isrc_].m_i_ * r1be * r1be * r1be;
a1c_0_wcache_ += mr1b3e * f01_jp_smem_[j_ + jstride_ * 2].m_j_ * dxb_0_ * (4.0 * r1ce + 1.25 * rbce - 0.25 * r1c2 / rbc2e * rbce + 0.25 * r1b2 / rbc2e * rbce) - 3.5 * (rbce * rbce * rbce) * r1be * f01_ip_[isrc_].m_i_ * f01_jp_smem_[j_ + jstride_ * 2].m_j_ * dxbc_0_ - mr1b3e * f01_jp_smem_[j_ + jstride_ * 2].m_j_ / m_0_ * (4.0 * (f01_ip_[isrc_].v_i_0_ * f01_jp_smem_[j_ + jstride_ * 2].v_j_0_ + f01_ip_[isrc_].v_i_1_ * f01_jp_smem_[j_ + jstride_ * 2].v_j_1_ + f01_ip_[isrc_].v_i_2_ * f01_jp_smem_[j_ + jstride_ * 2].v_j_2_) * dxb_0_ - 3.0 * (f01_ip_[isrc_].v_i_0_ * dxb_0_ + f01_ip_[isrc_].v_i_1_ * dxb_1_ + f01_ip_[isrc_].v_i_2_ * dxb_2_) * f01_jp_smem_[j_ + jstride_ * 2].v_j_0_ - 4.0 * (f01_jp_smem_[j_ + jstride_ * 2].v_j_0_ * dxb_0_ + f01_jp_smem_[j_ + jstride_ * 2].v_j_1_ * dxb_1_ + f01_jp_smem_[j_ + jstride_ * 2].v_j_2_ * dxb_2_) * f01_ip_[isrc_].v_i_0_);
a1c_1_wcache_ += mr1b3e * f01_jp_smem_[j_ + jstride_ * 2].m_j_ * dxb_1_ * (4.0 * r1ce + 1.25 * rbce - 0.25 * r1c2 / rbc2e * rbce + 0.25 * r1b2 / rbc2e * rbce) - 3.5 * (rbce * rbce * rbce) * r1be * f01_ip_[isrc_].m_i_ * f01_jp_smem_[j_ + jstride_ * 2].m_j_ * dxbc_1_ - mr1b3e * f01_jp_smem_[j_ + jstride_ * 2].m_j_ / m_0_ * (4.0 * (f01_ip_[isrc_].v_i_0_ * f01_jp_smem_[j_ + jstride_ * 2].v_j_0_ + f01_ip_[isrc_].v_i_1_ * f01_jp_smem_[j_ + jstride_ * 2].v_j_1_ + f01_ip_[isrc_].v_i_2_ * f01_jp_smem_[j_ + jstride_ * 2].v_j_2_) * dxb_1_ - 3.0 * (f01_ip_[isrc_].v_i_0_ * dxb_0_ + f01_ip_[isrc_].v_i_1_ * dxb_1_ + f01_ip_[isrc_].v_i_2_ * dxb_2_) * f01_jp_smem_[j_ + jstride_ * 2].v_j_1_ - 4.0 * (f01_jp_smem_[j_ + jstride_ * 2].v_j_0_ * dxb_0_ + f01_jp_smem_[j_ + jstride_ * 2].v_j_1_ * dxb_1_ + f01_jp_smem_[j_ + jstride_ * 2].v_j_2_ * dxb_2_) * f01_ip_[isrc_].v_i_1_);
a1c_2_wcache_ += mr1b3e * f01_jp_smem_[j_ + jstride_ * 2].m_j_ * dxb_2_ * (4.0 * r1ce + 1.25 * rbce - 0.25 * r1c2 / rbc2e * rbce + 0.25 * r1b2 / rbc2e * rbce) - 3.5 * (rbce * rbce * rbce) * r1be * f01_ip_[isrc_].m_i_ * f01_jp_smem_[j_ + jstride_ * 2].m_j_ * dxbc_2_ - mr1b3e * f01_jp_smem_[j_ + jstride_ * 2].m_j_ / m_0_ * (4.0 * (f01_ip_[isrc_].v_i_0_ * f01_jp_smem_[j_ + jstride_ * 2].v_j_0_ + f01_ip_[isrc_].v_i_1_ * f01_jp_smem_[j_ + jstride_ * 2].v_j_1_ + f01_ip_[isrc_].v_i_2_ * f01_jp_smem_[j_ + jstride_ * 2].v_j_2_) * dxb_2_ - 3.0 * (f01_ip_[isrc_].v_i_0_ * dxb_0_ + f01_ip_[isrc_].v_i_1_ * dxb_1_ + f01_ip_[isrc_].v_i_2_ * dxb_2_) * f01_jp_smem_[j_ + jstride_ * 2].v_j_2_ - 4.0 * (f01_jp_smem_[j_ + jstride_ * 2].v_j_0_ * dxb_0_ + f01_jp_smem_[j_ + jstride_ * 2].v_j_1_ * dxb_1_ + f01_jp_smem_[j_ + jstride_ * 2].v_j_2_ * dxb_2_) * f01_ip_[isrc_].v_i_2_);
        // loop 3
dxb_0_ = f01_ip_[isrc_].x_i_0_ - x_0_0_;
dxc_0_ = f01_jp_smem_[j_ + jstride_ * 3].x_j_0_ - x_0_0_;
dxbc_0_ = f01_jp_smem_[j_ + jstride_ * 3].x_j_0_ - f01_ip_[isrc_].x_i_0_;
dxb_1_ = f01_ip_[isrc_].x_i_1_ - x_0_1_;
dxc_1_ = f01_jp_smem_[j_ + jstride_ * 3].x_j_1_ - x_0_1_;
dxbc_1_ = f01_jp_smem_[j_ + jstride_ * 3].x_j_1_ - f01_ip_[isrc_].x_i_1_;
dxb_2_ = f01_ip_[isrc_].x_i_2_ - x_0_2_;
dxc_2_ = f01_jp_smem_[j_ + jstride_ * 3].x_j_2_ - x_0_2_;
dxbc_2_ = f01_jp_smem_[j_ + jstride_ * 3].x_j_2_ - f01_ip_[isrc_].x_i_2_;
r1b2 = dxb_0_ * dxb_0_ + dxb_1_ * dxb_1_ + dxb_2_ * dxb_2_;
r1b2e = r1b2 + eps2;
r1be = rsqrt(r1b2e);
r1c2 = dxc_0_ * dxc_0_ + dxc_1_ * dxc_1_ + dxc_2_ * dxc_2_;
r1c2e = r1c2 + eps2;
r1ce = rsqrt(r1c2e);
rbc2e = dxbc_0_ * dxbc_0_ + dxbc_1_ * dxbc_1_ + dxbc_2_ * dxbc_2_ + eps2;
rbce = rsqrt(rbc2e);
mr1b3e = f01_ip_[isrc_].m_i_ * r1be * r1be * r1be;
a1c_0_wcache_ += mr1b3e * f01_jp_smem_[j_ + jstride_ * 3].m_j_ * dxb_0_ * (4.0 * r1ce + 1.25 * rbce - 0.25 * r1c2 / rbc2e * rbce + 0.25 * r1b2 / rbc2e * rbce) - 3.5 * (rbce * rbce * rbce) * r1be * f01_ip_[isrc_].m_i_ * f01_jp_smem_[j_ + jstride_ * 3].m_j_ * dxbc_0_ - mr1b3e * f01_jp_smem_[j_ + jstride_ * 3].m_j_ / m_0_ * (4.0 * (f01_ip_[isrc_].v_i_0_ * f01_jp_smem_[j_ + jstride_ * 3].v_j_0_ + f01_ip_[isrc_].v_i_1_ * f01_jp_smem_[j_ + jstride_ * 3].v_j_1_ + f01_ip_[isrc_].v_i_2_ * f01_jp_smem_[j_ + jstride_ * 3].v_j_2_) * dxb_0_ - 3.0 * (f01_ip_[isrc_].v_i_0_ * dxb_0_ + f01_ip_[isrc_].v_i_1_ * dxb_1_ + f01_ip_[isrc_].v_i_2_ * dxb_2_) * f01_jp_smem_[j_ + jstride_ * 3].v_j_0_ - 4.0 * (f01_jp_smem_[j_ + jstride_ * 3].v_j_0_ * dxb_0_ + f01_jp_smem_[j_ + jstride_ * 3].v_j_1_ * dxb_1_ + f01_jp_smem_[j_ + jstride_ * 3].v_j_2_ * dxb_2_) * f01_ip_[isrc_].v_i_0_);
a1c_1_wcache_ += mr1b3e * f01_jp_smem_[j_ + jstride_ * 3].m_j_ * dxb_1_ * (4.0 * r1ce + 1.25 * rbce - 0.25 * r1c2 / rbc2e * rbce + 0.25 * r1b2 / rbc2e * rbce) - 3.5 * (rbce * rbce * rbce) * r1be * f01_ip_[isrc_].m_i_ * f01_jp_smem_[j_ + jstride_ * 3].m_j_ * dxbc_1_ - mr1b3e * f01_jp_smem_[j_ + jstride_ * 3].m_j_ / m_0_ * (4.0 * (f01_ip_[isrc_].v_i_0_ * f01_jp_smem_[j_ + jstride_ * 3].v_j_0_ + f01_ip_[isrc_].v_i_1_ * f01_jp_smem_[j_ + jstride_ * 3].v_j_1_ + f01_ip_[isrc_].v_i_2_ * f01_jp_smem_[j_ + jstride_ * 3].v_j_2_) * dxb_1_ - 3.0 * (f01_ip_[isrc_].v_i_0_ * dxb_0_ + f01_ip_[isrc_].v_i_1_ * dxb_1_ + f01_ip_[isrc_].v_i_2_ * dxb_2_) * f01_jp_smem_[j_ + jstride_ * 3].v_j_1_ - 4.0 * (f01_jp_smem_[j_ + jstride_ * 3].v_j_0_ * dxb_0_ + f01_jp_smem_[j_ + jstride_ * 3].v_j_1_ * dxb_1_ + f01_jp_smem_[j_ + jstride_ * 3].v_j_2_ * dxb_2_) * f01_ip_[isrc_].v_i_1_);
a1c_2_wcache_ += mr1b3e * f01_jp_smem_[j_ + jstride_ * 3].m_j_ * dxb_2_ * (4.0 * r1ce + 1.25 * rbce - 0.25 * r1c2 / rbc2e * rbce + 0.25 * r1b2 / rbc2e * rbce) - 3.5 * (rbce * rbce * rbce) * r1be * f01_ip_[isrc_].m_i_ * f01_jp_smem_[j_ + jstride_ * 3].m_j_ * dxbc_2_ - mr1b3e * f01_jp_smem_[j_ + jstride_ * 3].m_j_ / m_0_ * (4.0 * (f01_ip_[isrc_].v_i_0_ * f01_jp_smem_[j_ + jstride_ * 3].v_j_0_ + f01_ip_[isrc_].v_i_1_ * f01_jp_smem_[j_ + jstride_ * 3].v_j_1_ + f01_ip_[isrc_].v_i_2_ * f01_jp_smem_[j_ + jstride_ * 3].v_j_2_) * dxb_2_ - 3.0 * (f01_ip_[isrc_].v_i_0_ * dxb_0_ + f01_ip_[isrc_].v_i_1_ * dxb_1_ + f01_ip_[isrc_].v_i_2_ * dxb_2_) * f01_jp_smem_[j_ + jstride_ * 3].v_j_2_ - 4.0 * (f01_jp_smem_[j_ + jstride_ * 3].v_j_0_ * dxb_0_ + f01_jp_smem_[j_ + jstride_ * 3].v_j_1_ * dxb_1_ + f01_jp_smem_[j_ + jstride_ * 3].v_j_2_ * dxb_2_) * f01_ip_[isrc_].v_i_2_);
        // loop 4
dxb_0_ = f01_ip_[isrc_].x_i_0_ - x_0_0_;
dxc_0_ = f01_jp_smem_[j_ + jstride_ * 4].x_j_0_ - x_0_0_;
dxbc_0_ = f01_jp_smem_[j_ + jstride_ * 4].x_j_0_ - f01_ip_[isrc_].x_i_0_;
dxb_1_ = f01_ip_[isrc_].x_i_1_ - x_0_1_;
dxc_1_ = f01_jp_smem_[j_ + jstride_ * 4].x_j_1_ - x_0_1_;
dxbc_1_ = f01_jp_smem_[j_ + jstride_ * 4].x_j_1_ - f01_ip_[isrc_].x_i_1_;
dxb_2_ = f01_ip_[isrc_].x_i_2_ - x_0_2_;
dxc_2_ = f01_jp_smem_[j_ + jstride_ * 4].x_j_2_ - x_0_2_;
dxbc_2_ = f01_jp_smem_[j_ + jstride_ * 4].x_j_2_ - f01_ip_[isrc_].x_i_2_;
r1b2 = dxb_0_ * dxb_0_ + dxb_1_ * dxb_1_ + dxb_2_ * dxb_2_;
r1b2e = r1b2 + eps2;
r1be = rsqrt(r1b2e);
r1c2 = dxc_0_ * dxc_0_ + dxc_1_ * dxc_1_ + dxc_2_ * dxc_2_;
r1c2e = r1c2 + eps2;
r1ce = rsqrt(r1c2e);
rbc2e = dxbc_0_ * dxbc_0_ + dxbc_1_ * dxbc_1_ + dxbc_2_ * dxbc_2_ + eps2;
rbce = rsqrt(rbc2e);
mr1b3e = f01_ip_[isrc_].m_i_ * r1be * r1be * r1be;
a1c_0_wcache_ += mr1b3e * f01_jp_smem_[j_ + jstride_ * 4].m_j_ * dxb_0_ * (4.0 * r1ce + 1.25 * rbce - 0.25 * r1c2 / rbc2e * rbce + 0.25 * r1b2 / rbc2e * rbce) - 3.5 * (rbce * rbce * rbce) * r1be * f01_ip_[isrc_].m_i_ * f01_jp_smem_[j_ + jstride_ * 4].m_j_ * dxbc_0_ - mr1b3e * f01_jp_smem_[j_ + jstride_ * 4].m_j_ / m_0_ * (4.0 * (f01_ip_[isrc_].v_i_0_ * f01_jp_smem_[j_ + jstride_ * 4].v_j_0_ + f01_ip_[isrc_].v_i_1_ * f01_jp_smem_[j_ + jstride_ * 4].v_j_1_ + f01_ip_[isrc_].v_i_2_ * f01_jp_smem_[j_ + jstride_ * 4].v_j_2_) * dxb_0_ - 3.0 * (f01_ip_[isrc_].v_i_0_ * dxb_0_ + f01_ip_[isrc_].v_i_1_ * dxb_1_ + f01_ip_[isrc_].v_i_2_ * dxb_2_) * f01_jp_smem_[j_ + jstride_ * 4].v_j_0_ - 4.0 * (f01_jp_smem_[j_ + jstride_ * 4].v_j_0_ * dxb_0_ + f01_jp_smem_[j_ + jstride_ * 4].v_j_1_ * dxb_1_ + f01_jp_smem_[j_ + jstride_ * 4].v_j_2_ * dxb_2_) * f01_ip_[isrc_].v_i_0_);
a1c_1_wcache_ += mr1b3e * f01_jp_smem_[j_ + jstride_ * 4].m_j_ * dxb_1_ * (4.0 * r1ce + 1.25 * rbce - 0.25 * r1c2 / rbc2e * rbce + 0.25 * r1b2 / rbc2e * rbce) - 3.5 * (rbce * rbce * rbce) * r1be * f01_ip_[isrc_].m_i_ * f01_jp_smem_[j_ + jstride_ * 4].m_j_ * dxbc_1_ - mr1b3e * f01_jp_smem_[j_ + jstride_ * 4].m_j_ / m_0_ * (4.0 * (f01_ip_[isrc_].v_i_0_ * f01_jp_smem_[j_ + jstride_ * 4].v_j_0_ + f01_ip_[isrc_].v_i_1_ * f01_jp_smem_[j_ + jstride_ * 4].v_j_1_ + f01_ip_[isrc_].v_i_2_ * f01_jp_smem_[j_ + jstride_ * 4].v_j_2_) * dxb_1_ - 3.0 * (f01_ip_[isrc_].v_i_0_ * dxb_0_ + f01_ip_[isrc_].v_i_1_ * dxb_1_ + f01_ip_[isrc_].v_i_2_ * dxb_2_) * f01_jp_smem_[j_ + jstride_ * 4].v_j_1_ - 4.0 * (f01_jp_smem_[j_ + jstride_ * 4].v_j_0_ * dxb_0_ + f01_jp_smem_[j_ + jstride_ * 4].v_j_1_ * dxb_1_ + f01_jp_smem_[j_ + jstride_ * 4].v_j_2_ * dxb_2_) * f01_ip_[isrc_].v_i_1_);
a1c_2_wcache_ += mr1b3e * f01_jp_smem_[j_ + jstride_ * 4].m_j_ * dxb_2_ * (4.0 * r1ce + 1.25 * rbce - 0.25 * r1c2 / rbc2e * rbce + 0.25 * r1b2 / rbc2e * rbce) - 3.5 * (rbce * rbce * rbce) * r1be * f01_ip_[isrc_].m_i_ * f01_jp_smem_[j_ + jstride_ * 4].m_j_ * dxbc_2_ - mr1b3e * f01_jp_smem_[j_ + jstride_ * 4].m_j_ / m_0_ * (4.0 * (f01_ip_[isrc_].v_i_0_ * f01_jp_smem_[j_ + jstride_ * 4].v_j_0_ + f01_ip_[isrc_].v_i_1_ * f01_jp_smem_[j_ + jstride_ * 4].v_j_1_ + f01_ip_[isrc_].v_i_2_ * f01_jp_smem_[j_ + jstride_ * 4].v_j_2_) * dxb_2_ - 3.0 * (f01_ip_[isrc_].v_i_0_ * dxb_0_ + f01_ip_[isrc_].v_i_1_ * dxb_1_ + f01_ip_[isrc_].v_i_2_ * dxb_2_) * f01_jp_smem_[j_ + jstride_ * 4].v_j_2_ - 4.0 * (f01_jp_smem_[j_ + jstride_ * 4].v_j_0_ * dxb_0_ + f01_jp_smem_[j_ + jstride_ * 4].v_j_1_ * dxb_1_ + f01_jp_smem_[j_ + jstride_ * 4].v_j_2_ * dxb_2_) * f01_ip_[isrc_].v_i_2_);
        // loop 5
dxb_0_ = f01_ip_[isrc_].x_i_0_ - x_0_0_;
dxc_0_ = f01_jp_smem_[j_ + jstride_ * 5].x_j_0_ - x_0_0_;
dxbc_0_ = f01_jp_smem_[j_ + jstride_ * 5].x_j_0_ - f01_ip_[isrc_].x_i_0_;
dxb_1_ = f01_ip_[isrc_].x_i_1_ - x_0_1_;
dxc_1_ = f01_jp_smem_[j_ + jstride_ * 5].x_j_1_ - x_0_1_;
dxbc_1_ = f01_jp_smem_[j_ + jstride_ * 5].x_j_1_ - f01_ip_[isrc_].x_i_1_;
dxb_2_ = f01_ip_[isrc_].x_i_2_ - x_0_2_;
dxc_2_ = f01_jp_smem_[j_ + jstride_ * 5].x_j_2_ - x_0_2_;
dxbc_2_ = f01_jp_smem_[j_ + jstride_ * 5].x_j_2_ - f01_ip_[isrc_].x_i_2_;
r1b2 = dxb_0_ * dxb_0_ + dxb_1_ * dxb_1_ + dxb_2_ * dxb_2_;
r1b2e = r1b2 + eps2;
r1be = rsqrt(r1b2e);
r1c2 = dxc_0_ * dxc_0_ + dxc_1_ * dxc_1_ + dxc_2_ * dxc_2_;
r1c2e = r1c2 + eps2;
r1ce = rsqrt(r1c2e);
rbc2e = dxbc_0_ * dxbc_0_ + dxbc_1_ * dxbc_1_ + dxbc_2_ * dxbc_2_ + eps2;
rbce = rsqrt(rbc2e);
mr1b3e = f01_ip_[isrc_].m_i_ * r1be * r1be * r1be;
a1c_0_wcache_ += mr1b3e * f01_jp_smem_[j_ + jstride_ * 5].m_j_ * dxb_0_ * (4.0 * r1ce + 1.25 * rbce - 0.25 * r1c2 / rbc2e * rbce + 0.25 * r1b2 / rbc2e * rbce) - 3.5 * (rbce * rbce * rbce) * r1be * f01_ip_[isrc_].m_i_ * f01_jp_smem_[j_ + jstride_ * 5].m_j_ * dxbc_0_ - mr1b3e * f01_jp_smem_[j_ + jstride_ * 5].m_j_ / m_0_ * (4.0 * (f01_ip_[isrc_].v_i_0_ * f01_jp_smem_[j_ + jstride_ * 5].v_j_0_ + f01_ip_[isrc_].v_i_1_ * f01_jp_smem_[j_ + jstride_ * 5].v_j_1_ + f01_ip_[isrc_].v_i_2_ * f01_jp_smem_[j_ + jstride_ * 5].v_j_2_) * dxb_0_ - 3.0 * (f01_ip_[isrc_].v_i_0_ * dxb_0_ + f01_ip_[isrc_].v_i_1_ * dxb_1_ + f01_ip_[isrc_].v_i_2_ * dxb_2_) * f01_jp_smem_[j_ + jstride_ * 5].v_j_0_ - 4.0 * (f01_jp_smem_[j_ + jstride_ * 5].v_j_0_ * dxb_0_ + f01_jp_smem_[j_ + jstride_ * 5].v_j_1_ * dxb_1_ + f01_jp_smem_[j_ + jstride_ * 5].v_j_2_ * dxb_2_) * f01_ip_[isrc_].v_i_0_);
a1c_1_wcache_ += mr1b3e * f01_jp_smem_[j_ + jstride_ * 5].m_j_ * dxb_1_ * (4.0 * r1ce + 1.25 * rbce - 0.25 * r1c2 / rbc2e * rbce + 0.25 * r1b2 / rbc2e * rbce) - 3.5 * (rbce * rbce * rbce) * r1be * f01_ip_[isrc_].m_i_ * f01_jp_smem_[j_ + jstride_ * 5].m_j_ * dxbc_1_ - mr1b3e * f01_jp_smem_[j_ + jstride_ * 5].m_j_ / m_0_ * (4.0 * (f01_ip_[isrc_].v_i_0_ * f01_jp_smem_[j_ + jstride_ * 5].v_j_0_ + f01_ip_[isrc_].v_i_1_ * f01_jp_smem_[j_ + jstride_ * 5].v_j_1_ + f01_ip_[isrc_].v_i_2_ * f01_jp_smem_[j_ + jstride_ * 5].v_j_2_) * dxb_1_ - 3.0 * (f01_ip_[isrc_].v_i_0_ * dxb_0_ + f01_ip_[isrc_].v_i_1_ * dxb_1_ + f01_ip_[isrc_].v_i_2_ * dxb_2_) * f01_jp_smem_[j_ + jstride_ * 5].v_j_1_ - 4.0 * (f01_jp_smem_[j_ + jstride_ * 5].v_j_0_ * dxb_0_ + f01_jp_smem_[j_ + jstride_ * 5].v_j_1_ * dxb_1_ + f01_jp_smem_[j_ + jstride_ * 5].v_j_2_ * dxb_2_) * f01_ip_[isrc_].v_i_1_);
a1c_2_wcache_ += mr1b3e * f01_jp_smem_[j_ + jstride_ * 5].m_j_ * dxb_2_ * (4.0 * r1ce + 1.25 * rbce - 0.25 * r1c2 / rbc2e * rbce + 0.25 * r1b2 / rbc2e * rbce) - 3.5 * (rbce * rbce * rbce) * r1be * f01_ip_[isrc_].m_i_ * f01_jp_smem_[j_ + jstride_ * 5].m_j_ * dxbc_2_ - mr1b3e * f01_jp_smem_[j_ + jstride_ * 5].m_j_ / m_0_ * (4.0 * (f01_ip_[isrc_].v_i_0_ * f01_jp_smem_[j_ + jstride_ * 5].v_j_0_ + f01_ip_[isrc_].v_i_1_ * f01_jp_smem_[j_ + jstride_ * 5].v_j_1_ + f01_ip_[isrc_].v_i_2_ * f01_jp_smem_[j_ + jstride_ * 5].v_j_2_) * dxb_2_ - 3.0 * (f01_ip_[isrc_].v_i_0_ * dxb_0_ + f01_ip_[isrc_].v_i_1_ * dxb_1_ + f01_ip_[isrc_].v_i_2_ * dxb_2_) * f01_jp_smem_[j_ + jstride_ * 5].v_j_2_ - 4.0 * (f01_jp_smem_[j_ + jstride_ * 5].v_j_0_ * dxb_0_ + f01_jp_smem_[j_ + jstride_ * 5].v_j_1_ * dxb_1_ + f01_jp_smem_[j_ + jstride_ * 5].v_j_2_ * dxb_2_) * f01_ip_[isrc_].v_i_2_);
        // loop 6
dxb_0_ = f01_ip_[isrc_].x_i_0_ - x_0_0_;
dxc_0_ = f01_jp_smem_[j_ + jstride_ * 6].x_j_0_ - x_0_0_;
dxbc_0_ = f01_jp_smem_[j_ + jstride_ * 6].x_j_0_ - f01_ip_[isrc_].x_i_0_;
dxb_1_ = f01_ip_[isrc_].x_i_1_ - x_0_1_;
dxc_1_ = f01_jp_smem_[j_ + jstride_ * 6].x_j_1_ - x_0_1_;
dxbc_1_ = f01_jp_smem_[j_ + jstride_ * 6].x_j_1_ - f01_ip_[isrc_].x_i_1_;
dxb_2_ = f01_ip_[isrc_].x_i_2_ - x_0_2_;
dxc_2_ = f01_jp_smem_[j_ + jstride_ * 6].x_j_2_ - x_0_2_;
dxbc_2_ = f01_jp_smem_[j_ + jstride_ * 6].x_j_2_ - f01_ip_[isrc_].x_i_2_;
r1b2 = dxb_0_ * dxb_0_ + dxb_1_ * dxb_1_ + dxb_2_ * dxb_2_;
r1b2e = r1b2 + eps2;
r1be = rsqrt(r1b2e);
r1c2 = dxc_0_ * dxc_0_ + dxc_1_ * dxc_1_ + dxc_2_ * dxc_2_;
r1c2e = r1c2 + eps2;
r1ce = rsqrt(r1c2e);
rbc2e = dxbc_0_ * dxbc_0_ + dxbc_1_ * dxbc_1_ + dxbc_2_ * dxbc_2_ + eps2;
rbce = rsqrt(rbc2e);
mr1b3e = f01_ip_[isrc_].m_i_ * r1be * r1be * r1be;
a1c_0_wcache_ += mr1b3e * f01_jp_smem_[j_ + jstride_ * 6].m_j_ * dxb_0_ * (4.0 * r1ce + 1.25 * rbce - 0.25 * r1c2 / rbc2e * rbce + 0.25 * r1b2 / rbc2e * rbce) - 3.5 * (rbce * rbce * rbce) * r1be * f01_ip_[isrc_].m_i_ * f01_jp_smem_[j_ + jstride_ * 6].m_j_ * dxbc_0_ - mr1b3e * f01_jp_smem_[j_ + jstride_ * 6].m_j_ / m_0_ * (4.0 * (f01_ip_[isrc_].v_i_0_ * f01_jp_smem_[j_ + jstride_ * 6].v_j_0_ + f01_ip_[isrc_].v_i_1_ * f01_jp_smem_[j_ + jstride_ * 6].v_j_1_ + f01_ip_[isrc_].v_i_2_ * f01_jp_smem_[j_ + jstride_ * 6].v_j_2_) * dxb_0_ - 3.0 * (f01_ip_[isrc_].v_i_0_ * dxb_0_ + f01_ip_[isrc_].v_i_1_ * dxb_1_ + f01_ip_[isrc_].v_i_2_ * dxb_2_) * f01_jp_smem_[j_ + jstride_ * 6].v_j_0_ - 4.0 * (f01_jp_smem_[j_ + jstride_ * 6].v_j_0_ * dxb_0_ + f01_jp_smem_[j_ + jstride_ * 6].v_j_1_ * dxb_1_ + f01_jp_smem_[j_ + jstride_ * 6].v_j_2_ * dxb_2_) * f01_ip_[isrc_].v_i_0_);
a1c_1_wcache_ += mr1b3e * f01_jp_smem_[j_ + jstride_ * 6].m_j_ * dxb_1_ * (4.0 * r1ce + 1.25 * rbce - 0.25 * r1c2 / rbc2e * rbce + 0.25 * r1b2 / rbc2e * rbce) - 3.5 * (rbce * rbce * rbce) * r1be * f01_ip_[isrc_].m_i_ * f01_jp_smem_[j_ + jstride_ * 6].m_j_ * dxbc_1_ - mr1b3e * f01_jp_smem_[j_ + jstride_ * 6].m_j_ / m_0_ * (4.0 * (f01_ip_[isrc_].v_i_0_ * f01_jp_smem_[j_ + jstride_ * 6].v_j_0_ + f01_ip_[isrc_].v_i_1_ * f01_jp_smem_[j_ + jstride_ * 6].v_j_1_ + f01_ip_[isrc_].v_i_2_ * f01_jp_smem_[j_ + jstride_ * 6].v_j_2_) * dxb_1_ - 3.0 * (f01_ip_[isrc_].v_i_0_ * dxb_0_ + f01_ip_[isrc_].v_i_1_ * dxb_1_ + f01_ip_[isrc_].v_i_2_ * dxb_2_) * f01_jp_smem_[j_ + jstride_ * 6].v_j_1_ - 4.0 * (f01_jp_smem_[j_ + jstride_ * 6].v_j_0_ * dxb_0_ + f01_jp_smem_[j_ + jstride_ * 6].v_j_1_ * dxb_1_ + f01_jp_smem_[j_ + jstride_ * 6].v_j_2_ * dxb_2_) * f01_ip_[isrc_].v_i_1_);
a1c_2_wcache_ += mr1b3e * f01_jp_smem_[j_ + jstride_ * 6].m_j_ * dxb_2_ * (4.0 * r1ce + 1.25 * rbce - 0.25 * r1c2 / rbc2e * rbce + 0.25 * r1b2 / rbc2e * rbce) - 3.5 * (rbce * rbce * rbce) * r1be * f01_ip_[isrc_].m_i_ * f01_jp_smem_[j_ + jstride_ * 6].m_j_ * dxbc_2_ - mr1b3e * f01_jp_smem_[j_ + jstride_ * 6].m_j_ / m_0_ * (4.0 * (f01_ip_[isrc_].v_i_0_ * f01_jp_smem_[j_ + jstride_ * 6].v_j_0_ + f01_ip_[isrc_].v_i_1_ * f01_jp_smem_[j_ + jstride_ * 6].v_j_1_ + f01_ip_[isrc_].v_i_2_ * f01_jp_smem_[j_ + jstride_ * 6].v_j_2_) * dxb_2_ - 3.0 * (f01_ip_[isrc_].v_i_0_ * dxb_0_ + f01_ip_[isrc_].v_i_1_ * dxb_1_ + f01_ip_[isrc_].v_i_2_ * dxb_2_) * f01_jp_smem_[j_ + jstride_ * 6].v_j_2_ - 4.0 * (f01_jp_smem_[j_ + jstride_ * 6].v_j_0_ * dxb_0_ + f01_jp_smem_[j_ + jstride_ * 6].v_j_1_ * dxb_1_ + f01_jp_smem_[j_ + jstride_ * 6].v_j_2_ * dxb_2_) * f01_ip_[isrc_].v_i_2_);
        // loop 7
dxb_0_ = f01_ip_[isrc_].x_i_0_ - x_0_0_;
dxc_0_ = f01_jp_smem_[j_ + jstride_ * 7].x_j_0_ - x_0_0_;
dxbc_0_ = f01_jp_smem_[j_ + jstride_ * 7].x_j_0_ - f01_ip_[isrc_].x_i_0_;
dxb_1_ = f01_ip_[isrc_].x_i_1_ - x_0_1_;
dxc_1_ = f01_jp_smem_[j_ + jstride_ * 7].x_j_1_ - x_0_1_;
dxbc_1_ = f01_jp_smem_[j_ + jstride_ * 7].x_j_1_ - f01_ip_[isrc_].x_i_1_;
dxb_2_ = f01_ip_[isrc_].x_i_2_ - x_0_2_;
dxc_2_ = f01_jp_smem_[j_ + jstride_ * 7].x_j_2_ - x_0_2_;
dxbc_2_ = f01_jp_smem_[j_ + jstride_ * 7].x_j_2_ - f01_ip_[isrc_].x_i_2_;
r1b2 = dxb_0_ * dxb_0_ + dxb_1_ * dxb_1_ + dxb_2_ * dxb_2_;
r1b2e = r1b2 + eps2;
r1be = rsqrt(r1b2e);
r1c2 = dxc_0_ * dxc_0_ + dxc_1_ * dxc_1_ + dxc_2_ * dxc_2_;
r1c2e = r1c2 + eps2;
r1ce = rsqrt(r1c2e);
rbc2e = dxbc_0_ * dxbc_0_ + dxbc_1_ * dxbc_1_ + dxbc_2_ * dxbc_2_ + eps2;
rbce = rsqrt(rbc2e);
mr1b3e = f01_ip_[isrc_].m_i_ * r1be * r1be * r1be;
a1c_0_wcache_ += mr1b3e * f01_jp_smem_[j_ + jstride_ * 7].m_j_ * dxb_0_ * (4.0 * r1ce + 1.25 * rbce - 0.25 * r1c2 / rbc2e * rbce + 0.25 * r1b2 / rbc2e * rbce) - 3.5 * (rbce * rbce * rbce) * r1be * f01_ip_[isrc_].m_i_ * f01_jp_smem_[j_ + jstride_ * 7].m_j_ * dxbc_0_ - mr1b3e * f01_jp_smem_[j_ + jstride_ * 7].m_j_ / m_0_ * (4.0 * (f01_ip_[isrc_].v_i_0_ * f01_jp_smem_[j_ + jstride_ * 7].v_j_0_ + f01_ip_[isrc_].v_i_1_ * f01_jp_smem_[j_ + jstride_ * 7].v_j_1_ + f01_ip_[isrc_].v_i_2_ * f01_jp_smem_[j_ + jstride_ * 7].v_j_2_) * dxb_0_ - 3.0 * (f01_ip_[isrc_].v_i_0_ * dxb_0_ + f01_ip_[isrc_].v_i_1_ * dxb_1_ + f01_ip_[isrc_].v_i_2_ * dxb_2_) * f01_jp_smem_[j_ + jstride_ * 7].v_j_0_ - 4.0 * (f01_jp_smem_[j_ + jstride_ * 7].v_j_0_ * dxb_0_ + f01_jp_smem_[j_ + jstride_ * 7].v_j_1_ * dxb_1_ + f01_jp_smem_[j_ + jstride_ * 7].v_j_2_ * dxb_2_) * f01_ip_[isrc_].v_i_0_);
a1c_1_wcache_ += mr1b3e * f01_jp_smem_[j_ + jstride_ * 7].m_j_ * dxb_1_ * (4.0 * r1ce + 1.25 * rbce - 0.25 * r1c2 / rbc2e * rbce + 0.25 * r1b2 / rbc2e * rbce) - 3.5 * (rbce * rbce * rbce) * r1be * f01_ip_[isrc_].m_i_ * f01_jp_smem_[j_ + jstride_ * 7].m_j_ * dxbc_1_ - mr1b3e * f01_jp_smem_[j_ + jstride_ * 7].m_j_ / m_0_ * (4.0 * (f01_ip_[isrc_].v_i_0_ * f01_jp_smem_[j_ + jstride_ * 7].v_j_0_ + f01_ip_[isrc_].v_i_1_ * f01_jp_smem_[j_ + jstride_ * 7].v_j_1_ + f01_ip_[isrc_].v_i_2_ * f01_jp_smem_[j_ + jstride_ * 7].v_j_2_) * dxb_1_ - 3.0 * (f01_ip_[isrc_].v_i_0_ * dxb_0_ + f01_ip_[isrc_].v_i_1_ * dxb_1_ + f01_ip_[isrc_].v_i_2_ * dxb_2_) * f01_jp_smem_[j_ + jstride_ * 7].v_j_1_ - 4.0 * (f01_jp_smem_[j_ + jstride_ * 7].v_j_0_ * dxb_0_ + f01_jp_smem_[j_ + jstride_ * 7].v_j_1_ * dxb_1_ + f01_jp_smem_[j_ + jstride_ * 7].v_j_2_ * dxb_2_) * f01_ip_[isrc_].v_i_1_);
a1c_2_wcache_ += mr1b3e * f01_jp_smem_[j_ + jstride_ * 7].m_j_ * dxb_2_ * (4.0 * r1ce + 1.25 * rbce - 0.25 * r1c2 / rbc2e * rbce + 0.25 * r1b2 / rbc2e * rbce) - 3.5 * (rbce * rbce * rbce) * r1be * f01_ip_[isrc_].m_i_ * f01_jp_smem_[j_ + jstride_ * 7].m_j_ * dxbc_2_ - mr1b3e * f01_jp_smem_[j_ + jstride_ * 7].m_j_ / m_0_ * (4.0 * (f01_ip_[isrc_].v_i_0_ * f01_jp_smem_[j_ + jstride_ * 7].v_j_0_ + f01_ip_[isrc_].v_i_1_ * f01_jp_smem_[j_ + jstride_ * 7].v_j_1_ + f01_ip_[isrc_].v_i_2_ * f01_jp_smem_[j_ + jstride_ * 7].v_j_2_) * dxb_2_ - 3.0 * (f01_ip_[isrc_].v_i_0_ * dxb_0_ + f01_ip_[isrc_].v_i_1_ * dxb_1_ + f01_ip_[isrc_].v_i_2_ * dxb_2_) * f01_jp_smem_[j_ + jstride_ * 7].v_j_2_ - 4.0 * (f01_jp_smem_[j_ + jstride_ * 7].v_j_0_ * dxb_0_ + f01_jp_smem_[j_ + jstride_ * 7].v_j_1_ * dxb_1_ + f01_jp_smem_[j_ + jstride_ * 7].v_j_2_ * dxb_2_) * f01_ip_[isrc_].v_i_2_);

                           }
                       }
                   }
                   __syncthreads();
                   f01_result_smem_[tid_].a1c_0_ = a1c_0_wcache_;f01_result_smem_[tid_].a1c_1_ = a1c_1_wcache_;f01_result_smem_[tid_].a1c_2_ = a1c_2_wcache_;
                   __syncthreads();
                   if (jstride_ > 1) {if (tid_ < kbdim_ / 2) {f01_result_smem_[tid_].a1c_0_ += f01_result_smem_[tid_ + kbdim_ / 2].a1c_0_;f01_result_smem_[tid_].a1c_1_ += f01_result_smem_[tid_ + kbdim_ / 2].a1c_1_;f01_result_smem_[tid_].a1c_2_ += f01_result_smem_[tid_ + kbdim_ / 2].a1c_2_;}// __syncthreads(); // this is not necessary since kbdim_ / 2 <= warp size.
}if (jstride_ > 2) {if (tid_ < kbdim_ / 4) {f01_result_smem_[tid_].a1c_0_ += f01_result_smem_[tid_ + kbdim_ / 4].a1c_0_;f01_result_smem_[tid_].a1c_1_ += f01_result_smem_[tid_ + kbdim_ / 4].a1c_1_;f01_result_smem_[tid_].a1c_2_ += f01_result_smem_[tid_ + kbdim_ / 4].a1c_2_;}// __syncthreads(); // this is not necessary since kbdim_ / 4 <= warp size.
}
                   __syncthreads();

#if 1
                   if (tid_ < nvalidthread_) {
                       int idstoff_ = njdiv_ * kbdim_ * ibid_ + jbid_ + njdiv_ * (tid_ % njhsub_);
                       float4 *srcbuf_ = (float4 *) (f01_result_smem_ + tid_);
                       float4 *dstbuf_ = (float4 *) (f01_result_ + idstoff_);
                       for (int icpy_ = 0; icpy_ < sizeof(f01_result_t) / sizeof(float4); icpy_++) {
                           dstbuf_[icpy_] = srcbuf_[icpy_];
                       }
                   }
#else
                   if (tid_ < nvalidthread_) {
                       f01_result_[idst_] = f01_result_smem_[tid_];
                   }
#endif
               }

/*
 * njdiv_    : # of result fragments per result packet.
 * njdiv_ru_ : njdiv_ rounded up to a power of two.
 * rbdim_    : # of result fragments to be reduced to (rbdim_ / njdiv_) result packets.
 */
               __global__ void
               f01_reducer(int njdiv_, int njdiv_ru_, f01_result_t *f01_result_, f01_result_t *f01_result_sub_)
               {
                   extern __shared__ char smembuf_[];
                   int rbdim_ = blockDim.x;
                   f01_result_t * f01_result_smem_ = (f01_result_t *)smembuf_;
                   f01_result_t * f01_result_smem_packed_ = (f01_result_t *)(smembuf_ + rbdim_ * sizeof(f01_result_t));
                   int tid_ = threadIdx.x;
                   int bid_ = blockIdx.x;
                   int isrc_ = rbdim_ * bid_ + tid_;
                   int ndst_ = rbdim_ / njdiv_;
                   int idst_ = ndst_ * bid_ + tid_;
                   __syncthreads();
                   #if 1
                     {
                         float4 *srcbuf_ = (float4 *)(f01_result_sub_ + rbdim_ * bid_);
                         float4 *dstbuf_ = (float4 *)smembuf_;
                         for (int icpy = 0; icpy < sizeof(f01_result_t)/sizeof(float4); icpy++) {
                             dstbuf_[tid_] = srcbuf_[tid_];
                             dstbuf_ += rbdim_;
                             srcbuf_ += rbdim_;
                         }
                     }
#else
                     f01_result_smem_[tid_] = f01_result_sub_[isrc_];
#endif

                   __syncthreads();

                   int n_ = njdiv_ru_;
                   while (n_ > 1) {
                       n_ /= 2;
                       int ipartner_ = tid_ + n_;
                       if (tid_ % njdiv_ < n_ && ipartner_ % njdiv_ru_ < njdiv_) {
                           f01_result_smem_[tid_].a1c_0_ += f01_result_smem_[ipartner_].a1c_0_;f01_result_smem_[tid_].a1c_1_ += f01_result_smem_[ipartner_].a1c_1_;f01_result_smem_[tid_].a1c_2_ += f01_result_smem_[ipartner_].a1c_2_;
                       }
                       __syncthreads(); // this is not necessary if rbdim_ <= warp size.
                   }
                   __syncthreads();
                   if (tid_ % njdiv_ == 0) {
                       int ipack_ = tid_ / njdiv_;
                       f01_result_smem_packed_[ipack_] = f01_result_smem_[tid_];
                   }
                   __syncthreads();
#if 1
                   {
                       float4 *srcbuf_ = (float4 *) f01_result_smem_packed_;
                       float4 *dstbuf_ = (float4 *) (f01_result_ + ndst_ * bid_);
                       if (tid_ < ndst_) {
                           for (int icpy = 0; icpy < sizeof(f01_result_t) / sizeof(float4); icpy++) {
                               dstbuf_[tid_] = srcbuf_[tid_];
                               dstbuf_ += ndst_;
                               srcbuf_ += ndst_;
                           }
                       }
                   }
#else
                   if (tid_ < ndst_) {
                       f01_result_[idst_] = f01_result_smem_packed_[tid_];
                   }
#endif
               }
